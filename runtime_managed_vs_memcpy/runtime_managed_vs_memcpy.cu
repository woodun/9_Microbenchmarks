#include "hip/hip_runtime.h"
#include <cstdio>
#include <ctime>
#include <vector>
#include <algorithm>
#include <stdlib.h>
// utilities
#include <hip/hip_runtime_api.h>
#include <time.h>
#include <sys/time.h>

/////////////////////////////L1 is enabled. "ALL_CCFLAGS += -Xptxas -dlcm=ca"
//////////////large vs small data.

void init_cpu_data(long long int* A, long long int size, long long int stride){
	
	for (long long int i = 0; i < size; i++){
		A[i]=1;
	}
	
	/*
	for (long long int i = 0; i < size - stride; i++){
		A[i]=(i + stride);
	}
			
	for (long long int i = size - stride; i < size; i++){
		A[i]=0;
	}
	*/
}

long long unsigned time_diff(timespec start, timespec end){
	struct timespec temp;
	if ((end.tv_nsec - start.tv_nsec) < 0){
		temp.tv_sec = end.tv_sec - start.tv_sec - 1;
		temp.tv_nsec = 1000000000 + end.tv_nsec - start.tv_nsec;
	} 
	else{
		temp.tv_sec = end.tv_sec - start.tv_sec;
		temp.tv_nsec = end.tv_nsec - start.tv_nsec;
	}
	
	long long unsigned time_interval_ns = temp.tv_nsec;
	long long unsigned time_interval_s = temp.tv_sec;
	time_interval_s = time_interval_s * 1000000000;
	
	return time_interval_s + time_interval_ns;
}

//__global__ void Page_visitor(long long int *A, long long int *B, long long int data_stride, long long int clock_count){
__global__ void Page_visitor(long long int *A, long long int data_stride, long long int clock_count){////load-compute -store
		
	/*
	long long int index = threadIdx.x;
	
	/////////////////////////////////time
	long long int start_time = 0;//////clock
	long long int end_time = 0;//////clock
	long long int time_interval = 0;//////clock
	
	if(index = 0){
		start_time= clock64();
	}
	__syncthreads();
	*/
	
	long long int index = (blockIdx.x * blockDim.x + threadIdx.x) * data_stride;
	
	long long int value = A[index];
	
	/*
	//////////////////////////////////////////////sleep
	long long int start_clock = clock64();
    long long int clock_offset = 0;
    while (clock_offset < clock_count)
    {
        clock_offset = clock64() - start_clock;
    }
	*/
	
	//////////////////////////////////////////////loop
	long long int clock_offset = 0;
    while (clock_offset < clock_count){/////////////////what's the time overhead for addition and multiplication?
        clock_offset++;
		value = value + threadIdx.x;
    }
	
	/*
	if(threadIdx.x == 0){/////%tid %ntid %laneid %warpid %nwarpid %ctaid %nctaid %smid %nsmid %gridid
		int smid = 1;
		asm("mov.u32 %0, %smid;" : "=r"(smid) );
		printf("blockIdx.x: %d, smid: %d\n", blockIdx.x, smid);
		if(blockIdx.x == 55){
			int nsmid = 1;
			asm("mov.u32 %0, %smid;" : "=r"(nsmid) );
			printf("nsmid: %d\n", nsmid);
		}
	}
	*/
	
    //d_o[0] = clock_offset;
	//////////////////////////////////////////////sleep
	
	A[index] = value;
	
	/*
	__syncthreads();
	__syncthreads();
	/////////////////////////////////time
	if(index = 0){
		start_time= clock64();
		time_interval = end_time - start_time;//////clock
	}	
	//B[0] = time_interval;
	*/
}

int main(int argc, char **argv)
{
	printf("\n");
	
    // set device
    hipDeviceProp_t device_prop;
    //long long int dev_id = findCudaDevice(argc, (const char **) argv);
	long long int dev_id = 0;
    checkCudaErrors(hipGetDeviceProperties(&device_prop, dev_id));
	
	int peak_clk = 1;//kHz
	checkCudaErrors(hipDeviceGetAttribute(&peak_clk, hipDeviceAttributeClockRate, dev_id));
	float clock_rate = (float) peak_clk;
	
	printf("clock_rate:%f\n", clock_rate);

    if (!device_prop.managedMemory) { 
        // This samples requires being run on a device that supports Unified Memory
        fprintf(stderr, "Unified Memory not supported on this device\n");

        exit(EXIT_WAIVED);
    }

    if (device_prop.computeMode == hipComputeModeProhibited)
    {
        // This sample requires being run with a default or process exclusive mode
        fprintf(stderr, "This sample requires a device in either default or process exclusive mode\n");

        exit(EXIT_WAIVED);
    }
	
	if (device_prop.concurrentManagedAccess == 1){
		printf("This device supports concurrent Managed Access.\n");
    }else{
		printf("This device does not support concurrent Managed Access.\n");
	}
	
	int value1 = 1;
	checkCudaErrors(hipDeviceGetAttribute(&value1, hipDeviceAttributeConcurrentManagedAccess, dev_id));
	printf("hipDeviceAttributeConcurrentManagedAccess = %d\n", value1);	
	
	//plain managed
	printf("###################\n#########################managed\n");
	for(long long int data_stride = 1 * 1 * 1; data_stride <= 1 * 512 * 1024; data_stride = data_stride * 2){
	for(long long int mod = 536870912; mod <= 536870912; mod = mod * 2){////134217728 = 1gb, 268435456 = 2gb, 536870912 = 4gb, 1073741824 = 8gb, 2147483648 = 16gb, 4294967296 = 32gb, 8589934592 = 64gb. (index)
	for(long long int clock_count = 1; clock_count <= 1024; clock_count = clock_count * 2){
		///////////////////////////////////////////////////////////////////CPU data begin		
		//long long int data_size = mod;
		long long int data_size = data_stride;
		data_size = data_size * 32;
		data_size = data_size * 64;
		//long long int iterations = mod / data_stride;////32 * 32 * 4 / 32 * 2 = 256
	
		long long int *CPU_data_in;
		//CPU_data_in = (long long int*)malloc(sizeof(long long int) * data_size);
		checkCudaErrors(hipMallocManaged(&CPU_data_in, sizeof(long long int) * data_size));/////////////using unified memory		
		init_cpu_data(CPU_data_in, data_size, data_stride);				
		///////////////////////////////////////////////////////////////////CPU data end	
	
		///////////////////////////////////////////////////////////////////GPU data in	
		//long long int *GPU_data_in;
		//checkCudaErrors(hipMalloc(&GPU_data_in, sizeof(long long int) * data_size));	
		//hipMemcpy(GPU_data_in, CPU_data_in, sizeof(long long int) * data_size, hipMemcpyHostToDevice);
		
		/*
		///////////////////////////////////////////////////////////////////GPU data out
		long long int *GPU_data_out;
		//checkCudaErrors(hipMalloc(&GPU_data_out, sizeof(long long int) * data_size));
		checkCudaErrors(hipMallocManaged(&GPU_data_out, sizeof(long long int) * data_size));/////////////using unified memory		
		*/
		
		/////////////////////////////////time
		struct timespec ts1;
		clock_gettime(CLOCK_REALTIME, &ts1);

		Page_visitor<<<32, 64>>>(CPU_data_in, data_stride, clock_count);///////////////1024 per block max
		///////////////////////////////////////////////////32 * 64 * 1 * 512 * 1024 = 8gb.
		hipDeviceSynchronize();
				
		/////////////////////////////////time
		struct timespec ts2;
		clock_gettime(CLOCK_REALTIME, &ts2);
		
		//printf("###################data_stride%lld#########################clock_count:%lld\n", data_stride, clock_count);
		//printf("*\n*\n*\nruntime:  %lluns\n", time_diff(ts1, ts2));
		printf("%llu\n", time_diff(ts1, ts2));
		
		//checkCudaErrors(hipFree(GPU_data_in));
		checkCudaErrors(hipFree(CPU_data_in));
		//free(CPU_data_in);
		checkCudaErrors(hipFree(GPU_data_out));
	}
	}
	}

	printf("###################\n#########################memcpy + kernel\n");
	for(long long int data_stride = 1 * 1 * 1; data_stride <= 1 * 512 * 1024; data_stride = data_stride * 2){////////question: when using smaller stride to migrate the whole 2M, is managed still better than memcpy?
	for(long long int mod = 536870912; mod <= 536870912; mod = mod * 2){////134217728 = 1gb, 268435456 = 2gb, 536870912 = 4gb, 1073741824 = 8gb, 2147483648 = 16gb, 4294967296 = 32gb, 8589934592 = 64gb. (index)
	for(long long int clock_count = 1; clock_count <= 1024; clock_count = clock_count * 2){
		///////////////////////////////////////////////////////////////////CPU data begin		
		//long long int data_size = mod;
		long long int data_size = data_stride;
		data_size = data_size * 32;
		data_size = data_size * 64;
		//long long int iterations = mod / data_stride;////32 * 32 * 4 / 32 * 2 = 256
	
		long long int *CPU_data_in;
		CPU_data_in = (long long int*)malloc(sizeof(long long int) * data_size);//////////////mempcy
		//checkCudaErrors(hipMallocManaged(&CPU_data_in, sizeof(long long int) * data_size));/////////////using unified memory		
		init_cpu_data(CPU_data_in, data_size, data_stride);				
		///////////////////////////////////////////////////////////////////CPU data end	
	
		///////////////////////////////////////////////////////////////////GPU data in	
		long long int *GPU_data_in;
		checkCudaErrors(hipMalloc(&GPU_data_in, sizeof(long long int) * data_size));	
		//hipMemcpy(GPU_data_in, CPU_data_in, sizeof(long long int) * data_size, hipMemcpyHostToDevice);///////moved down
		
		/*
		///////////////////////////////////////////////////////////////////GPU data out
		long long int *GPU_data_out;
		checkCudaErrors(hipMalloc(&GPU_data_out, sizeof(long long int) * data_size));//////////////mempcy
		//checkCudaErrors(hipMallocManaged(&GPU_data_out, sizeof(long long int) * data_size));/////////////using unified memory		
		*/
		
		/////////////////////////////////time
		struct timespec ts1;
		clock_gettime(CLOCK_REALTIME, &ts1);
		
		hipMemcpy(GPU_data_in, CPU_data_in, sizeof(long long int) * data_size, hipMemcpyHostToDevice);
		
		struct timespec ts2;
		clock_gettime(CLOCK_REALTIME, &ts2);
  
		Page_visitor<<<32, 64>>>(GPU_data_in, data_stride, clock_count);///////////////1024 per block max
		///////////////////////////////////////////////////32 * 512 * 2 = 32gb, 32 * 128 * 2 = 8gb, 32 * 64 * 2 = 4gb, 32 * 32 * 2 = 2gb
		hipDeviceSynchronize();
				
		/////////////////////////////////time
		struct timespec ts3;
		clock_gettime(CLOCK_REALTIME, &ts3);
		
		//printf("###################data_stride%lld#########################clock_count:%lld\n", data_stride, clock_count);
		//printf("*\n*\n*\nruntime:  %lluns\n", time_diff(ts1, ts2));
		printf("%llu %llu %llu\n", time_diff(ts1, ts2), time_diff(ts2, ts3), time_diff(ts1, ts3));		
		
		checkCudaErrors(hipFree(GPU_data_in));
		free(CPU_data_in);
		//checkCudaErrors(hipFree(CPU_data_in));		
		checkCudaErrors(hipFree(GPU_data_out));
	}
	}
	}
		
	/////////////////////what happens when migration full 2m pages (not just 64k)
	/////////////////////what happens when page fault intensity is smaller?		
		
    exit(EXIT_SUCCESS);
}