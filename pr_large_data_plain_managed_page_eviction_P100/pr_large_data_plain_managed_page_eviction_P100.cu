#include "hip/hip_runtime.h"
#include <cstdio>
#include <ctime>
#include <vector>
#include <algorithm>
#include <stdlib.h>
// utilities
#include <hip/hip_runtime_api.h>
#include <time.h>

/////////////////////////////change the data size to larger than 16 gb to test for different memories. L1 is enabled. "ALL_CCFLAGS += -Xptxas -dlcm=ca"

void init_cpu_data(long long int* A, long long int size, long long int stride, long long int mod){
	if(1){////////////normal
		for (long long int i = 0; i < size - stride; i = i + stride){
			A[i]=(i + stride);
		}
		
		//for (long long int i = 3; i < size - stride; i = i + stride){
		//	A[i]=(i + stride);
		//}
				
		A[size - stride]=0;
		//A[size - stride + 3]=0;
	}
	
	if(0){////////////reversed
		//for (long long int i = 0; i <= size - stride; i = i + stride){
		//	A[i]=(i - stride);
		//}
		
		for (long long int i = 3; i <= size - stride + 3; i = i + stride){
			A[i]=(i - stride);
		}
		
		//A[0]=size - stride;
		A[3]=size - stride + 3;
	}
	
	/////54521859 returned page fault starting point for 2147483648.
	///////////////////2147483648 - 54521859 = 2092961789.
	///////////////////2092961789 -4096 + 3 = 1996 * 1M = 15968 MB (out of 16280 MB out of 16384 MB)
	/////2202267651 returned page fault starting point for 4294967296
	///////////////////4294967296 - 2202267651 = 2092699645.
	///////////////////2092699645 -4096 + 3 = 1995.75 * 1M = 15966 MB (out of 16280 MB out of 16384 MB)
}

timespec time_diff(timespec start, timespec end){
	timespec temp;
	if ((end.tv_nsec - start.tv_nsec) < 0){
		temp.tv_sec = end.tv_sec - start.tv_sec - 1;
		temp.tv_nsec = 1000000000 + end.tv_nsec - start.tv_nsec;
	} 
	else{
		temp.tv_sec = end.tv_sec - start.tv_sec;
		temp.tv_nsec = end.tv_nsec - start.tv_nsec;
	}
	return temp;
}

__device__ void P_chasing2(int mark, long long int *A, long long int iterations, long long int *B, long long int starting_index, float clock_rate, long long int data_stride){		
	
	long long int j = starting_index;
	
	//long long int start_time = 0;//////clock
	//long long int end_time = 0;//////clock
	//start_time = clock64();//////clock
			
	for (long long int it = 0; it < iterations; it++){
		j = A[j];
	}
	
	//end_time = clock64();//////clock
	//long long int total_time = end_time - start_time;//////clock
	//printf("*\n*\n*\nruntime%d: %f\n", mark, total_time / ((double)clock_rate / 1000000));//////clock, average latency //////////the print will flush the L1?!
	
	B[mark] = j;
}

__global__ void tlb_latency_test(long long int *A, long long int iterations, long long int *B, float clock_rate, long long int mod, long long int data_stride){
			
	P_chasing2(1, A, iterations/8, B, 0, clock_rate, data_stride);
	P_chasing2(2, A, iterations/8, B, 536870912, clock_rate, data_stride);
	P_chasing2(3, A, iterations/8, B, 1073741824, clock_rate, data_stride);
	P_chasing2(4, A, iterations/8, B, 1610612736, clock_rate, data_stride);
	P_chasing2(5, A, iterations/8, B, 2147483648, clock_rate, data_stride);
	P_chasing2(6, A, iterations/8, B, 2684354560, clock_rate, data_stride);
	P_chasing2(7, A, iterations/8, B, 3221225472, clock_rate, data_stride);
	P_chasing2(8, A, iterations/8, B, 3758096384, clock_rate, data_stride);
	P_chasing2(1, A, iterations/8, B, 0, clock_rate, data_stride);
	P_chasing2(2, A, iterations/8, B, 536870912, clock_rate, data_stride);
	P_chasing2(3, A, iterations/8, B, 1073741824, clock_rate, data_stride);
	P_chasing2(4, A, iterations/8, B, 1610612736, clock_rate, data_stride);
	P_chasing2(5, A, iterations/8, B, 2147483648, clock_rate, data_stride);
	P_chasing2(6, A, iterations/8, B, 2684354560, clock_rate, data_stride);
	P_chasing2(7, A, iterations/8, B, 3221225472, clock_rate, data_stride);
	P_chasing2(8, A, iterations/8, B, 3758096384, clock_rate, data_stride);
	//P_chasing2(1, A, iterations, B, 0, clock_rate, data_stride);
	//P_chasing2(0, A, iterations, B, mod - data_stride + 3, clock_rate, data_stride);
	
	__syncthreads();
}

int main(int argc, char **argv)
{
    // set device
    hipDeviceProp_t device_prop;
    //int dev_id = findCudaDevice(argc, (const char **) argv);
	int dev_id = 0;
    checkCudaErrors(hipGetDeviceProperties(&device_prop, dev_id));
	
	int peak_clk = 1;//kHz
	checkCudaErrors(hipDeviceGetAttribute(&peak_clk, hipDeviceAttributeClockRate, dev_id));
	float clock_rate = (float) peak_clk;
	
	printf("clock_rate_out_kernel:%f\n", clock_rate);

    if (!device_prop.managedMemory) { 
        // This samples requires being run on a device that supports Unified Memory
        fprintf(stderr, "Unified Memory not supported on this device\n");

        exit(EXIT_WAIVED);
    }

    if (device_prop.computeMode == hipComputeModeProhibited)
    {
        // This sample requires being run with a default or process exclusive mode
        fprintf(stderr, "This sample requires a device in either default or process exclusive mode\n");

        exit(EXIT_WAIVED);
    }
	
	if (device_prop.concurrentManagedAccess == 1){
		printf("This device supports concurrent Managed Access.\n");
    }else{
		printf("This device does not support concurrent Managed Access.\n");
	}
	
	int value1 = 1;
	checkCudaErrors(hipDeviceGetAttribute(&value1, hipDeviceAttributeConcurrentManagedAccess, dev_id));
	printf("hipDeviceAttributeConcurrentManagedAccess = %d\n", value1);	
	
	///////////////////////////////////////////////////////////////////GPU data out
	long long int *GPU_data_out;
	checkCudaErrors(hipMalloc(&GPU_data_out, sizeof(long long int) * 256));			
	
	int counter = 0;	
	for(long long int data_stride = 1 * 256 * 1024; data_stride <= 1 * 256 * 1024; data_stride = data_stride * 2){/////////32mb stride

	//plain managed
	printf("*\n*\n*\n plain managed\n");
	for(long long int mod = 4294967296; mod <= 4294967296; mod = mod * 2){////134217728 = 1gb, 268435456 = 2gb, 536870912 = 4gb, 1073741824 = 8gb, 2147483648 = 16gb, 4294967296 = 32gb, 8589934592 = 64gb. (index)
		counter++;
		///////////////////////////////////////////////////////////////////CPU data begin
		long long int data_size = mod;
		long long int iterations = mod / data_stride;
			
		long long int *CPU_data_in;
		//CPU_data_in = (int*)malloc(sizeof(int) * data_size);
		checkCudaErrors(hipMallocManaged(&CPU_data_in, sizeof(long long int) * data_size));/////////////using unified memory		
		init_cpu_data(CPU_data_in, data_size, data_stride, mod);		
		///////////////////////////////////////////////////////////////////CPU data end	
	
		///////////////////////////////////////////////////////////////////GPU data in	
		//int *GPU_data_in;
		//checkCudaErrors(hipMalloc(&GPU_data_in, sizeof(int) * data_size));	
		//hipMemcpy(GPU_data_in, CPU_data_in, sizeof(int) * data_size, hipMemcpyHostToDevice);		
		
		printf("###################data_stride%lld#########################\n", data_stride);
		printf("###############Mod%lld##############%lld\n", mod, iterations);		
						
		tlb_latency_test<<<1, 1>>>(CPU_data_in, iterations, GPU_data_out, clock_rate, mod, data_stride);///kernel is here	
		hipDeviceSynchronize();
		
		init_cpu_data(CPU_data_in, data_size, data_stride, mod);
		
		//checkCudaErrors(hipFree(GPU_data_in));
		checkCudaErrors(hipFree(CPU_data_in));
		//free(CPU_data_in);		
	}
	}
			
	checkCudaErrors(hipFree(GPU_data_out));	
	
    exit(EXIT_SUCCESS);
}