#include "hip/hip_runtime.h"
#include <cstdio>
#include <ctime>
#include <vector>
#include <algorithm>
#include <stdlib.h>
// utilities
#include <hip/hip_runtime_api.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_cooperative_groups.h>

using namespace cooperative_groups;

/////////////////////////////L1 is enabled. "ALL_CCFLAGS += -Xptxas -dlcm=ca"
//////////////large vs small data.

void init_cpu_data(long long int* A, long long int size, double stride){
	
	for (long long int i = 0; i < size; i++){
		A[i]=1;
	}
	
	/*
	for (long long int i = 0; i < size - stride; i++){
		A[i]=(i + stride);
	}
			
	for (long long int i = size - stride; i < size; i++){
		A[i]=0;
	}
	*/
}

__global__ void gpu_initialization(long long int *A, double data_stride, long long int data_size){			

	long long int index = (blockIdx.x * blockDim.x + threadIdx.x);
	long long int thread_num =  gridDim.x * blockDim.x;
	
	for(long long int it = 0; it < data_size; it = it + thread_num){
		A[index + it]=23;
	}
}

long long unsigned time_diff(timespec start, timespec end){
	struct timespec temp;
	if ((end.tv_nsec - start.tv_nsec) < 0){
		temp.tv_sec = end.tv_sec - start.tv_sec - 1;
		temp.tv_nsec = 1000000000 + end.tv_nsec - start.tv_nsec;
	} 
	else{
		temp.tv_sec = end.tv_sec - start.tv_sec;
		temp.tv_nsec = end.tv_nsec - start.tv_nsec;
	}
	
	long long unsigned time_interval_ns = temp.tv_nsec;
	long long unsigned time_interval_s = temp.tv_sec;
	time_interval_s = time_interval_s * 1000000000;
	
	return time_interval_s + time_interval_ns;
}

__global__ void baseline(long long int *A1, long long int *B1, double data_stride, long long int clock_count){////load-compute-store
			
	//thread_block block = this_thread_block();	
	
	double temp = (blockIdx.x * 512 + threadIdx.x) * data_stride;
	long long int index = __double2ll_rd(temp);
	
	long long int value1;
	
	double temp2 = (blockIdx.x * 512 + threadIdx.x * 16) * data_stride;
	long long int prefetch_index = __double2ll_rd(temp2);

	value1 = A1[index];
		
	long long int clock_offset = 0;
    while (clock_offset < clock_count){/////////////////what's the time overhead for addition and multiplication?
        clock_offset++;
		value1 = value1 + threadIdx.x;
    }

	B1[index] = value1;	
}

//262144 2m
//__global__ void Page_visitor(long long int *A, long long int *B, long long int data_stride, long long int clock_count){
__global__ void page_visitor(long long int *A1, long long int *B1, double data_stride, long long int clock_count){////vertical
			
	//thread_block block = this_thread_block();	
	
	double temp = (blockIdx.x * 512 + threadIdx.x) * data_stride;
	long long int index = __double2ll_rd(temp);
	
	long long int value1;
	
	double temp2 = (blockIdx.x * 512 + threadIdx.x * 16) * data_stride;//////////////vertical	
	long long int prefetch_index = __double2ll_rd(temp2);	
	
	//if(threadIdx.x < 480){
	if(threadIdx.x > 31){
	//if(0){
		value1 = A1[index];
		
	}else{
		value1 = A1[index];		
		B1[prefetch_index] = 0;
	}
	
	//block.sync();
		
	long long int clock_offset = 0;
    while (clock_offset < clock_count){/////////////////what's the time overhead for addition and multiplication?
        clock_offset++;
		value1 = value1 + threadIdx.x;
    }

	B1[index] = value1;	
}

__global__ void page_visitor2(long long int *A1, long long int *B1, double data_stride, long long int clock_count, long long int offset){////horizontal
			
	//thread_block block = this_thread_block();	
	
	double temp = (blockIdx.x * 512 + threadIdx.x) * data_stride;
	long long int index = __double2ll_rd(temp);
	
	long long int value1;
	long long int value2;	
	
	double temp2 = ( (blockIdx.x + offset) * 512 + threadIdx.x * 16) * data_stride;//////////////horizontal
	long long int prefetch_index = __double2ll_rd(temp2);	
	
	//if(threadIdx.x < 480){
	if(threadIdx.x > 31){
	//if(0){
		value1 = A1[index];
		
	}else{
		value1 = A1[index];
		if(blockIdx.x < 4194304 - offset){
		value2 = A1[prefetch_index];
		}
	}
	
	//block.sync();
		
	long long int clock_offset = 0;
    while (clock_offset < clock_count){/////////////////what's the time overhead for addition and multiplication?
        clock_offset++;
		value1 = value1 + threadIdx.x;
    }

	if(threadIdx.x > 31){
		B1[index] = value1;	
	}else{
		B1[index] = value1;
		if(blockIdx.x < 4194304 - offset){
		B1[prefetch_index] = value2;
		}		
	}
}

__global__ void page_visitor3(long long int *A1, long long int *B1, double data_stride, long long int clock_count, long long int offset, long long int rate){////vertical with offset
			
	//thread_block block = this_thread_block();	
	
	double temp = (blockIdx.x * 512 + threadIdx.x) * data_stride;
	long long int index = __double2ll_rd(temp);
	
	long long int value1;	
	
	double temp2 = ( (blockIdx.x + offset) * 512 + threadIdx.x * 16) * data_stride;//////////////horizontal
	long long int prefetch_index = __double2ll_rd(temp2);	
	
	value1 = A1[index];		
	
	if(threadIdx.x < 32){			
		if(blockIdx.x < 4194304 - offset){
			if(blockIdx.x % 8 == 0){
				B1[prefetch_index] = 0;
			}
		}
	}
	
	/*
	if(blockIdx.x < 4194304 - offset){
		B1[index] = 0;			
	}
	*/
	
	//block.sync();
	//__threadfence_block();
		
	long long int clock_offset = 0;
    while (clock_offset < clock_count){/////////////////what's the time overhead for addition and multiplication?
        clock_offset++;
		value1 = value1 + threadIdx.x;
    }
	
	//__threadfence_block();

	B1[index] = value1;
}

__global__ void page_visitor4(long long int *A1, long long int *B1, double data_stride, long long int clock_count, long long int offset, long long int time){////vertical with offset and time
			
	thread_block block = this_thread_block();	
	
	double temp = (blockIdx.x * 512 + threadIdx.x) * data_stride;
	long long int index = __double2ll_rd(temp);
	
	long long int value1;	
	
	double temp2 = ( (blockIdx.x + offset) * 512 + threadIdx.x * 16) * data_stride;//////////////horizontal
	long long int prefetch_index = __double2ll_rd(temp2);	
	
	//if(threadIdx.x < 480){
	if(threadIdx.x > 31){
	//if(0){
		value1 = A1[index];
		
	}else{
		value1 = A1[index];
	}
			
	long long int clock_offset = 0;
    while (clock_offset < clock_count - time){/////////////////what's the time overhead for addition and multiplication?
        clock_offset++;
		value1 = value1 + threadIdx.x;
    }
	
	if(threadIdx.x < 32){
		if(blockIdx.x < 4194304 - offset){//////////////how about negative offset?
		B1[prefetch_index] = 0;
		}
	}
	
	block.sync();////////////////////////////////////try to sync here?
	
	clock_offset = 0;
    while (clock_offset < time){/////////////////what's the time overhead for addition and multiplication?
        clock_offset++;
		value1 = value1 + threadIdx.x;
    }
	
	//block.sync();////////////////////////////////////try to sync here?

	if(threadIdx.x > 31){
		B1[index] = value1;	
	}else{
		B1[index] = value1;
	}
}

__global__ void page_visitor5(long long int *A1, long long int *B, double data_stride, long long int clock_count, long long int offset){////load-compute-store
			
	//thread_block block = this_thread_block();
	
	double temp = (blockIdx.x * 512 + (threadIdx.x - 32) ) * data_stride;
	long long int index = __double2ll_rd(temp);

	long long int value1;
	
	double temp2 = ( (blockIdx.x + offset) * 512 + threadIdx.x * 16) * data_stride;
	long long int prefetch_index = __double2ll_rd(temp2);
	long long int value2;
	
	if(threadIdx.x > 31){
		value1 = A1[index];
		
		//__threadfence_block();
	}
	
	//block.sync();/////////////how to vote inside/outside blocks?	
		
	if(threadIdx.x < 32){
		if(blockIdx.x < 4194304 - offset){//////////////questions: how about negative offset?		
			B[prefetch_index] = 0;//////////////////////questions: try for horizontal using proxy.			
			
			//__threadfence_block();
		}		
	}	
	
	//block.sync();
		
	if(threadIdx.x > 31){
		//////////////////////////////////////////////loop
		long long int clock_offset2 = 0;
		while (clock_offset2 < clock_count){/////////////////what's the time overhead for addition and multiplication?
			clock_offset2++;
			value1 = value1 + threadIdx.x;
		}
	}
	
	//block.sync();
	//__threadfence_block();
	
	if(threadIdx.x > 31){
		B[index] = value1;
	}
}

__global__ void page_visitor7(long long int *A1, long long int *B, double data_stride, long long int clock_count, long long int offset, long long int time){////load-compute-store
			
	//thread_block block = this_thread_block();
	__shared__ int signal;
	
	double temp = (blockIdx.x * 512 + (threadIdx.x - 32) ) * data_stride;
	long long int index = __double2ll_rd(temp);

	long long int value1;
	
	double temp2 = ( (blockIdx.x + offset) * 512 + threadIdx.x * 16) * data_stride;
	long long int prefetch_index = __double2ll_rd(temp2);
	long long int value2;
	
	if(threadIdx.x > 31){//////////////////question: for non-proxy, remove thread limit?
		value1 = A1[index];
		
		//__threadfence_block();
	}
	
	//block.sync();/////////////how to vote inside/outside blocks?	
	
	if(threadIdx.x > 31){
		//////////////////////////////////////////////loop
		long long int clock_offset1 = 0;
		while (clock_offset1 < clock_count - time){/////////////////what's the time overhead for addition and multiplication?
			clock_offset1++;
			value1 = value1 + threadIdx.x;
		}
	}
	
	if(threadIdx.x < 33){
		if(threadIdx.x > 31){
			signal = value1;
		}
		__threadfence_block();
		
		if(threadIdx.x < 32){//////////////////proxy
			if(blockIdx.x < 4194304 - offset){//////////////questions: how about negative offset?		
				B[prefetch_index] = 0;//////////////////////questions: try for horizontal using proxy.			
			}
		}
	}
	
	if(threadIdx.x > 31){
		//////////////////////////////////////////////loop
		long long int clock_offset2 = 0;
		while (clock_offset2 < time){/////////////////what's the time overhead for addition and multiplication?
			clock_offset2++;
			value1 = value1 + threadIdx.x;
		}
	}
	
	//block.sync();
	//__threadfence_block();
	
	if(threadIdx.x > 31){
		B[index] = value1;
	}
}

 __global__ void page_visitor6(long long int *A1, long long int *B, double data_stride, long long int clock_count, long long int offset){////load-compute-store
		
	__shared__ int signal;
	__shared__ int signal2;
	__shared__ int trigger;
	
	signal = 0;
	signal2 = 0;
	trigger = 0;
	
	thread_block block = this_thread_block();
	
	double temp = (blockIdx.x * 512 + (threadIdx.x - 32) ) * data_stride;
	long long int index = __double2ll_rd(temp);

	long long int value1;
	
	double temp2 = ( (blockIdx.x + offset) * 512 + threadIdx.x * 16) * data_stride;
	long long int prefetch_index = __double2ll_rd(temp2);
	long long int value2;
	
	
	if(threadIdx.x > 31){
		signal = 1;
		value1 = A1[index];
	}
	
	//block.sync();/////////////how to vote inside/outside blocks?
	
	if(threadIdx.x < 32){
		if(blockIdx.x < 4194304 - offset){//////////////questions: how about negative offset?
			if(signal == 1){
				B[prefetch_index] = 0;//////////////////////questions: try for horizontal using proxy.
				signal2 = 1;
			}
		}
	}
	
	//block.sync();
	
	if(threadIdx.x > 31){
		if(signal2 == 1){
			//////////////////////////////////////////////loop
			long long int clock_offset2 = 0;
			while (clock_offset2 < clock_count){/////////////////what's the time overhead for addition and multiplication?
				clock_offset2++;
				value1 = value1 + threadIdx.x;
			}
		}
	}
	
	//block.sync();
	
	if(threadIdx.x > 31){
		if(signal == 1){
			B[index] = value1;
		}
	}
}
 
 
int main(int argc, char **argv)
{
	printf("\n");
	
    // set device
    hipDeviceProp_t device_prop;
    //long long int dev_id = findCudaDevice(argc, (const char **) argv);
	long long int dev_id = 0;
    checkCudaErrors(hipGetDeviceProperties(&device_prop, dev_id));
	
	//int peak_clk = 1;//kHz
	//checkCudaErrors(hipDeviceGetAttribute(&peak_clk, hipDeviceAttributeClockRate, dev_id));
	//float clock_rate = (float) peak_clk;
	//printf("clock_rate:%f\n", clock_rate);

    if (!device_prop.managedMemory) { 
        // This samples requires being run on a device that supports Unified Memory
        fprintf(stderr, "Unified Memory not supported on this device\n");

        exit(EXIT_WAIVED);
    }

    if (device_prop.computeMode == hipComputeModeProhibited)
    {
        // This sample requires being run with a default or process exclusive mode
        fprintf(stderr, "This sample requires a device in either default or process exclusive mode\n");

        exit(EXIT_WAIVED);
    }
	
	/*
	if (device_prop.concurrentManagedAccess == 1){
		printf("This device supports concurrent Managed Access.\n");
    }else{
		printf("This device does not support concurrent Managed Access.\n");
	}
	*/
	
	int value1 = 1;
	checkCudaErrors(hipDeviceGetAttribute(&value1, hipDeviceAttributeConcurrentManagedAccess, dev_id));
	//printf("hipDeviceAttributeConcurrentManagedAccess = %d\n", value1);	
	
	//changeable: block size, number of blocks(8192 * 512 max?), data size, data stride, computation length (best not too long nor too short?), ways of implementation (stand alone or incorporated? verticle or parallel prefetch?), span of prefetching (fetch for other blocks even verticlly)? additional warp with vote, different vote and prefetch locations.
	//plain managed
	//when was 64k and 4k pages used?
	//how to decrease the overhead of sync?
	//printf("###################\n#########################managed\n");
	
	/*
	for(long long int factor = 1; factor <= 128; factor = factor * 2){
	for(double data_stride = 0.25 * factor; data_stride <= 1 * 1 * 4 * factor; data_stride = data_stride * 2){///134217728 = 1gb, 268435456 = 2gb, 536870912 = 4gb, 1073741824 = 8gb, 2147483648 = 16gb, 4294967296 = 32gb, 8589934592 = 64gb. (index)
	for(long long int clock_count = 64; clock_count <= 16384; clock_count = clock_count * 2){
	*/

	///*
	printf("############approach\n");
	for(long long int time = 32; time <= 32; time = time * 2){
	printf("\n####################time: %llu\n", time);
	
	long long int coverage2 = 0;
	for(long long int coverage = 1; coverage <= 1; coverage = coverage * 2){///////////////8192 is 2m.
		//coverage2++;
		//if(coverage2 == 2){
		//	coverage = 1;
		//}
		printf("############coverage: %llu\n", coverage);
		
	for(long long int rate = 1; rate <= 1; rate = rate * 2){
		printf("############rate: %llu\n", rate);
		
	long long int offset2 = 0;
	for(long long int offset = 0; offset <= 0; offset = offset + 2){///////8
	//for(long long int offset = 0; offset <= 256; offset = offset + 8){
		//offset2++;
		//if(offset2 == 2){
		//	offset = 1;
		//}
	//printf("############offset: %llu\n", offset);
	
	for(long long int factor = 16384; factor <= 16384; factor = factor * 2){/////////////16384 (128k) max
	//printf("####################factor: %llu\n", factor);
	
	for(double data_stride = 1 * 1 * 1 * factor; data_stride <= 1 * 1 * 1 * factor; data_stride = data_stride * 2){///134217728 = 1gb, 268435456 = 2gb, 536870912 = 4gb, 1073741824 = 8gb, 2147483648 = 16gb, 4294967296 = 32gb, 8589934592 = 64gb. (index)
	//printf("\n");

	for(long long int clock_count = 8192; clock_count <= 8192; clock_count = clock_count * 2){
		
	///long long int time2 = time;
	//if(time2 > clock_count){
	//	time2 = clock_count;
	//}

		///////////////////////////////////////////////////////////////////CPU data begin
		double temp = data_stride * 512;
		long long int data_size = (long long int) temp;
		data_size = data_size * 8192 * 512 / factor;
		
		long long int *CPU_data_in1;
		checkCudaErrors(hipMallocManaged(&CPU_data_in1, sizeof(long long int) * data_size));/////////////using unified memory
		///////////////////////////////////////////////////////////////////CPU data end
		
		long long int *GPU_data_out1;
		checkCudaErrors(hipMallocManaged(&GPU_data_out1, sizeof(long long int) * data_size));/////////////using unified memory
		///////////////////////////////////////////////////////////////////GPU data out	end
		
		if(1){
			double scale = 1;
			if(data_stride < 1){
				scale = data_stride;/////////make sure threadIdx is smaller than data_size in the initialization
			}
			
			gpu_initialization<<<8192 * 512 * scale / factor, 512>>>(GPU_data_out1, data_stride, data_size);///1024 per block max
			hipDeviceSynchronize();
			gpu_initialization<<<8192 * 512 * scale / factor, 512>>>(CPU_data_in1, data_stride, data_size);///1024 per block max
			hipDeviceSynchronize();
		}else{
			init_cpu_data(GPU_data_out1, data_size, data_stride);
			init_cpu_data(CPU_data_in1, data_size, data_stride);		
		}
		
		/////////////////////////////////time
		struct timespec ts1;
		clock_gettime(CLOCK_REALTIME, &ts1);

		////may want to use more thread to see clock_count effect
		//page_visitor7<<<8192 * 512 / factor, 512 + 32>>>(CPU_data_in1, GPU_data_out1, data_stride, clock_count, offset, time);
		//page_visitor5<<<8192 * 512 / factor, 512 + 32>>>(CPU_data_in1, GPU_data_out1, data_stride, clock_count, offset);
		page_visitor3<<<8192 * 512 / factor, 512>>>(CPU_data_in1, GPU_data_out1, data_stride, clock_count, offset, rate);
		//page_visitor3<<<8192 * 512 / factor, 512>>>(CPU_data_in1, GPU_data_out1, data_stride, clock_count, offset);
		hipDeviceSynchronize();
				
		/////////////////////////////////time
		struct timespec ts2;
		clock_gettime(CLOCK_REALTIME, &ts2);
		
		//printf("###################data_stride%lld#########################clock_count:%lld\n", data_stride, clock_count);
		//printf("*\n*\n*\nruntime:  %lluns\n", time_diff(ts1, ts2));
		printf("%llu ", time_diff(ts1, ts2));
		fflush(stdout);
		
		checkCudaErrors(hipFree(CPU_data_in1));		
		checkCudaErrors(hipFree(GPU_data_out1));
	}
	}
	}
	}
	}
	}
	}
	//*/
	
	///*
	printf("\n############baseline\n");
	for(long long int factor = 16384; factor <= 16384; factor = factor * 2){/////////////16384 max
	//printf("####################factor: %llu\n", factor);
		
	for(double data_stride = 1 * 1 * 1 * factor; data_stride <= 1 * 1 * 1 * factor; data_stride = data_stride * 2){///134217728 = 1gb, 268435456 = 2gb, 536870912 = 4gb, 1073741824 = 8gb, 2147483648 = 16gb, 4294967296 = 32gb, 8589934592 = 64gb. (index) 262144 = 2m. 16384 = 128k.
	//printf("\n");
	
	for(long long int clock_count = 8192; clock_count <= 8192; clock_count = clock_count * 2){///////8192 all factors variable4

		///////////////////////////////////////////////////////////////////CPU data begin
		double temp = data_stride * 512;
		long long int data_size = (long long int) temp;
		data_size = data_size * 8192 * 512 / factor;
		
		long long int *CPU_data_in1;
		checkCudaErrors(hipMallocManaged(&CPU_data_in1, sizeof(long long int) * data_size));/////////////using unified memory
		///////////////////////////////////////////////////////////////////CPU data end
		
		long long int *GPU_data_out1;
		checkCudaErrors(hipMallocManaged(&GPU_data_out1, sizeof(long long int) * data_size));/////////////using unified memory
		///////////////////////////////////////////////////////////////////GPU data out	end
		
		if(1){
			double scale = 1;
			if(data_stride < 1){
				scale = data_stride;/////////make sure threadIdx is smaller than data_size in the initialization
			}
			
			gpu_initialization<<<8192 * 512 * scale / factor, 512>>>(GPU_data_out1, data_stride, data_size);////////////1024 per block max
			hipDeviceSynchronize();
			gpu_initialization<<<8192 * 512 * scale / factor, 512>>>(CPU_data_in1, data_stride, data_size);//////////1024 per block max
			hipDeviceSynchronize();
		}else{
			init_cpu_data(GPU_data_out1, data_size, data_stride);
			init_cpu_data(CPU_data_in1, data_size, data_stride);		
		}
		
		/////////////////////////////////time
		struct timespec ts1;
		clock_gettime(CLOCK_REALTIME, &ts1);

		////may want to use more thread to see clock_count effect
		baseline<<<8192 * 512 / factor, 512>>>(CPU_data_in1, GPU_data_out1, data_stride, clock_count);///1024 per block max
		hipDeviceSynchronize();
				
		/////////////////////////////////time
		struct timespec ts2;
		clock_gettime(CLOCK_REALTIME, &ts2);
		
		//printf("###################data_stride%lld#########################clock_count:%lld\n", data_stride, clock_count);
		//printf("*\n*\n*\nruntime:  %lluns\n", time_diff(ts1, ts2));
		printf("%llu ", time_diff(ts1, ts2));
		fflush(stdout);
		
		checkCudaErrors(hipFree(CPU_data_in1));		
		checkCudaErrors(hipFree(GPU_data_out1));
	}	
	}	
	}
	//*/
	printf("\n");
	
	exit(EXIT_SUCCESS);
}