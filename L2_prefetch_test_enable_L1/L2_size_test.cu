#include "hip/hip_runtime.h"
#include <cstdio>
#include <ctime>
#include <vector>
#include <algorithm>
#include <stdlib.h>
// utilities
#include <hip/hip_runtime_api.h>
#include <time.h>

///////////when L1 is enabled. every miss will cause L2 to fetch 4 cache lines * 32 bytes to fill the 1 cache line * 128 byte in L1. Is it true? Change the starting offset to see.

void init_cpu_data(int* A, int size, int stride, int mod){
	for (int i = 0; i < size; ++i){
		A[i]=(i + stride) % mod;
   	}
}

//////////min page size 4kb = 4096b = 32 * 128.
__device__ void P_chasing(int mark, int *A, int iterations, int *B, int starting_index, float clock_rate, int data_stride){
	
	int k = starting_index;/////make them in the same page, and miss near in cache lines
	for (int it = 0; it < iterations; it++){/////////////warmup
		k = A[k];
	}
	
	int j = starting_index;/////make them in the same page, and miss near in cache lines
	
	long long int start_time = 0;//////clock
	long long int end_time = 0;//////clock
	start_time = clock64();//////clock
			
	for (int it = 0; it < iterations; it++){
		j = A[j];
	}
	
	end_time=clock64();//////clock
	long long int total_time = end_time - start_time;//////clock
	printf("inside%d:%fms\n", mark, (total_time / (float)clock_rate) / ((float)iterations));//////clock, average latency
	
	B[0] = j;
}

__global__ void tlb_latency_test(int *A, int iterations, int *B, float clock_rate, int mod, int data_stride){	
	
	P_chasing(0, A, iterations, B, 0, clock_rate, data_stride);////////saturate the L1 not L2
	P_chasing(7, A, iterations, B, 7, clock_rate, data_stride);////////access different parts of the 128 byte on L2
	P_chasing(9, A, iterations, B, 31, clock_rate, data_stride);////////access different parts of the 128 byte on L2
	
	 __syncthreads();
}

int main(int argc, char **argv)
{
	printf("\n");
	
    // set device
    hipDeviceProp_t device_prop;
    //int dev_id = findCudaDevice(argc, (const char **) argv);
	int dev_id = 0;
    checkCudaErrors(hipGetDeviceProperties(&device_prop, dev_id));
	
	int peak_clk = 1;//kHz
	checkCudaErrors(hipDeviceGetAttribute(&peak_clk, hipDeviceAttributeClockRate, dev_id));
	float clock_rate = (float) peak_clk;
	
	//printf("clock_rate_out_kernel:%f\n", clock_rate);

    if (!device_prop.managedMemory) { 
        // This samples requires being run on a device that supports Unified Memory
        fprintf(stderr, "Unified Memory not supported on this device\n");

        exit(EXIT_WAIVED);
    }

    if (device_prop.computeMode == hipComputeModeProhibited)
    {
        // This sample requires being run with a default or process exclusive mode
        fprintf(stderr, "This sample requires a device in either default or process exclusive mode\n");

        exit(EXIT_WAIVED);
    }
	
	///////////////////////////////////////////////////////////////////GPU data out
	int *GPU_data_out;
	checkCudaErrors(hipMalloc(&GPU_data_out, sizeof(int) * 1));
	
	printf("################fixing data range, changing stride############################\n");	
	for(int data_stride = 32; data_stride <= 32; data_stride = data_stride + 1){/////////stride shall be L1 cache line size.
		printf("###################data_stride%d#########################\n", data_stride);
	//for(int mod = 1024 * 256 * 2; mod > 0; mod = mod - 32 * 1024){/////kepler L2 1.5m
	for(int mod = 1024 * 256 * 1; mod >= 1024 * 256 * 1; mod = mod / 2){/////kepler L2 1.5m ////////saturate the L1 not L2
		///////////////////////////////////////////////////////////////////CPU data begin
		int data_size = 512 * 1024 * 30;/////size = iteration * stride = 30 2mb pages.		
		//int iterations = data_size / data_stride;
		int iterations = mod / data_stride;
	
		int *CPU_data_in;
		CPU_data_in = (int*)malloc(sizeof(int) * data_size);	
		init_cpu_data(CPU_data_in, data_size, data_stride, mod);
		///////////////////////////////////////////////////////////////////CPU data end	
	
		///////////////////////////////////////////////////////////////////GPU data in	
		int *GPU_data_in;
		checkCudaErrors(hipMalloc(&GPU_data_in, sizeof(int) * data_size));	
		hipMemcpy(GPU_data_in, CPU_data_in, sizeof(int) * data_size, hipMemcpyHostToDevice);
		
		tlb_latency_test<<<1, 1>>>(GPU_data_in, iterations, GPU_data_out, clock_rate, mod, data_stride);//////////////////////////////////////////////kernel is here	
		hipDeviceSynchronize();
		
		checkCudaErrors(hipFree(GPU_data_in));
		free(CPU_data_in);
	}
		printf("############################################\n\n");
	}
			
	checkCudaErrors(hipFree(GPU_data_out));	
	//free(CPU_data_out);
	
    exit(EXIT_SUCCESS);
}
