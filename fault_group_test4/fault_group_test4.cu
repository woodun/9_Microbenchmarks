#include "hip/hip_runtime.h"
#include <cstdio>
#include <ctime>
#include <vector>
#include <algorithm>
#include <stdlib.h>
// utilities
#include <hip/hip_runtime_api.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime_api.h>

using namespace cooperative_groups;

/////////////////////////////L1 is enabled. "ALL_CCFLAGS += -Xptxas -dlcm=ca"
//////////////large vs small data.
//////creating 2 blocks doing exactly the same thing? method in the blog.
//////nvprof --profile-from-start off --print-gpu-trace --log-file 4warpsall.txt --csv ./fault_group_test15

void init_cpu_data(long long int* A, long long int size, double stride){
	
	for (long long int i = 0; i < size; i++){
		A[i]=1;
	}
	
	/*
	for (long long int i = 0; i < size - stride; i++){
		A[i]=(i + stride);
	}
			
	for (long long int i = size - stride; i < size; i++){
		A[i]=0;
	}
	*/
}

__global__ void gpu_initialization(long long int *A, double data_stride, long long int data_size){			

	long long int index = (blockIdx.x * blockDim.x + threadIdx.x);
	long long int thread_num =  gridDim.x * blockDim.x;
	
	for(long long int it = 0; it < data_size; it = it + thread_num){
		A[index + it]=23;
	}
}

long long unsigned time_diff(timespec start, timespec end){
	struct timespec temp;
	if ((end.tv_nsec - start.tv_nsec) < 0){
		temp.tv_sec = end.tv_sec - start.tv_sec - 1;
		temp.tv_nsec = 1000000000 + end.tv_nsec - start.tv_nsec;
	} 
	else{
		temp.tv_sec = end.tv_sec - start.tv_sec;
		temp.tv_nsec = end.tv_nsec - start.tv_nsec;
	}
	
	long long unsigned time_interval_ns = temp.tv_nsec;
	long long unsigned time_interval_s = temp.tv_sec;
	time_interval_s = time_interval_s * 1000000000;
	
	return time_interval_s + time_interval_ns;
}

//#define stride 512

__global__ void stream_thread(long long int *ptr, const long long int size, 
                              long long int *output, const long long int val) 
{ 
  long long int tid = threadIdx.x + blockIdx.x * blockDim.x; 
  long long int n = size / sizeof(long long int); 
  long long int accum = 0; 

  for(; tid < n; tid += blockDim.x * gridDim.x) 
    if (1) accum += ptr[tid]; 
      else ptr[tid] = val;  

  if (1) 
    output[threadIdx.x + blockIdx.x * blockDim.x] = accum; 
}


//#define STRIDE_64K 65536

__global__ void stream_warp(long long int *ptr, const long long int size, long long int *output, const long long int val, long long int STRIDE_64K) 
{ 
  int lane_id = threadIdx.x & 31; 
  long long int warp_id = (threadIdx.x + blockIdx.x * blockDim.x) >> 5; 
  int warps_per_grid = (blockDim.x * gridDim.x) >> 5; 
  long long int warp_total = (size + STRIDE_64K-1) / STRIDE_64K; 

  long long int n = size / sizeof(long long int); 
  long long int accum = 0; 

  for(; warp_id < warp_total; warp_id += warps_per_grid) { 
    #pragma unroll
    for(int rep = 0; rep < STRIDE_64K/sizeof(long long int)/32; rep++) {
      long long int ind = warp_id * STRIDE_64K/sizeof(long long int) + rep * 32 + lane_id;
      if (ind < n) { 
        if (1) accum += ptr[ind]; 
        else ptr[ind] = val; 
      }
    } 
  } 

  if (1) 
    output[threadIdx.x + blockIdx.x * blockDim.x] = accum; 
}


int main(int argc, char **argv)
{
	printf("\n");
	
    // set device
    hipDeviceProp_t device_prop;
    //long long int dev_id = findCudaDevice(argc, (const char **) argv);
	long long int dev_id = 0;
    checkCudaErrors(hipGetDeviceProperties(&device_prop, dev_id));
	
	//int peak_clk = 1;//kHz
	//checkCudaErrors(hipDeviceGetAttribute(&peak_clk, hipDeviceAttributeClockRate, dev_id));
	//float clock_rate = (float) peak_clk;
	//printf("clock_rate:%f\n", clock_rate);

    if (!device_prop.managedMemory) { 
        // This samples requires being run on a device that supports Unified Memory
        fprintf(stderr, "Unified Memory not supported on this device\n");

        exit(EXIT_WAIVED);
    }

    if (device_prop.computeMode == hipComputeModeProhibited)
    {
        // This sample requires being run with a default or process exclusive mode
        fprintf(stderr, "This sample requires a device in either default or process exclusive mode\n");

        exit(EXIT_WAIVED);
    }
	
	/*
	if (device_prop.concurrentManagedAccess == 1){
		printf("This device supports concurrent Managed Access.\n");
    }else{
		printf("This device does not support concurrent Managed Access.\n");
	}
	*/
	
	int value1 = 1;
	checkCudaErrors(hipDeviceGetAttribute(&value1, hipDeviceAttributeConcurrentManagedAccess, dev_id));
	//printf("hipDeviceAttributeConcurrentManagedAccess = %d\n", value1);	
		
	

	/*
	//printf("############approach\n");
	for(long long int time = 0; time <= 0; time = time + 1){
	//printf("\n####################time: %llu\n", time);
	
	//long long int coverage2 = 0;
	for(long long int coverage = 1; coverage <= 1; coverage = coverage * 2){///////////////8192 is 2m.
		//coverage2++;
		//if(coverage2 == 2){
		//	coverage = 1;
		//}
		//printf("############coverage: %llu\n", coverage);
		
	for(long long int rate = 1; rate <= 1; rate = rate * 2){
		//printf("############rate: %llu\n", rate);
		
	//long long int offset2 = 0;
	//for(long long int offset = 0; offset <= 0; offset = offset * 2){///////8
	for(long long int offset = 0; offset <= 0; offset = offset + 8){
		//offset2++;
		//if(offset2 == 2){
		//	offset = 1;
		//}
	//printf("############offset: %llu\n", offset);
	
	for(long long int factor = 1; factor <= 1; factor = factor * 2){/////////////16384 (128k) max
	//printf("####################factor: %llu\n", factor);
	
	//for(double data_stride = 2684354560 * factor; data_stride <= 4294967296 * factor; data_stride = data_stride + 536870912){
	for(double data_stride = 536870912 * factor; data_stride <= 2147483648 * factor; data_stride = data_stride + 536870912){///134217728 = 1gb, 268435456 = 2gb, 536870912 = 4gb, 1073741824 = 8gb, 2147483648 = 16gb, 4294967296 = 32gb, 8589934592 = 64gb. (index)
	//printf("\n");

	for(long long int clock_count = 32; clock_count <= 32; clock_count = clock_count * 2){
		
	///long long int time2 = time;
	//if(time2 > clock_count){
	//	time2 = clock_count;
	//}

		///////////////////////////////////////////////////////////////////CPU data begin
		//double temp = data_stride * 512;
		double temp = data_stride;
		long long int data_size = (long long int) temp;		
		//data_size = data_size * 8192 * 128 / factor;
		data_size = data_size / factor;
		
		long long int *CPU_data_in1;
		checkCudaErrors(hipMallocManaged(&CPU_data_in1, sizeof(long long int) * data_size));/////////////using unified memory
		///////////////////////////////////////////////////////////////////CPU data end
		
		long long int *GPU_data_out1;
		checkCudaErrors(hipMallocManaged(&GPU_data_out1, sizeof(long long int) * data_size));/////////////using unified memory
		///////////////////////////////////////////////////////////////////GPU data out	end
		
		if(1){
			double scale = 1;
			if(data_stride < 1){
				scale = data_stride;/////////make sure threadIdx is smaller than data_size in the initialization
			}
			
			gpu_initialization<<<8192 * scale / factor, 512>>>(GPU_data_out1, data_stride, data_size);///1024 per block max
			hipDeviceSynchronize();
			if(0){
			gpu_initialization<<<8192 * scale / factor, 512>>>(CPU_data_in1, data_stride, data_size);///1024 per block max
			hipDeviceSynchronize();
			}else{
			init_cpu_data(CPU_data_in1, data_size, data_stride);
			}
		}else{
			init_cpu_data(GPU_data_out1, data_size, data_stride);
			init_cpu_data(CPU_data_in1, data_size, data_stride);		
		}
		
		/////////////////////////////////time
		struct timespec ts1;
		clock_gettime(CLOCK_REALTIME, &ts1);
		
		stream_thread<<<512, 512>>>(CPU_data_in1, 8 * data_size, GPU_data_out1, 7);

		hipDeviceSynchronize();
				
		/////////////////////////////////time
		struct timespec ts2;
		clock_gettime(CLOCK_REALTIME, &ts2);
		
		//printf("###################data_stride%lld#########################clock_count:%lld\n", data_stride, clock_count);
		//printf("*\n*\n*\nruntime:  %lluns\n", time_diff(ts1, ts2));
		printf("%llu ", time_diff(ts1, ts2));
		fflush(stdout);
		
		checkCudaErrors(hipFree(CPU_data_in1));		
		checkCudaErrors(hipFree(GPU_data_out1));
	}
	}
	}
	}
	}
	}
	}
	printf("\n");
	*/
	
	//printf("%d\n",atoll(argv[1]));
	///*
	//////nvprof --profile-from-start off --print-gpu-trace --log-file prof512512size8gpage256.txt --csv ./fault_group_test4
	///also do for less than 16 warps same/diff cores
	////256	512	1024	2048	4096(5)	8192	16384	32768	65536(9)	131072	262144	524288	1048576	2097152(not index but real size)
	////1694768538 1363231797 1252858568 1227351872 1240169346 1272947382 1233840312 1203557761 1113896111 1339415345 1362196167 1236045906 (512 512)
	////3358277712 3273436872 2704454491 2736675685 2653977370 2627856647 3346580241 2936864316 3259807074 3837481549 3687183218 3129213694 (1 512)
	////2864158121 2808228031 2391946181 2611531792 2498506627 4187674088 4210352111 3878525896 3340698560 3407195743 2510971485 2192341097 (16 32)
	////4104063378 4184730646 3189396480 3005161192 2883182901 2699462209 2709909614 3610128080 3875286731 3876505935 3562113597 3241507431 (1 256)
	////
	//for(long long int STRIDE_64K = 256; STRIDE_64K <= 524288; STRIDE_64K = STRIDE_64K * 2){
	for(long long int STRIDE_64K = atoll(argv[1]); STRIDE_64K <= atoll(argv[1]); STRIDE_64K = STRIDE_64K * 2){
	//printf("############approach\n");
	for(long long int time = 0; time <= 0; time = time + 1){
	//printf("\n####################time: %llu\n", time);
	
	//long long int coverage2 = 0;
	for(long long int coverage = 1; coverage <= 1; coverage = coverage * 2){///////////////8192 is 2m.
		//coverage2++;
		//if(coverage2 == 2){
		//	coverage = 1;
		//}
		//printf("############coverage: %llu\n", coverage);
		
	for(long long int rate = 1; rate <= 1; rate = rate * 2){
		//printf("############rate: %llu\n", rate);
		
	//long long int offset2 = 0;
	//for(long long int offset = 0; offset <= 0; offset = offset * 2){///////8
	for(long long int offset = 0; offset <= 0; offset = offset + 8){
		//offset2++;
		//if(offset2 == 2){
		//	offset = 1;
		//}
	//printf("############offset: %llu\n", offset);
	
	for(long long int factor = 1; factor <= 1; factor = factor * 2){/////////////16384 (128k) max
	//printf("####################factor: %llu\n", factor);
	
	//for(double data_stride = 2684354560 * factor; data_stride <= 4294967296 * factor; data_stride = data_stride + 536870912){
	for(double data_stride = 1073741824 * factor; data_stride <= 1073741824 * factor; data_stride = data_stride + 536870912){///134217728 = 1gb, 268435456 = 2gb, 536870912 = 4gb, 1073741824 = 8gb, 2147483648 = 16gb, 4294967296 = 32gb, 8589934592 = 64gb. (index)
	//printf("\n");

	for(long long int clock_count = 32; clock_count <= 32; clock_count = clock_count * 2){
		
	///long long int time2 = time;
	//if(time2 > clock_count){
	//	time2 = clock_count;
	//}

		///////////////////////////////////////////////////////////////////CPU data begin
		//double temp = data_stride * 512;
		double temp = data_stride;
		long long int data_size = (long long int) temp;		
		//data_size = data_size * 8192 * 128 / factor;
		data_size = data_size / factor;
		long long int data_size2 = 512 * 8192 ;	
		
		long long int *CPU_data_in1;
		checkCudaErrors(hipMallocManaged(&CPU_data_in1, sizeof(long long int) * data_size));/////////////using unified memory
		///////////////////////////////////////////////////////////////////CPU data end
		
		long long int *GPU_data_out1;
		if(0){
			checkCudaErrors(hipMallocManaged(&GPU_data_out1, sizeof(long long int) * data_size2));/////////////using unified memory
		}else{
			checkCudaErrors(hipMalloc(&GPU_data_out1, sizeof(long long int) * data_size2));/////////////not using unified memory
		}
		///////////////////////////////////////////////////////////////////GPU data out	end
		
		if(1){
			double scale = 1;
			if(data_stride < 1){
				scale = data_stride;/////////make sure threadIdx is smaller than data_size in the initialization
			}
			
			//gpu_initialization<<<8192 * scale / factor, 512>>>(GPU_data_out1, data_stride, data_size);///1024 per block max
			//hipDeviceSynchronize();
			if(0){
			gpu_initialization<<<8192 * scale / factor, 512>>>(CPU_data_in1, data_stride, data_size);///1024 per block max
			hipDeviceSynchronize();
			}else{
			init_cpu_data(CPU_data_in1, data_size, data_stride);
			}
		}else{
			init_cpu_data(GPU_data_out1, data_size2, data_stride);
			init_cpu_data(CPU_data_in1, data_size, data_stride);		
		}
		
		hipProfilerStart();////////////////////////////////start
		/////////////////////////////////time
		struct timespec ts1;
		clock_gettime(CLOCK_REALTIME, &ts1);
		
		stream_warp<<<16, 32>>>(CPU_data_in1, 8 * data_size, GPU_data_out1, 7, STRIDE_64K);

		hipDeviceSynchronize();
				
		/////////////////////////////////time
		struct timespec ts2;
		clock_gettime(CLOCK_REALTIME, &ts2);
		
		//printf("###################data_stride%lld#########################clock_count:%lld\n", data_stride, clock_count);
		//printf("*\n*\n*\nruntime:  %lluns\n", time_diff(ts1, ts2));
		printf("%llu ", time_diff(ts1, ts2));
		fflush(stdout);
		
		checkCudaErrors(hipFree(CPU_data_in1));		
		checkCudaErrors(hipFree(GPU_data_out1));
		hipProfilerStop();/////////////////////////////////stop
	}
	}
	}
	}
	}
	}
	}
	}
	printf("\n");
	//*/
	
	exit(EXIT_SUCCESS);
}