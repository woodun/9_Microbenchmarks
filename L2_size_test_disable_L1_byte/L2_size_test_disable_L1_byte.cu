#include "hip/hip_runtime.h"
#include <cstdio>
#include <ctime>
#include <vector>
#include <algorithm>
#include <stdlib.h>
// utilities
#include <hip/hip_runtime_api.h>
#include <time.h>

/////////////saturate L2 with long consecutive data. this one use the method in the paper which initialize the data multiple times. L1 is disabled with "ALL_CCFLAGS += -Xptxas -dlcm=cg"
/////////////see what happens with byte data in the ptx. can we eliminate the multiplication? is there change in latency?

void init_cpu_data(unsigned char* A, unsigned char size, unsigned char stride, int mod){
	for (unsigned char i = 0; i <= size; ++i){
		A[i] = (unsigned char)((i + stride) % mod);
   	}
}

void init_cpu_data2(int* A, int size, int stride, int mod){
	for (int i = 0; i < size; ++i){
		A[i]=(i + stride) % mod;
   	}
}

//////////min page size 4kb = 4096b = 32 * 128.
__device__ void P_chasing(int mark, unsigned char *A, int iterations, unsigned char *B, unsigned char starting_index, float clock_rate, unsigned char data_stride){
	
	unsigned char k = starting_index;/////make them in the same page, and miss near in cache lines
	for (int it = 0; it < iterations; it++){/////////////warmup
		k = A[k];
	}
	B[0] = k;///////////////it will disappear without this line.
	
	unsigned char j = starting_index;/////make them in the same page, and miss near in cache lines
	//int stride = 0;///////////
	
	long long int start_time = 0;//////clock
	long long int end_time = 0;//////clock
	start_time = clock64();//////clock
			
	for (int it = 0; it < iterations; it++){/////////////warmup
		j = A[j];
	}
	
	end_time=clock64();//////clock
	long long int total_time = end_time - start_time;//////clock
	printf("inside%d:%fms\n", mark, (total_time / (float)clock_rate) / ((float)iterations));//////clock, average latency
	
	B[0] = j;
}

__global__ void tlb_latency_test(unsigned char *A, int iterations, unsigned char *B, float clock_rate, int mod, unsigned char data_stride){	
	
	P_chasing(mod, A, iterations, B, 0, clock_rate, data_stride);
	
	 __syncthreads();
}




//////////min page size 4kb = 4096b = 32 * 128.
__device__ void P_chasing2(int mark, int *A, int iterations, int *B, int starting_index, float clock_rate, int data_stride){
	
	int k = starting_index;/////make them in the same page, and miss near in cache lines
	for (int it = 0; it < mark; it++){/////////////warmup
		k = A[k];
	}
	B[0] = k;///////////////it will disappear without this line.
	
	int j = starting_index;/////make them in the same page, and miss near in cache lines
	
	long long int start_time = 0;//////clock
	long long int end_time = 0;//////clock
	start_time = clock64();//////clock
			
	for (int it = 0; it < iterations; it++){
		j = A[j];
	}
	
	end_time=clock64();//////clock
	long long int total_time = end_time - start_time;//////clock
	printf("inside%d:%fms\n", mark, (total_time / (float)clock_rate) / ((float)iterations));//////clock, average latency
	
	B[0] = j;
}

__global__ void tlb_latency_test2(int *A, int iterations, int *B, float clock_rate, int mod, int data_stride){	
	
	P_chasing2(mod, A, iterations, B, 0, clock_rate, data_stride);
	
	 __syncthreads();
}





int main(int argc, char **argv)
{
	printf("\n");
	
    // set device
    hipDeviceProp_t device_prop;
    //int dev_id = findCudaDevice(argc, (const char **) argv);
	int dev_id = 0;
    checkCudaErrors(hipGetDeviceProperties(&device_prop, dev_id));
	
	int peak_clk = 1;//kHz
	checkCudaErrors(hipDeviceGetAttribute(&peak_clk, hipDeviceAttributeClockRate, dev_id));
	float clock_rate = (float) peak_clk;
	
	//printf("clock_rate_out_kernel:%f\n", clock_rate);

    if (!device_prop.managedMemory) { 
        // This samples requires being run on a device that supports Unified Memory
        fprintf(stderr, "Unified Memory not supported on this device\n");

        exit(EXIT_WAIVED);
    }

    if (device_prop.computeMode == hipComputeModeProhibited)
    {
        // This sample requires being run with a default or process exclusive mode
        fprintf(stderr, "This sample requires a device in either default or process exclusive mode\n");

        exit(EXIT_WAIVED);
    }

	if(1){
	printf("################byte data############################\n");
	
	///////////////////////////////////////////////////////////////////GPU data out
	unsigned char *GPU_data_out;
	checkCudaErrors(hipMalloc(&GPU_data_out, sizeof(unsigned char) * 1));	
	
	printf("################fixing data range, changing stride############################\n");
	//for(int mod = 1024 * 256 * 8; mod > 0; mod = mod / 2){/////volta L2 6m
	//for(int mod = 1024 * 256 * 7 ; mod >= 1024 * 256 * 6; mod = mod - 256 * 128){/////volta L2 6m
	for(unsigned char data_stride = 4; data_stride <= 128; data_stride = data_stride * 2){
		printf("###################data_stride%d#########################\n", data_stride / 4);
	//for(int mod = 1024 * 256 * 2; mod > 0; mod = mod - 32 * 1024){/////kepler L2 1.5m
	for(int mod = 256; mod >= 256; mod = mod / 2){/////kepler L2 1.5m //////////////1024 * 4 * 3 /////////8 /////////// 1024 * 256 * 1.5 / 1024 * 4 * 3 / 8 = 4 sets? 
		///////////////////////////////////////////////////////////////////CPU data begin
		int data_size = 512 * 1024 * 30 * 4;/////size = iteration * stride = 30 2mb pages.		
		//int iterations = data_size / data_stride;
		int iterations = mod * 1024;
	
		unsigned char *CPU_data_in = (unsigned char*) malloc(sizeof(unsigned char) * data_size);	
		init_cpu_data(CPU_data_in, 255, data_stride, mod);
		///////////////////////////////////////////////////////////////////CPU data end	
	
		///////////////////////////////////////////////////////////////////GPU data in	
		unsigned char *GPU_data_in;
		checkCudaErrors(hipMalloc(&GPU_data_in, sizeof(unsigned char) * data_size));	
		hipMemcpy(GPU_data_in, CPU_data_in, sizeof(unsigned char) * data_size, hipMemcpyHostToDevice);
		
		tlb_latency_test<<<1, 1>>>(GPU_data_in, iterations, GPU_data_out, clock_rate, mod / 4, data_stride);//////////////////////////////////////////////kernel is here	
		hipDeviceSynchronize();
		
		checkCudaErrors(hipFree(GPU_data_in));
		free(CPU_data_in);
	}
		printf("############################################\n\n");
	}
	
	printf("\n\n################fixing stride, changing data range############################\n\n");
	//for(int mod = 1024 * 256 * 8; mod > 0; mod = mod / 2){/////volta L2 6m
	//for(int mod = 1024 * 256 * 7 ; mod >= 1024 * 256 * 6; mod = mod - 256 * 128){/////volta L2 6m
	for(unsigned char data_stride = 4; data_stride <= 4; data_stride = data_stride * 2){
		printf("###################data_stride%d#########################\n", data_stride / 4);
	for(int mod = 256; mod >= 4; mod = mod / 2){/////kepler L2 1.5m
	//for(int mod = 1024 * 256 * 6; mod > 0; mod = mod / 2){/////kepler L2 1.5m //////////////1024 * 256 * 6 / 128 = 1024 * 2 * 6 ///////8 /////// 1024 * 256 * 1.5 / 1024 * 2 * 6 / 8 = 4 sets? 
		///////////////////////////////////////////////////////////////////CPU data begin
		int data_size = 1024 * 512 * 30 * 4;/////size = iteration * stride = 30 2mb pages.		
		//int iterations = data_size / data_stride;
		//int iterations = data_size;
		int iterations = mod * 1024;
	
		unsigned char *CPU_data_in = (unsigned char*) malloc(sizeof(unsigned char) * data_size);	
		init_cpu_data(CPU_data_in, 255, data_stride, mod);
		///////////////////////////////////////////////////////////////////CPU data end	
	
		///////////////////////////////////////////////////////////////////GPU data in	
		unsigned char *GPU_data_in;
		checkCudaErrors(hipMalloc(&GPU_data_in, sizeof(unsigned char) * data_size));	
		hipMemcpy(GPU_data_in, CPU_data_in, sizeof(unsigned char) * data_size, hipMemcpyHostToDevice);
		
		tlb_latency_test<<<1, 1>>>(GPU_data_in, iterations, GPU_data_out, clock_rate, mod / 4, data_stride);//////////////////////////////////////////////kernel is here	
		hipDeviceSynchronize();
		
		checkCudaErrors(hipFree(GPU_data_in));
		free(CPU_data_in);
	}
		printf("############################################\n\n");
	}
			
	checkCudaErrors(hipFree(GPU_data_out));	
	//free(CPU_data_out);
	}
	
	if(1){
	printf("################float data############################\n");
	
	int *GPU_data_out;
	checkCudaErrors(hipMalloc(&GPU_data_out, sizeof(int) * 1));	
	
	printf("################fixing data range, changing stride############################\n");
	//for(int mod = 1024 * 256 * 8; mod > 0; mod = mod / 2){/////volta L2 6m
	//for(int mod = 1024 * 256 * 7 ; mod >= 1024 * 256 * 6; mod = mod - 256 * 128){/////volta L2 6m
	for(int data_stride = 1; data_stride <= 1024; data_stride = data_stride * 2){
		printf("###################data_stride%d#########################\n", data_stride);
	//for(int mod = 1024 * 256 * 2; mod > 0; mod = mod - 32 * 1024){/////kepler L2 1.5m
	for(int mod = 1024 * 256 * 6; mod >= 1024 * 256 * 6; mod = mod / 2){/////kepler L2 1.5m //////////////1024 * 4 * 3 /////////8 /////////// 1024 * 256 * 1.5 / 1024 * 4 * 3 / 8 = 4 sets? 
		///////////////////////////////////////////////////////////////////CPU data begin
		int data_size = 512 * 1024 * 30;/////size = iteration * stride = 30 2mb pages.		
		//int iterations = data_size / data_stride;
		int iterations = mod * 2;
	
		int *CPU_data_in;
		CPU_data_in = (int*)malloc(sizeof(int) * data_size);	
		init_cpu_data2(CPU_data_in, data_size, data_stride, mod);
		///////////////////////////////////////////////////////////////////CPU data end	
	
		///////////////////////////////////////////////////////////////////GPU data in	
		int *GPU_data_in;
		checkCudaErrors(hipMalloc(&GPU_data_in, sizeof(int) * data_size));	
		hipMemcpy(GPU_data_in, CPU_data_in, sizeof(int) * data_size, hipMemcpyHostToDevice);
		
		tlb_latency_test2<<<1, 1>>>(GPU_data_in, iterations, GPU_data_out, clock_rate, mod, data_stride);//////////////////////////////////////////////kernel is here	
		hipDeviceSynchronize();
		
		checkCudaErrors(hipFree(GPU_data_in));
		free(CPU_data_in);
	}
		printf("############################################\n\n");
	}	
	
	printf("\n\n################fixing stride, changing data range############################\n\n");
	//for(int mod = 1024 * 256 * 8; mod > 0; mod = mod / 2){/////volta L2 6m
	//for(int mod = 1024 * 256 * 7 ; mod >= 1024 * 256 * 6; mod = mod - 256 * 128){/////volta L2 6m
	for(int data_stride = 4; data_stride <= 4; data_stride = data_stride * 2){
		printf("###################data_stride%d#########################\n", data_stride);
	for(int mod = 1024 * 256 * 1.5 + 32 * 1024; mod > 1024 * 256 * 1.5 - 16 * 1024; mod = mod - 1024){/////kepler L2 1.5m
	//for(int mod = 1024 * 256 * 6; mod > 0; mod = mod / 2){/////kepler L2 1.5m //////////////1024 * 256 * 6 / 128 = 1024 * 2 * 6 ///////8 /////// 1024 * 256 * 1.5 / 1024 * 2 * 6 / 8 = 4 sets? 
		///////////////////////////////////////////////////////////////////CPU data begin
		int data_size = 1024 * 512 * 30;/////size = iteration * stride = 30 2mb pages.		
		//int iterations = data_size / data_stride;
		//int iterations = data_size;
		int iterations = mod * 2;
	
		int *CPU_data_in;
		CPU_data_in = (int*)malloc(sizeof(int) * data_size);	
		init_cpu_data2(CPU_data_in, data_size, data_stride, mod);
		///////////////////////////////////////////////////////////////////CPU data end	
	
		///////////////////////////////////////////////////////////////////GPU data in	
		int *GPU_data_in;
		checkCudaErrors(hipMalloc(&GPU_data_in, sizeof(int) * data_size));	
		hipMemcpy(GPU_data_in, CPU_data_in, sizeof(int) * data_size, hipMemcpyHostToDevice);
		
		tlb_latency_test2<<<1, 1>>>(GPU_data_in, iterations, GPU_data_out, clock_rate, mod, data_stride);//////////////////////////////////////////////kernel is here	
		hipDeviceSynchronize();
		
		checkCudaErrors(hipFree(GPU_data_in));
		free(CPU_data_in);
	}
		printf("############################################\n\n");
	}
			
	checkCudaErrors(hipFree(GPU_data_out));
	}
	
    exit(EXIT_SUCCESS);
}
