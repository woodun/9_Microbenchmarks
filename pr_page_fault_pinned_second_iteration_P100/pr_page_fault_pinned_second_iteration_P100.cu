#include "hip/hip_runtime.h"
#include <cstdio>
#include <ctime>
#include <vector>
#include <algorithm>
#include <stdlib.h>
// utilities
#include <hip/hip_runtime_api.h>
#include <time.h>

///////////per request timing. L1 enabled. Pinned memory experience similar latency patterns with plain managed & copied memory. 
///////////However, as shown in the second iteration, it does not produce L2 cache hits. Meanwhile its memory access latency is much longer.
///////////It's probably because it's accessing the host memory directly.
///////////In the first iteration, it seems that the host does prefetch the L2 tlb on the host side.
///////////However, with increased data size eventually the L2 tlb will also be missed by almost all requests. 
///////////(Moreover, in the second iteration the L2 tlb miss rate is much less. 
///////////So is the latency observed in the first iteration really l2 tlb miss latency or is it also a page table context switch latency?)
///////////Sometimes there are requests with even greater latency than the l2 tlb miss.
///////////It could be the l3 tlb on the host or still the page table context switch. (Actually the 1200s happens randomly. Thus it is not another condition.)

//typedef unsigned char byte;

void init_cpu_data(int* A, long long int size, int stride, long long int mod){
	for (long int i = 0; i < size; i = i + stride){
		A[i]=(i + stride) % mod;
   	}
}

__device__ void P_chasing0(int mark, int *A, int iterations, int *B, int *C, long long int *D, int starting_index, float clock_rate, int data_stride){	
	
	int j = starting_index;/////make them in the same page, and miss near in cache lines
			
	for (int it = 0; it < iterations; it++){	
		j = A[j];		
	}	
		
	B[0] = j;
}

//////////min page size 4kb = 4096b = 32 * 128.
__device__ void P_chasing1(int mark, int *A, int iterations, int *B, int *C, long long int *D, int starting_index, float clock_rate, int data_stride){	
	
	int j = starting_index;/////make them in the same page, and miss near in cache lines
	
	//long long int start_time = 0;//////clock
	//long long int end_time = 0;//////clock
	//start_time = clock64();//////clock
			
	for (int it = 0; it < iterations; it++){
		j = A[j];
	}
	
	//end_time=clock64();//////clock
	//long long int total_time = end_time - start_time;//////clock
	//printf("inside%d:%fms\n", mark, (total_time / (float)clock_rate) / ((float)iterations));//////clock, average latency //////////the print will flush the L1?! (
	
	B[0] = j;
	//B[1] = (int) total_time;
}

//////////min page size 4kb = 4096b = 32 * 128.
__device__ void P_chasing2(int mark, int *A, long long int iterations, int *B, int *C, long long int *D, int starting_index, float clock_rate, int data_stride){//////what is the effect of warmup outside vs inside?
	
	//////shared memory: 0xc000 max (49152 Bytes = 48KB)
	__shared__ long long int s_tvalue[1024 * 4];/////must be enough to contain the number of iterations.
	__shared__ int s_index[1024 * 4];
	//__shared__ int s_index[1];
	
	int j = starting_index;/////make them in the same page, and miss near in cache lines
	//int j = B[0];
	
	long long int start_time = 0;//////clock
	long long int end_time = 0;//////clock
	long long int time_interval = 0;//////clock
	//long long int total_time = end_time - start_time;//////clock
	
	/*		
	for (int it = 0; it < iterations; it++){
		
		start_time = clock64();//////clock		
		j = A[j];
		//s_index[it] = j;
		end_time=clock64();//////clock		
		s_tvalue[it] = end_time - start_time;
	}
	*/
	
		asm(".reg .u64 t1;\n\t"
		".reg .u64 t2;\n\t");
	
	for (long long int it = 0; it < iterations; it++){
		
		/*
		asm("mul.wide.u32 	t1, %3, %5;\n\t"	
		"add.u64 	t2, t1, %4;\n\t"		
		"mov.u64 	%0, %clock64;\n\t"		
		"ld.global.u32 	%2, [t2];\n\t"
		"mov.u64 	%1, %clock64;"
		: "=l"(start_time), "=l"(end_time), "=r"(j) : "r"(j), "l"(A), "r"(4));
		*/

		asm("mul.wide.u32 	t1, %2, %4;\n\t"	
		"add.u64 	t2, t1, %3;\n\t"		
		"mov.u64 	%0, %clock64;\n\t"		
		"ld.global.u32 	%1, [t2];\n\t"		
		: "=l"(start_time), "=r"(j) : "r"(j), "l"(A), "r"(4));
		
		s_index[it] = j;////what if without this? ///Then it is not accurate and cannot get the access time at all, due to the ILP. (another way is to use average time, but inevitably containing other instructions:setp, add).
		
		asm volatile ("mov.u64 %0, %clock64;": "=l"(end_time));
		
		time_interval = end_time - start_time;
		//if(it >= 4 * 1024){
		s_tvalue[it] = time_interval;
		//}
	}
	
	//printf("inside%d:%fms\n", mark, (total_time / (float)clock_rate) / ((float)iterations));//////clock, average latency
	
	B[0] = j;
	
	for (long long int it = 0; it < iterations; it++){		
		C[it] = s_index[it];
		D[it] = s_tvalue[it];
	}
}

__global__ void tlb_latency_test(int *A, long long int iterations, int *B, int *C, long long int *D, float clock_rate, long long int mod, int data_stride){
	
	int reduced_iter = iterations;
	if(reduced_iter > 512){
		reduced_iter = 512;
	}
	
	///////////kepler L2 has 48 * 1024 = 49152 cache lines. But we only have 1024 * 4 slots in shared memory.
	P_chasing1(0, A, iterations + 0, B, C, D, 0, clock_rate, data_stride);////////saturate the L2
	P_chasing2(0, A, reduced_iter, B, C, D, 0, clock_rate, data_stride);////////partially print the data
	
	 __syncthreads();
}

int main(int argc, char **argv)
{
	printf("\n");
	
    // set device
    hipDeviceProp_t device_prop;
    //int dev_id = findCudaDevice(argc, (const char **) argv);
	int dev_id = 0;
    checkCudaErrors(hipGetDeviceProperties(&device_prop, dev_id));
	
	int peak_clk = 1;//kHz
	checkCudaErrors(hipDeviceGetAttribute(&peak_clk, hipDeviceAttributeClockRate, dev_id));
	float clock_rate = (float) peak_clk;
	
	//printf("clock_rate_out_kernel:%f\n", clock_rate);

    if (!device_prop.managedMemory) { 
        // This samples requires being run on a device that supports Unified Memory
        fprintf(stderr, "Unified Memory not supported on this device\n");

        exit(EXIT_WAIVED);
    }

    if (device_prop.computeMode == hipComputeModeProhibited)
    {
        // This sample requires being run with a default or process exclusive mode
        fprintf(stderr, "This sample requires a device in either default or process exclusive mode\n");

        exit(EXIT_WAIVED);
    }
	
	if (device_prop.concurrentManagedAccess == 1){
		printf("This device supports concurrent Managed Access.\n");
    }else{
		printf("This device does not support concurrent Managed Access.\n");
	}
	
	
	///////////////////////////////////////////////////////////////////GPU data out
	int *GPU_data_out;
	checkCudaErrors(hipMalloc(&GPU_data_out, sizeof(int) * 2));			
	
	FILE * pFile;
    pFile = fopen ("output.txt","w");		
	
	int counter = 0;
	for(int data_stride = 2 * 16 * 1024; data_stride <= 32 * 256 * 1024; data_stride = data_stride * 2){/////////32mb stride
		//printf("###################data_stride%d#########################\n", data_stride);
	//for(int mod = 1024 * 256 * 2; mod > 0; mod = mod - 32 * 1024){/////kepler L2 1.5m
	for(long long int mod = 2 * 256 * 1024 * 8; mod <= 2147483648; mod = mod * 2){////268435456 = 1gb, 536870912 = 2gb, 1073741824 = 4gb, 2147483648 = 8gb, 4294967296 = 16gb.
		counter++;
		///////////////////////////////////////////////////////////////////CPU data begin
		//int data_size = 2 * 256 * 1024 * 32;/////size = iteration * stride = 32 2mb pages.
		long long int data_size = mod;
		//int iterations = data_size / data_stride;
		//int iterations = 1024 * 256 * 8;
		long long int iterations = mod / data_stride;////32 * 32 * 4 / 32 * 2 = 256
	
		int *CPU_data_in;
		//CPU_data_in = (int*)malloc(sizeof(int) * data_size);
		//checkCudaErrors(hipMallocManaged(&CPU_data_in, sizeof(int) * data_size));/////////////using unified memory
		checkCudaErrors(hipHostAlloc((void**)&CPU_data_in, sizeof(int) * data_size, hipHostMallocDefault));//////////using pinned memory
		init_cpu_data(CPU_data_in, data_size, data_stride, mod);
		
		int *CPU_data_out_index;
		CPU_data_out_index = (int*)malloc(sizeof(int) * iterations);
		long long int *CPU_data_out_time;
		CPU_data_out_time = (long long int*)malloc(sizeof(long long int) * iterations);
		///////////////////////////////////////////////////////////////////CPU data end	
	
		///////////////////////////////////////////////////////////////////GPU data in	
		//int *GPU_data_in;
		//checkCudaErrors(hipMalloc(&GPU_data_in, sizeof(int) * data_size));	
		//hipMemcpy(GPU_data_in, CPU_data_in, sizeof(int) * data_size, hipMemcpyHostToDevice);
		
		///////////////////////////////////////////////////////////////////GPU data out
		int *GPU_data_out_index;
		checkCudaErrors(hipMalloc(&GPU_data_out_index, sizeof(int) * iterations));
		long long int *GPU_data_out_time;
		checkCudaErrors(hipMalloc(&GPU_data_out_time, sizeof(long long int) * iterations));
		
		tlb_latency_test<<<1, 1>>>(CPU_data_in, iterations, GPU_data_out, GPU_data_out_index, GPU_data_out_time, clock_rate, mod, data_stride);///////////////kernel is here	
		hipDeviceSynchronize();
				
		hipMemcpy(CPU_data_out_index, GPU_data_out_index, sizeof(int) * iterations, hipMemcpyDeviceToHost);
		hipMemcpy(CPU_data_out_time, GPU_data_out_time, sizeof(long long int) * iterations, hipMemcpyDeviceToHost);
				
		fprintf(pFile, "###################data_stride%d#########################\n", data_stride);
		fprintf (pFile, "###############Mod%d##############%d\n", mod, iterations);
		for (long long int it = 0; it < iterations; it++){			
			fprintf (pFile, "%d %fms %lldcycles\n", CPU_data_out_index[it], CPU_data_out_time[it] / (float)clock_rate, CPU_data_out_time[it]);
			//fprintf (pFile, "%d %fms\n", it, CPU_data_out_time[it] / (float)clock_rate);
			//printf ("%d %fms\n", CPU_data_out_index[it], CPU_data_out_time[it] / (float)clock_rate);
		}
		
		checkCudaErrors(hipFree(GPU_data_out_index));
		checkCudaErrors(hipFree(GPU_data_out_time));
		//checkCudaErrors(hipFree(GPU_data_in));
		//checkCudaErrors(hipFree(CPU_data_in));
		checkCudaErrors(hipHostFree(CPU_data_in));//////using pinned memory
		//free(CPU_data_in);
		free(CPU_data_out_index);
		free(CPU_data_out_time);
	}
		//printf("############################################\n\n");
	}
			
	checkCudaErrors(hipFree(GPU_data_out));	
	//free(CPU_data_out);
	fclose (pFile);
	
    exit(EXIT_SUCCESS);
}
