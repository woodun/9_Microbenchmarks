#include "hip/hip_runtime.h"
#include <cstdio>
#include <ctime>
#include <vector>
#include <algorithm>
#include <stdlib.h>
// utilities
#include <hip/hip_runtime_api.h>
#include <time.h>

/////////////////////////////access consecutive data. change the strides to see L1 cache line size. question: why it does not show difference when passing the L2 cache line size?


void init_cpu_data(int* A, int size, int stride){
	for (int i = 0; i < size; ++i){
		A[i]=(i + stride) % size;
   	}
}


//////////min page size 4kb = 4096b = 32 * 128.
__device__ void P_chasing(int mark, int *A, int iterations, int *B, int starting_index, float clock_rate){
	
	int j = starting_index;/////make them in the same page, and miss near in cache lines
	
	long long int start_time = 0;//////clock
	long long int end_time = 0;//////clock
	start_time = clock64();//////clock
	
	for (int it =0; it < iterations; it ++){
		j = A[j];
	}
	
	end_time=clock64();//////clock
	long long int total_time = end_time - start_time;//////clock
	printf("inside%d:%fms\n", mark, (total_time / (float)clock_rate) / (float)iterations);//////clock, average latency
	
	B[0] = j;
}

//////////////////////////////////////////////////////4 * (8) * 32 * 32 = 128kb ///////////////////48 * 128kb = 6144kb ///////////12 * 128kb = 1536kb ////////////// 16 * 64 = 1024 = 4kb
__global__ void tlb_latency_test_stride1(int *A, int iterations, int *B, float clock_rate){	

	printf("stride1:\n");

	int index = 0;
	
	long long int start_time = 0;///////////clock
	long long int end_time = 0;///////////clock	
	start_time = clock64();///////////clock
		
	P_chasing(15, A, 16, B, 15 * 524288, clock_rate);/////warmup
	P_chasing(16, A, 16, B, 16 * 524288, clock_rate);/////try to generate TLB miss and cache miss
	P_chasing(0, A, 1, B, 0 * 524288 + 0 * 1, clock_rate);/////warmup TLB
	P_chasing(0, A, 16, B, 0 * 524288 + 1 * 1, clock_rate);/////try to generate TLB hit and cache miss ///////(1)
	P_chasing(0, A, 16, B, 0 * 524288 + 1 * 1, clock_rate);/////try to generate TLB hit and cache hit ///////(1)
	P_chasing(16, A, 16, B, 16 * 524288 + 16 * 32, clock_rate);/////try to generate TLB hit and cache miss
	P_chasing(16, A, 16, B, 16 * 524288, clock_rate);/////try to generate TLB hit and cache hit
	P_chasing(16, A, 16, B, 16 * 524288, clock_rate);/////try to generate TLB hit and cache hit
	
	end_time=clock64();///////////clock
		
	long long int total_time = end_time - start_time;///////////clock
	//printf("outside1:%fms\n", total_time / (float)clock_rate);///////////clock
	printf("\n");
	
	__syncthreads();
}

__global__ void tlb_latency_test_stride2(int *A, int iterations, int *B, float clock_rate){
	
	printf("stride2:\n");

	int index = 0;
	
	long long int start_time = 0;///////////clock
	long long int end_time = 0;///////////clock	
	start_time = clock64();///////////clock
		
	P_chasing(15, A, 16, B, 15 * 524288, clock_rate);/////warmup
	P_chasing(16, A, 16, B, 16 * 524288, clock_rate);/////try to generate TLB miss and cache miss
	P_chasing(0, A, 1, B, 0 * 524288 + 0 * 2, clock_rate);/////warmup TLB
	P_chasing(0, A, 16, B, 0 * 524288 + 1 * 2, clock_rate);/////try to generate TLB hit and cache miss ///////(1)
	P_chasing(0, A, 16, B, 0 * 524288 + 1 * 2, clock_rate);/////try to generate TLB hit and cache hit ///////(1)
	P_chasing(16, A, 16, B, 16 * 524288 + 16 * 32, clock_rate);/////try to generate TLB hit and cache miss
	P_chasing(16, A, 16, B, 16 * 524288, clock_rate);/////try to generate TLB hit and cache hit
	P_chasing(16, A, 16, B, 16 * 524288, clock_rate);/////try to generate TLB hit and cache hit
	
	end_time=clock64();///////////clock
		
	long long int total_time = end_time - start_time;///////////clock
	//printf("outside1:%fms\n", total_time / (float)clock_rate);///////////clock
	printf("\n");
	
	__syncthreads();
}

__global__ void tlb_latency_test_stride4(int *A, int iterations, int *B, float clock_rate){
	
	printf("stride4:\n");

	int index = 0;
	
	long long int start_time = 0;///////////clock
	long long int end_time = 0;///////////clock	
	start_time = clock64();///////////clock
		
	P_chasing(15, A, 16, B, 15 * 524288, clock_rate);/////warmup
	P_chasing(16, A, 16, B, 16 * 524288, clock_rate);/////try to generate TLB miss and cache miss
	P_chasing(0, A, 1, B, 0 * 524288 + 0 * 4, clock_rate);/////warmup TLB
	P_chasing(0, A, 16, B, 0 * 524288 + 1 * 4, clock_rate);/////try to generate TLB hit and cache miss ///////(1)
	P_chasing(0, A, 16, B, 0 * 524288 + 1 * 4, clock_rate);/////try to generate TLB hit and cache hit ///////(1)
	P_chasing(16, A, 16, B, 16 * 524288 + 16 * 32, clock_rate);/////try to generate TLB hit and cache miss
	P_chasing(16, A, 16, B, 16 * 524288, clock_rate);/////try to generate TLB hit and cache hit
	P_chasing(16, A, 16, B, 16 * 524288, clock_rate);/////try to generate TLB hit and cache hit
	
	end_time=clock64();///////////clock
		
	long long int total_time = end_time - start_time;///////////clock
	//printf("outside1:%fms\n", total_time / (float)clock_rate);///////////clock
	printf("\n");
	
	__syncthreads();
}

__global__ void tlb_latency_test_stride8(int *A, int iterations, int *B, float clock_rate){
	
	printf("stride8:\n");

	int index = 0;
	
	long long int start_time = 0;///////////clock
	long long int end_time = 0;///////////clock	
	start_time = clock64();///////////clock
		
	P_chasing(15, A, 16, B, 15 * 524288, clock_rate);/////warmup
	P_chasing(16, A, 16, B, 16 * 524288, clock_rate);/////try to generate TLB miss and cache miss
	P_chasing(0, A, 1, B, 0 * 524288 + 0 * 8, clock_rate);/////warmup TLB
	P_chasing(0, A, 16, B, 0 * 524288 + 1 * 8, clock_rate);/////try to generate TLB hit and cache miss ///////(1)
	P_chasing(0, A, 16, B, 0 * 524288 + 1 * 8, clock_rate);/////try to generate TLB hit and cache hit ///////(1)
	P_chasing(16, A, 16, B, 16 * 524288 + 16 * 32, clock_rate);/////try to generate TLB hit and cache miss
	P_chasing(16, A, 16, B, 16 * 524288, clock_rate);/////try to generate TLB hit and cache hit
	P_chasing(16, A, 16, B, 16 * 524288, clock_rate);/////try to generate TLB hit and cache hit
	
	end_time=clock64();///////////clock
		
	long long int total_time = end_time - start_time;///////////clock
	//printf("outside1:%fms\n", total_time / (float)clock_rate);///////////clock
	printf("\n");
	
	__syncthreads();
}

__global__ void tlb_latency_test_stride16(int *A, int iterations, int *B, float clock_rate){
	
	printf("stride16:\n");

	int index = 0;
	
	long long int start_time = 0;///////////clock
	long long int end_time = 0;///////////clock	
	start_time = clock64();///////////clock
		
	P_chasing(15, A, 16, B, 15 * 524288, clock_rate);/////warmup
	P_chasing(16, A, 16, B, 16 * 524288, clock_rate);/////try to generate TLB miss and cache miss
	P_chasing(0, A, 1, B, 0 * 524288 + 0 * 16, clock_rate);/////warmup TLB
	P_chasing(0, A, 16, B, 0 * 524288 + 1 * 16, clock_rate);/////try to generate TLB hit and cache miss ///////(1)
	P_chasing(0, A, 16, B, 0 * 524288 + 1 * 16, clock_rate);/////try to generate TLB hit and cache hit ///////(1)
	P_chasing(16, A, 16, B, 16 * 524288 + 16 * 32, clock_rate);/////try to generate TLB hit and cache miss
	P_chasing(16, A, 16, B, 16 * 524288, clock_rate);/////try to generate TLB hit and cache hit
	P_chasing(16, A, 16, B, 16 * 524288, clock_rate);/////try to generate TLB hit and cache hit
	
	end_time=clock64();///////////clock
		
	long long int total_time = end_time - start_time;///////////clock
	//printf("outside1:%fms\n", total_time / (float)clock_rate);///////////clock
	printf("\n");
	
	__syncthreads();
}

__global__ void tlb_latency_test_stride32(int *A, int iterations, int *B, float clock_rate){
	
	printf("stride32:\n");

	int index = 0;
	
	long long int start_time = 0;///////////clock
	long long int end_time = 0;///////////clock	
	start_time = clock64();///////////clock
		
	P_chasing(15, A, 16, B, 15 * 524288, clock_rate);/////warmup
	P_chasing(16, A, 16, B, 16 * 524288, clock_rate);/////try to generate TLB miss and cache miss
	P_chasing(0, A, 1, B, 0 * 524288 + 0 * 32, clock_rate);/////warmup TLB
	P_chasing(0, A, 16, B, 0 * 524288 + 1 * 32, clock_rate);/////try to generate TLB hit and cache miss ///////(1)
	P_chasing(0, A, 16, B, 0 * 524288 + 1 * 32, clock_rate);/////try to generate TLB hit and cache hit ///////(1)
	P_chasing(16, A, 16, B, 16 * 524288 + 16 * 32, clock_rate);/////try to generate TLB hit and cache miss
	P_chasing(16, A, 16, B, 16 * 524288, clock_rate);/////try to generate TLB hit and cache hit
	P_chasing(16, A, 16, B, 16 * 524288, clock_rate);/////try to generate TLB hit and cache hit
	
	end_time=clock64();///////////clock
		
	long long int total_time = end_time - start_time;///////////clock
	//printf("outside1:%fms\n", total_time / (float)clock_rate);///////////clock
	printf("\n");
	
	__syncthreads();
}

__global__ void tlb_latency_test_stride64(int *A, int iterations, int *B, float clock_rate){
	
	printf("stride64:\n");

	int index = 0;
	
	long long int start_time = 0;///////////clock
	long long int end_time = 0;///////////clock	
	start_time = clock64();///////////clock
		
	P_chasing(15, A, 8, B, 15 * 524288, clock_rate);/////warmup
	P_chasing(16, A, 8, B, 16 * 524288, clock_rate);/////try to generate TLB miss and cache miss
	P_chasing(0, A, 1, B, 0 * 524288 + 0 * 64, clock_rate);/////warmup TLB
	P_chasing(0, A, 8, B, 0 * 524288 + 1 * 64, clock_rate);/////try to generate TLB hit and cache miss ///////(1)
	P_chasing(0, A, 8, B, 0 * 524288 + 1 * 64, clock_rate);/////try to generate TLB hit and cache hit ///////(1)
	P_chasing(16, A, 8, B, 16 * 524288 + 16 * 32, clock_rate);/////try to generate TLB hit and cache miss
	P_chasing(16, A, 8, B, 16 * 524288, clock_rate);/////try to generate TLB hit and cache hit
	P_chasing(16, A, 8, B, 16 * 524288, clock_rate);/////try to generate TLB hit and cache hit
	
	end_time=clock64();///////////clock
		
	long long int total_time = end_time - start_time;///////////clock
	//printf("outside1:%fms\n", total_time / (float)clock_rate);///////////clock
	printf("\n");
	
	__syncthreads();
}

int main(int argc, char **argv)
{
	printf("\n");
	
    // set device
    hipDeviceProp_t device_prop;
    //int dev_id = findCudaDevice(argc, (const char **) argv);
	int dev_id = 1;
    checkCudaErrors(hipGetDeviceProperties(&device_prop, dev_id));
	
	int peak_clk = 1;//kHz
	checkCudaErrors(hipDeviceGetAttribute(&peak_clk, hipDeviceAttributeClockRate, dev_id));
	float clock_rate = (float) peak_clk;
	
	//printf("clock_rate_out_kernel:%f\n", clock_rate);

    if (!device_prop.managedMemory) { 
        // This samples requires being run on a device that supports Unified Memory
        fprintf(stderr, "Unified Memory not supported on this device\n");

        exit(EXIT_WAIVED);
    }

    if (device_prop.computeMode == hipComputeModeProhibited)
    {
        // This sample requires being run with a default or process exclusive mode
        fprintf(stderr, "This sample requires a device in either default or process exclusive mode\n");

        exit(EXIT_WAIVED);
    }
	
	///////////////////////////////////////////////////////////////////GPU output data
	int *GPU_data_out;
	checkCudaErrors(hipMalloc(&GPU_data_out, sizeof(int) * 1));
		
	////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////_stride64
	///////////////////////////////////////////////////////////////////CPU data begin
	int iterations_stride64 = 16384 * 100 / 2;	
	int data_stride_stride64 = 64;/////256b. Pointing to the next cacheline.	
	int data_size_stride64 = iterations_stride64 * data_stride_stride64;/////size = iteration * stride = 100 2mb pages.
	
	int *CPU_data_in_stride64;	
	CPU_data_in_stride64 = (int*)malloc(sizeof(int) * data_size_stride64);	
	init_cpu_data(CPU_data_in_stride64, data_size_stride64, data_stride_stride64);
	///////////////////////////////////////////////////////////////////CPU data end
	
	///////////////////////////////////////////////////////////////////GPU input data
	int *GPU_data_in_stride64;	
	checkCudaErrors(hipMalloc(&GPU_data_in_stride64, sizeof(int) * data_size_stride64));	
	checkCudaErrors(hipMemcpy(GPU_data_in_stride64, CPU_data_in_stride64, sizeof(int) * data_size_stride64, hipMemcpyHostToDevice));
		
	tlb_latency_test_stride64<<<1, 1>>>(GPU_data_in_stride64, iterations_stride64, GPU_data_out, clock_rate);//////////////////////////////////////////////kernel is here
	
	checkCudaErrors(hipFree(GPU_data_in_stride64));	
	free(CPU_data_in_stride64);
	////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////_stride64
	
	////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////_stride32
	///////////////////////////////////////////////////////////////////CPU data begin
	int iterations_stride32 = 1 * 16384 * 100;	
	int data_stride_stride32 = 32;/////64b. Pointing to the next cacheline.	
	int data_size_stride32 = iterations_stride32 * data_stride_stride32;/////size = iteration * stride = 100 2mb pages.
	
	int *CPU_data_in_stride32;	
	CPU_data_in_stride32 = (int*)malloc(sizeof(int) * data_size_stride32);	
	init_cpu_data(CPU_data_in_stride32, data_size_stride32, data_stride_stride32);
	///////////////////////////////////////////////////////////////////CPU data end
	
	///////////////////////////////////////////////////////////////////GPU input data
	int *GPU_data_in_stride32;	
	checkCudaErrors(hipMalloc(&GPU_data_in_stride32, sizeof(int) * data_size_stride32));	
	checkCudaErrors(hipMemcpy(GPU_data_in_stride32, CPU_data_in_stride32, sizeof(int) * data_size_stride32, hipMemcpyHostToDevice));
		
	tlb_latency_test_stride32<<<1, 1>>>(GPU_data_in_stride32, iterations_stride32, GPU_data_out, clock_rate);//////////////////////////////////////////////kernel is here
	
	checkCudaErrors(hipFree(GPU_data_in_stride32));	
	free(CPU_data_in_stride32);
	////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////_stride32
	
	////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////_stride16
	///////////////////////////////////////////////////////////////////CPU data begin
	int iterations_stride16 = 2 * 16384 * 100;	
	int data_stride_stride16 = 16;/////64b. Pointing to the next cacheline.	
	int data_size_stride16 = iterations_stride16 * data_stride_stride16;/////size = iteration * stride = 100 2mb pages.
	
	int *CPU_data_in_stride16;	
	CPU_data_in_stride16 = (int*)malloc(sizeof(int) * data_size_stride16);	
	init_cpu_data(CPU_data_in_stride16, data_size_stride16, data_stride_stride16);
	///////////////////////////////////////////////////////////////////CPU data end
	
	///////////////////////////////////////////////////////////////////GPU input data
	int *GPU_data_in_stride16;	
	checkCudaErrors(hipMalloc(&GPU_data_in_stride16, sizeof(int) * data_size_stride16));	
	checkCudaErrors(hipMemcpy(GPU_data_in_stride16, CPU_data_in_stride16, sizeof(int) * data_size_stride16, hipMemcpyHostToDevice));
		
	tlb_latency_test_stride16<<<1, 1>>>(GPU_data_in_stride16, iterations_stride16, GPU_data_out, clock_rate);//////////////////////////////////////////////kernel is here
	
	checkCudaErrors(hipFree(GPU_data_in_stride16));	
	free(CPU_data_in_stride16);
	////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////_stride16
	
	////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////_stride8
	///////////////////////////////////////////////////////////////////CPU data begin
	int iterations_stride8 = 4 * 16384 * 100;	
	int data_stride_stride8 = 8;/////64b. Pointing to the next cacheline.	
	int data_size_stride8 = iterations_stride8 * data_stride_stride8;/////size = iteration * stride = 100 2mb pages.
	
	int *CPU_data_in_stride8;	
	CPU_data_in_stride8 = (int*)malloc(sizeof(int) * data_size_stride8);	
	init_cpu_data(CPU_data_in_stride8, data_size_stride8, data_stride_stride8);
	///////////////////////////////////////////////////////////////////CPU data end
	
	///////////////////////////////////////////////////////////////////GPU input data
	int *GPU_data_in_stride8;	
	checkCudaErrors(hipMalloc(&GPU_data_in_stride8, sizeof(int) * data_size_stride8));	
	checkCudaErrors(hipMemcpy(GPU_data_in_stride8, CPU_data_in_stride8, sizeof(int) * data_size_stride8, hipMemcpyHostToDevice));
		
	tlb_latency_test_stride8<<<1, 1>>>(GPU_data_in_stride8, iterations_stride8, GPU_data_out, clock_rate);//////////////////////////////////////////////kernel is here
	
	checkCudaErrors(hipFree(GPU_data_in_stride8));	
	free(CPU_data_in_stride8);
	////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////_stride8
	
	////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////_stride4
	///////////////////////////////////////////////////////////////////CPU data begin
	int iterations_stride4 = 8 * 16384 * 100;	
	int data_stride_stride4 = 4;/////64b. Pointing to the next cacheline.	
	int data_size_stride4 = iterations_stride4 * data_stride_stride4;/////size = iteration * stride = 100 2mb pages.
	
	int *CPU_data_in_stride4;	
	CPU_data_in_stride4 = (int*)malloc(sizeof(int) * data_size_stride4);	
	init_cpu_data(CPU_data_in_stride4, data_size_stride4, data_stride_stride4);
	///////////////////////////////////////////////////////////////////CPU data end
	
	///////////////////////////////////////////////////////////////////GPU input data
	int *GPU_data_in_stride4;	
	checkCudaErrors(hipMalloc(&GPU_data_in_stride4, sizeof(int) * data_size_stride4));	
	checkCudaErrors(hipMemcpy(GPU_data_in_stride4, CPU_data_in_stride4, sizeof(int) * data_size_stride4, hipMemcpyHostToDevice));
		
	tlb_latency_test_stride4<<<1, 1>>>(GPU_data_in_stride4, iterations_stride4, GPU_data_out, clock_rate);//////////////////////////////////////////////kernel is here
	
	checkCudaErrors(hipFree(GPU_data_in_stride4));	
	free(CPU_data_in_stride4);
	////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////_stride4
	
	////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////_stride2
	///////////////////////////////////////////////////////////////////CPU data begin
	int iterations_stride2 = 16 * 16384 * 100;	
	int data_stride_stride2 = 2;/////64b. Pointing to the next cacheline.	
	int data_size_stride2 = iterations_stride2 * data_stride_stride2;/////size = iteration * stride = 100 2mb pages.
	
	int *CPU_data_in_stride2;	
	CPU_data_in_stride2 = (int*)malloc(sizeof(int) * data_size_stride2);	
	init_cpu_data(CPU_data_in_stride2, data_size_stride2, data_stride_stride2);
	///////////////////////////////////////////////////////////////////CPU data end
	
	///////////////////////////////////////////////////////////////////GPU input data
	int *GPU_data_in_stride2;	
	checkCudaErrors(hipMalloc(&GPU_data_in_stride2, sizeof(int) * data_size_stride2));	
	checkCudaErrors(hipMemcpy(GPU_data_in_stride2, CPU_data_in_stride2, sizeof(int) * data_size_stride2, hipMemcpyHostToDevice));
		
	tlb_latency_test_stride2<<<1, 1>>>(GPU_data_in_stride2, iterations_stride2, GPU_data_out, clock_rate);//////////////////////////////////////////////kernel is here
	
	checkCudaErrors(hipFree(GPU_data_in_stride2));	
	free(CPU_data_in_stride2);
	////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////_stride2
	
	////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////_stride1
	///////////////////////////////////////////////////////////////////CPU data begin
	int iterations_stride1 = 32 * 16384 * 100;	
	int data_stride_stride1 = 1;/////64b. Pointing to the next cacheline.	
	int data_size_stride1 = iterations_stride1 * data_stride_stride1;/////size = iteration * stride = 100 2mb pages.
	
	int *CPU_data_in_stride1;	
	CPU_data_in_stride1 = (int*)malloc(sizeof(int) * data_size_stride1);	
	init_cpu_data(CPU_data_in_stride1, data_size_stride1, data_stride_stride1);
	///////////////////////////////////////////////////////////////////CPU data end
	
	///////////////////////////////////////////////////////////////////GPU input data
	int *GPU_data_in_stride1;	
	checkCudaErrors(hipMalloc(&GPU_data_in_stride1, sizeof(int) * data_size_stride1));	
	checkCudaErrors(hipMemcpy(GPU_data_in_stride1, CPU_data_in_stride1, sizeof(int) * data_size_stride1, hipMemcpyHostToDevice));
		
	tlb_latency_test_stride1<<<1, 1>>>(GPU_data_in_stride1, iterations_stride1, GPU_data_out, clock_rate);//////////////////////////////////////////////kernel is here
	
	checkCudaErrors(hipFree(GPU_data_in_stride1));	
	free(CPU_data_in_stride1);
	////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////_stride1
	
	
	checkCudaErrors(hipFree(GPU_data_out));
		
    exit(EXIT_SUCCESS);
}
