#include "hip/hip_runtime.h"
#include <cstdio>
#include <ctime>
#include <vector>
#include <algorithm>
#include <stdlib.h>
// utilities
#include <hip/hip_runtime_api.h>
#include <time.h>

/////////////////////////////access consecutive data. change the strides to see L1 cache line size. question: why it does not show difference when passing the L2 cache line size?


void init_cpu_data(int* A, int size, int stride){
	for (int i = 0; i < size; ++i){
		A[i]=(i + stride) % size;
   	}
}


//////////min page size 4kb = 4096b = 32 * 128.
__device__ void P_chasing(int mark, int *A, int iterations, int *B, int starting_index, float clock_rate){
	
	int j = starting_index;/////make them in the same page, and miss near in cache lines
	
	long long int start_time = 0;//////clock
	long long int end_time = 0;//////clock
	start_time = clock64();//////clock
	
	for (int it =0; it < iterations; it ++){
		j = A[j];
	}
	
	end_time=clock64();//////clock
	long long int total_time = end_time - start_time;//////clock
	printf("inside%d:%fms\n", mark, (total_time / (float)clock_rate) / (float)iterations);//////clock, average latency
	
	B[0] = j;
}

__global__ void tlb_latency_test_stride(int *A, int iterations, int *B, float clock_rate, int iter, int stride){
	
	printf("stride%d:\n", stride);
			
	P_chasing(-7, A, 16, B, 7 * 524288, clock_rate);/////warmup	
	
	P_chasing(8, A, 16, B, 8 * 524288, clock_rate);/////try to generate TLB miss and cache miss
	P_chasing(8, A, 16, B, 8 * 524288, clock_rate);/////try to generate TLB hit and cache hit
	P_chasing(8, A, 16, B, 8 * 524288, clock_rate);/////try to generate TLB hit and cache hit
	P_chasing(8, A, 16, B, 8 * 524288, clock_rate);/////try to generate TLB hit and cache hit
	
	P_chasing(-16, A, 1, B, 16 * 524288 + 0 * stride, clock_rate);/////warmup TLB
	P_chasing(-16, A, 1, B, 16 * 524288 + 31 * stride, clock_rate);/////warmup TLB
	P_chasing(16, A, 16, B, 16 * 524288 + 1 * stride, clock_rate);/////try to generate TLB hit and cache miss ///////
	P_chasing(16, A, 16, B, 16 * 524288 + 1 * stride, clock_rate);/////try to generate TLB hit and cache hit ///////
	P_chasing(16, A, 16, B, 16 * 524288 + 0 * stride, clock_rate);/////try to generate TLB hit and cache hit ///////
	P_chasing(16, A, 16, B, 16 * 524288 + 0 * stride, clock_rate);/////try to generate TLB hit and cache hit ///////
	
	P_chasing(-1, A, 1, B, 0 * 524288 + 0 * stride, clock_rate);/////warmup TLB
	P_chasing(-1, A, 1, B, 0 * 524288 + 31 * stride, clock_rate);/////warmup TLB
	P_chasing(0, A, 16, B, 0 * 524288 + 1 * stride, clock_rate);/////try to generate TLB hit and cache miss ///////(1)
	P_chasing(0, A, 16, B, 0 * 524288 + 1 * stride, clock_rate);/////try to generate TLB hit and cache hit ///////(1)
	P_chasing(0, A, 16, B, 0 * 524288 + 0 * stride, clock_rate);/////try to generate TLB hit and cache hit ///////(1)
	P_chasing(0, A, 16, B, 0 * 524288 + 0 * stride, clock_rate);/////try to generate TLB hit and cache hit ///////(1)
	
	printf("\n");	
	__syncthreads();
}

int main(int argc, char **argv)
{
	printf("\n");
	
    // set device
    hipDeviceProp_t device_prop;
    //int dev_id = findCudaDevice(argc, (const char **) argv);
	int dev_id = 0;
    checkCudaErrors(hipGetDeviceProperties(&device_prop, dev_id));
	
	int peak_clk = 1;//kHz
	checkCudaErrors(hipDeviceGetAttribute(&peak_clk, hipDeviceAttributeClockRate, dev_id));
	float clock_rate = (float) peak_clk;
	
	//printf("clock_rate_out_kernel:%f\n", clock_rate);

    if (!device_prop.managedMemory) { 
        // This samples requires being run on a device that supports Unified Memory
        fprintf(stderr, "Unified Memory not supported on this device\n");

        exit(EXIT_WAIVED);
    }

    if (device_prop.computeMode == hipComputeModeProhibited)
    {
        // This sample requires being run with a default or process exclusive mode
        fprintf(stderr, "This sample requires a device in either default or process exclusive mode\n");

        exit(EXIT_WAIVED);
    }
	
	///////////////////////////////////////////////////////////////////GPU output data
	int *GPU_data_out;
	checkCudaErrors(hipMalloc(&GPU_data_out, sizeof(int) * 1));
		
	int data_size = 512 * 1024 * 100;/////size = iteration * stride = 100 2mb pages.
	for(int data_stride = 64; data_stride > 0; data_stride = data_stride / 2){	
		///////////////////////////////////////////////////////////////////CPU data begin
		int iterations_stride = data_size / data_stride;		
		int *CPU_data_in;	
		CPU_data_in = (int*)malloc(sizeof(int) * data_size);	
		init_cpu_data(CPU_data_in, data_size, data_stride);
		
		///////////////////////////////////////////////////////////////////GPU input data
		int *GPU_data_in;	
		checkCudaErrors(hipMalloc(&GPU_data_in, sizeof(int) * data_size));	
		checkCudaErrors(hipMemcpy(GPU_data_in, CPU_data_in, sizeof(int) * data_size, hipMemcpyHostToDevice));
		
		tlb_latency_test_stride<<<1, 1>>>(GPU_data_in, iterations_stride, GPU_data_out, clock_rate, 16, data_stride);//////////////////////////////////////////////kernel is here
		hipDeviceSynchronize();
	
		checkCudaErrors(hipFree(GPU_data_in));	
		free(CPU_data_in);
	}	
	
	checkCudaErrors(hipFree(GPU_data_out));
		
    exit(EXIT_SUCCESS);
}
