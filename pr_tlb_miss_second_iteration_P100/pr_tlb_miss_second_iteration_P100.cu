#include "hip/hip_runtime.h"
#include <cstdio>
#include <ctime>
#include <vector>
#include <algorithm>
#include <stdlib.h>
// utilities
#include <hip/hip_runtime_api.h>
#include <time.h>


///////////per request timing. L1 enabled. P100.
///////////using more than 8gb.


//typedef unsigned char byte;

void shuffle(int *array, size_t n)
{
    if (n > 1) 
    {
        size_t i;
        for (i = 0; i < n - 1; i++) 
        {
          size_t j = i + rand() / (RAND_MAX / (n - i) + 1);
          int t = array[j];
          array[j] = array[i];
          array[i] = t;
        }
    }
}

void init_cpu_data(unsigned *A, unsigned size, unsigned stride, unsigned mod, unsigned iterations){
	for (unsigned i = 0; i < size - stride; i = i + stride){
		A[i]=(i + stride);
   	}
	
	for (unsigned i = 7; i < size - stride; i = i + stride){
		A[i]=(i + stride);
   	}
	
	int rand_sequence[iterations];
	
	//////random sequence offset 0
	for(int i = 0; i < iterations; i++){
		rand_sequence[i] = i;
	}
	//srand (time(NULL));
	srand (0);
	shuffle(rand_sequence, iterations);
	
	unsigned previous_rand_num;
	unsigned rand_num = rand_sequence[0] * stride;	
	for(unsigned i = 1; i < iterations; i++){		
		previous_rand_num = rand_num;		
		rand_num = rand_sequence[i] * stride;		
		A[previous_rand_num]=rand_num;
	}
	
	//////random sequence offset 7	
	for(int i = 0; i < iterations; i++){
		rand_sequence[i] = i;
	}
	//srand (time(NULL));
	//shuffle(rand_sequence, iterations);
	
	rand_num = rand_sequence[0] * stride + 7;	
	for(unsigned i = 1; i < iterations; i++){		
		previous_rand_num = rand_num;		
		rand_num = rand_sequence[i] * stride + 7;		
		A[previous_rand_num]=rand_num;
	}
  
	/*
	///////manually set the nodes
	A[32]=104333344;
	A[104333344]=200802336;
	A[200802336]=353370144;
	A[353370144]=372244512;
	A[372244512]=110100512;
	A[110100512]=182452256;
	A[182452256]=333971488;
	A[333971488]=225443872;
	A[225443872]=155189280;
	A[155189280]=104333344;
	*/
	
	for (unsigned i = size - stride; i < size; i++){
		A[i]=0;
   	}
}

__device__ void P_chasing0(int mark, unsigned *A, int iterations, int *B, int *C, unsigned *D, int starting_index, float clock_rate, int data_stride){	
	
	int j = starting_index;/////make them in the same page, and miss near in cache lines
			
	for (int it = 0; it < iterations; it++){	
		j = A[j];		
	}	
		
	B[0] = j;
}

//////////min page size 4kb = 4096b = 32 * 128.
__device__ void P_chasing1(int mark, unsigned *A, unsigned iterations, unsigned *B, unsigned *C, long long int *D, unsigned starting_index, float clock_rate, unsigned data_stride){
	
	unsigned j = starting_index;/////make them in the same page, and miss near in cache lines
	
	//unsigned start_time = 0;//////clock
	//unsigned end_time = 0;//////clock
	//start_time = clock64();//////clock
			
	for (unsigned it = 0; it < iterations; it++){
		j = A[j];
	}
	
	//end_time=clock64();//////clock
	//unsigned total_time = end_time - start_time;//////clock
	//printf("inside%d:%fms\n", mark, (total_time / (float)clock_rate) / ((float)iterations));//////clock, average latency //////////the print will flush the L1?! (
	
	B[0] = j;
	//B[1] = (int) total_time;
}

//////////min page size 4kb = 4096b = 32 * 128.
__device__ void P_chasing2(int mark, unsigned *A, unsigned iterations, unsigned *B, unsigned *C, long long int *D, unsigned starting_index, float clock_rate, unsigned data_stride){//////what is the effect of warmup outside vs inside?
	
	//////shared memory: 0xc000 max (49152 Bytes = 48KB)
	__shared__ long long int s_tvalue[1024 * 4];/////must be enough to contain the number of iterations.
	__shared__ unsigned s_index[1024 * 4];
	//__shared__ unsigned s_index[1];
	
	unsigned j = starting_index;/////make them in the same page, and miss near in cache lines
	//int j = B[0];
	
	long long int start_time = 0;//////clock
	long long int end_time = 0;//////clock
	long long int time_interval = 0;//////clock
	//unsigned total_time = end_time - start_time;//////clock
	
	/*		
	for (int it = 0; it < iterations; it++){
		
		start_time = clock64();//////clock		
		j = A[j];
		//s_index[it] = j;
		end_time=clock64();//////clock		
		s_tvalue[it] = end_time - start_time;
	}
	*/
	
		asm(".reg .u64 t1;\n\t"
		".reg .u64 t2;\n\t");
	
	for (unsigned it = 0; it < iterations; it++){
		
		/*
		asm("mul.wide.u32 	t1, %3, %5;\n\t"	
		"add.u64 	t2, t1, %4;\n\t"		
		"mov.u64 	%0, %clock64;\n\t"		
		"ld.global.u32 	%2, [t2];\n\t"
		"mov.u64 	%1, %clock64;"
		: "=l"(start_time), "=l"(end_time), "=r"(j) : "r"(j), "l"(A), "r"(4));
		*/

		asm("mul.wide.u32 	t1, %2, %4;\n\t"	
		"add.u64 	t2, t1, %3;\n\t"		
		"mov.u64 	%0, %clock64;\n\t"		
		"ld.global.u32 	%1, [t2];\n\t"		
		: "=l"(start_time), "=r"(j) : "r"(j), "l"(A), "r"(4));
		
		s_index[it] = j;////what if without this? ///Then it is not accurate and cannot get the access time at all, due to the ILP. (another way is to use average time, but inevitably containing other instructions:setp, add).
		
		asm volatile ("mov.u64 %0, %clock64;": "=l"(end_time));
		
		time_interval = end_time - start_time;
		//if(it >= 4 * 1024){
		s_tvalue[it] = time_interval;
		//}
	}
	
	//printf("inside%d:%fms\n", mark, (total_time / (float)clock_rate) / ((float)iterations));//////clock, average latency
	
	B[0] = j;
	
	for (unsigned it = 0; it < iterations; it++){		
		C[it] = s_index[it];
		D[it] = s_tvalue[it];
	}
}

__global__ void tlb_latency_test(unsigned *A, unsigned iterations, unsigned *B, unsigned *C, long long int *D, float clock_rate, unsigned mod, int data_stride){
	
	unsigned reduced_iter = iterations;
	if(reduced_iter > 512){
		reduced_iter = 512;
	}else if(reduced_iter < 16){
		reduced_iter = 16;
	}
	
	///////////kepler L2 has 48 * 1024 = 49152 cache lines. But we only have 1024 * 4 slots in shared memory.
	P_chasing1(0, A, iterations + 0, B, C, D, 0, clock_rate, data_stride);////////saturate the L2
	P_chasing2(0, A, reduced_iter, B, C, D, 0, clock_rate, data_stride);////////partially print the data
	
	 __syncthreads();
}

int main(int argc, char **argv)
{
	printf("\n");
	
    // set device
    hipDeviceProp_t device_prop;
    //int dev_id = findCudaDevice(argc, (const char **) argv);
	int dev_id = 0;
    checkCudaErrors(hipGetDeviceProperties(&device_prop, dev_id));
	
	int peak_clk = 1;//kHz
	checkCudaErrors(hipDeviceGetAttribute(&peak_clk, hipDeviceAttributeClockRate, dev_id));
	float clock_rate = (float) peak_clk;
	
	//printf("clock_rate_out_kernel:%f\n", clock_rate);

    if (!device_prop.managedMemory) { 
        // This samples requires being run on a device that supports Unified Memory
        fprintf(stderr, "Unified Memory not supported on this device\n");

        exit(EXIT_WAIVED);
    }

    if (device_prop.computeMode == hipComputeModeProhibited)
    {
        // This sample requires being run with a default or process exclusive mode
        fprintf(stderr, "This sample requires a device in either default or process exclusive mode\n");

        exit(EXIT_WAIVED);
    }
	
	///////////////////////////////////////////////////////////////////GPU data out
	unsigned *GPU_data_out;
	checkCudaErrors(hipMalloc(&GPU_data_out, sizeof(unsigned) * 2));			
	
	FILE * pFile;
    pFile = fopen ("output.txt","w");		
	
	unsigned counter = 0;
	for(unsigned data_stride = 2 * 256 * 1024; data_stride <= 2 * 256 * 1024; data_stride = data_stride * 2){/////////32mb stride
		//data_stride = data_stride + 32;///offset a cache line, trying to cause L2 miss but tlb hit.
		//printf("###################data_stride%d#########################\n", data_stride);
	//for(int mod = 1024 * 256 * 2; mod > 0; mod = mod - 32 * 1024){/////kepler L2 1.5m = 12288 cache lines, L1 16k = 128 cache lines.
	for(unsigned mod2 = 2 * 256 * 1024; mod2 <= 1073741824; mod2 = mod2 * 2){////268435456 = 1gb, 536870912 = 2gb, 1073741824 = 4gb, 2147483648 = 8gb, 4294967296 = 16gb.
		counter++;
		///////////////////////////////////////////////////////////////////CPU data begin
		//int data_size = 2 * 256 * 1024 * 32;/////size = iteration * stride = 32 2mb pages.
		unsigned mod = mod2;
		if(mod > 3221225472){
			mod = 3221225472;
		}
		unsigned data_size = mod;
		if(data_size < 4194304){//////////data size at least 16mb to prevent L2 prefetch
			data_size = 4194304;
		}
		//int iterations = data_size / data_stride;
		//int iterations = 1024 * 256 * 8;
		unsigned iterations = mod / data_stride;////32 * 32 * 4 / 32 * 2 = 256
	
		unsigned *CPU_data_in;
		CPU_data_in = (unsigned*)malloc(sizeof(unsigned) * data_size);
		init_cpu_data(CPU_data_in, data_size, data_stride, mod, iterations);
		
		
		unsigned reduced_iter = iterations;
		if(reduced_iter > 512){
			reduced_iter = 512;
		}else if(reduced_iter < 16){
			reduced_iter = 16;
		}
		
		unsigned *CPU_data_out_index;
		CPU_data_out_index = (unsigned*)malloc(sizeof(unsigned) * reduced_iter);
		long long int *CPU_data_out_time;
		CPU_data_out_time = (long long int*)malloc(sizeof(long long int) * reduced_iter);
		///////////////////////////////////////////////////////////////////CPU data end	
	
		///////////////////////////////////////////////////////////////////GPU data in	
		unsigned *GPU_data_in;
		checkCudaErrors(hipMalloc(&GPU_data_in, sizeof(unsigned) * data_size));	
		hipMemcpy(GPU_data_in, CPU_data_in, sizeof(unsigned) * data_size, hipMemcpyHostToDevice);
		
		///////////////////////////////////////////////////////////////////GPU data out
		unsigned *GPU_data_out_index;
		checkCudaErrors(hipMalloc(&GPU_data_out_index, sizeof(unsigned) * reduced_iter));
		long long int *GPU_data_out_time;
		checkCudaErrors(hipMalloc(&GPU_data_out_time, sizeof(long long int) * reduced_iter));
		
		tlb_latency_test<<<1, 1>>>(GPU_data_in, iterations, GPU_data_out, GPU_data_out_index, GPU_data_out_time, clock_rate, mod, data_stride);///////////////kernel is here	
		hipDeviceSynchronize();
				
		hipMemcpy(CPU_data_out_index, GPU_data_out_index, sizeof(unsigned) * reduced_iter, hipMemcpyDeviceToHost);
		hipMemcpy(CPU_data_out_time, GPU_data_out_time, sizeof(long long int) * reduced_iter, hipMemcpyDeviceToHost);
				

		fprintf(pFile, "###################data_stride%u#########################\n", data_stride);
		fprintf (pFile, "###############Mod%u##############%u\n", mod, iterations);
		for (unsigned it = 0; it < reduced_iter; it++){			
			fprintf (pFile, "%u %fms %lldcycles\n", CPU_data_out_index[it], (double)CPU_data_out_time[it] / (float)clock_rate, CPU_data_out_time[it]);
			//fprintf (pFile, "%d %fms\n", it, CPU_data_out_time[it] / (float)clock_rate);
			//printf ("%d %fms\n", CPU_data_out_index[it], CPU_data_out_time[it] / (float)clock_rate);
		}
		
		checkCudaErrors(hipFree(GPU_data_out_index));
		checkCudaErrors(hipFree(GPU_data_out_time));
		checkCudaErrors(hipFree(GPU_data_in));
		free(CPU_data_in);
		free(CPU_data_out_index);
		free(CPU_data_out_time);
	}
	
	for(unsigned mod2 = 1; mod2 <= 1; mod2 = mod2 * 2){////268435456 = 1gb, 536870912 = 2gb, 1073741824 = 4gb, 2147483648 = 8gb, 4294967296 = 16gb.
		counter++;
		///////////////////////////////////////////////////////////////////CPU data begin
		//int data_size = 2 * 256 * 1024 * 32;/////size = iteration * stride = 32 2mb pages.
		unsigned mod = 2147483648;
		if(mod > 3221225472){
			mod = 3221225472;
		}
		unsigned data_size = mod;
		if(data_size < 4194304){//////////data size at least 16mb to prevent L2 prefetch
			data_size = 4194304;
		}
		//int iterations = data_size / data_stride;
		//int iterations = 1024 * 256 * 8;
		unsigned iterations = mod / data_stride;////32 * 32 * 4 / 32 * 2 = 256
	
		unsigned *CPU_data_in;
		CPU_data_in = (unsigned*)malloc(sizeof(unsigned) * data_size);
		init_cpu_data(CPU_data_in, data_size, data_stride, mod, iterations);
		
		
		unsigned reduced_iter = iterations;
		if(reduced_iter > 512){
			reduced_iter = 512;
		}else if(reduced_iter < 16){
			reduced_iter = 16;
		}
		
		unsigned *CPU_data_out_index;
		CPU_data_out_index = (unsigned*)malloc(sizeof(unsigned) * reduced_iter);
		long long int *CPU_data_out_time;
		CPU_data_out_time = (long long int*)malloc(sizeof(long long int) * reduced_iter);
		///////////////////////////////////////////////////////////////////CPU data end	
	
		///////////////////////////////////////////////////////////////////GPU data in	
		unsigned *GPU_data_in;
		checkCudaErrors(hipMalloc(&GPU_data_in, sizeof(unsigned) * data_size));	
		hipMemcpy(GPU_data_in, CPU_data_in, sizeof(unsigned) * data_size, hipMemcpyHostToDevice);
		
		///////////////////////////////////////////////////////////////////GPU data out
		unsigned *GPU_data_out_index;
		checkCudaErrors(hipMalloc(&GPU_data_out_index, sizeof(unsigned) * reduced_iter));
		long long int *GPU_data_out_time;
		checkCudaErrors(hipMalloc(&GPU_data_out_time, sizeof(long long int) * reduced_iter));
		
		tlb_latency_test<<<1, 1>>>(GPU_data_in, iterations, GPU_data_out, GPU_data_out_index, GPU_data_out_time, clock_rate, mod, data_stride);///////////////kernel is here	
		hipDeviceSynchronize();
				
		hipMemcpy(CPU_data_out_index, GPU_data_out_index, sizeof(unsigned) * reduced_iter, hipMemcpyDeviceToHost);
		hipMemcpy(CPU_data_out_time, GPU_data_out_time, sizeof(long long int) * reduced_iter, hipMemcpyDeviceToHost);
				

		fprintf(pFile, "###################data_stride%u#########################\n", data_stride);
		fprintf (pFile, "###############Mod%u##############%u\n", mod, iterations);
		for (unsigned it = 0; it < reduced_iter; it++){			
			fprintf (pFile, "%u %fms %lldcycles\n", CPU_data_out_index[it], (double)CPU_data_out_time[it] / (float)clock_rate, CPU_data_out_time[it]);
			//fprintf (pFile, "%d %fms\n", it, CPU_data_out_time[it] / (float)clock_rate);
			//printf ("%d %fms\n", CPU_data_out_index[it], CPU_data_out_time[it] / (float)clock_rate);
		}
		
		checkCudaErrors(hipFree(GPU_data_out_index));
		checkCudaErrors(hipFree(GPU_data_out_time));
		checkCudaErrors(hipFree(GPU_data_in));
		free(CPU_data_in);
		free(CPU_data_out_index);
		free(CPU_data_out_time);
	}
	
	for(unsigned mod2 = 1; mod2 <= 1; mod2 = mod2 * 2){////268435456 = 1gb, 536870912 = 2gb, 1073741824 = 4gb, 2147483648 = 8gb, 4294967296 = 16gb.
		counter++;
		///////////////////////////////////////////////////////////////////CPU data begin
		//int data_size = 2 * 256 * 1024 * 32;/////size = iteration * stride = 32 2mb pages.
		unsigned mod = 3221225472;
		if(mod > 3221225472){
			mod = 3221225472;
		}
		unsigned data_size = mod;
		if(data_size < 4194304){//////////data size at least 16mb to prevent L2 prefetch
			data_size = 4194304;
		}
		//int iterations = data_size / data_stride;
		//int iterations = 1024 * 256 * 8;
		unsigned iterations = mod / data_stride;////32 * 32 * 4 / 32 * 2 = 256
	
		unsigned *CPU_data_in;
		CPU_data_in = (unsigned*)malloc(sizeof(unsigned) * data_size);
		init_cpu_data(CPU_data_in, data_size, data_stride, mod, iterations);
		
		
		unsigned reduced_iter = iterations;
		if(reduced_iter > 512){
			reduced_iter = 512;
		}else if(reduced_iter < 16){
			reduced_iter = 16;
		}
		
		unsigned *CPU_data_out_index;
		CPU_data_out_index = (unsigned*)malloc(sizeof(unsigned) * reduced_iter);
		long long int *CPU_data_out_time;
		CPU_data_out_time = (long long int*)malloc(sizeof(long long int) * reduced_iter);
		///////////////////////////////////////////////////////////////////CPU data end	
	
		///////////////////////////////////////////////////////////////////GPU data in	
		unsigned *GPU_data_in;
		checkCudaErrors(hipMalloc(&GPU_data_in, sizeof(unsigned) * data_size));	
		hipMemcpy(GPU_data_in, CPU_data_in, sizeof(unsigned) * data_size, hipMemcpyHostToDevice);
		
		///////////////////////////////////////////////////////////////////GPU data out
		unsigned *GPU_data_out_index;
		checkCudaErrors(hipMalloc(&GPU_data_out_index, sizeof(unsigned) * reduced_iter));
		long long int *GPU_data_out_time;
		checkCudaErrors(hipMalloc(&GPU_data_out_time, sizeof(long long int) * reduced_iter));
		
		tlb_latency_test<<<1, 1>>>(GPU_data_in, iterations, GPU_data_out, GPU_data_out_index, GPU_data_out_time, clock_rate, mod, data_stride);///////////////kernel is here	
		hipDeviceSynchronize();
				
		hipMemcpy(CPU_data_out_index, GPU_data_out_index, sizeof(unsigned) * reduced_iter, hipMemcpyDeviceToHost);
		hipMemcpy(CPU_data_out_time, GPU_data_out_time, sizeof(long long int) * reduced_iter, hipMemcpyDeviceToHost);
				

		fprintf(pFile, "###################data_stride%u#########################\n", data_stride);
		fprintf (pFile, "###############Mod%u##############%u\n", mod, iterations);
		for (unsigned it = 0; it < reduced_iter; it++){			
			fprintf (pFile, "%u %fms %lldcycles\n", CPU_data_out_index[it], (double)CPU_data_out_time[it] / (float)clock_rate, CPU_data_out_time[it]);
			//fprintf (pFile, "%d %fms\n", it, CPU_data_out_time[it] / (float)clock_rate);
			//printf ("%d %fms\n", CPU_data_out_index[it], CPU_data_out_time[it] / (float)clock_rate);
		}
		
		checkCudaErrors(hipFree(GPU_data_out_index));
		checkCudaErrors(hipFree(GPU_data_out_time));
		checkCudaErrors(hipFree(GPU_data_in));
		free(CPU_data_in);
		free(CPU_data_out_index);
		free(CPU_data_out_time);
	}
		//printf("############################################\n\n");
	}
			
	checkCudaErrors(hipFree(GPU_data_out));	
	//free(CPU_data_out);
	fclose (pFile);
	
    exit(EXIT_SUCCESS);
}
