#include "hip/hip_runtime.h"
#include <cstdio>
#include <ctime>
#include <vector>
#include <algorithm>
#include <stdlib.h>
// utilities
#include <hip/hip_runtime_api.h>
#include <time.h>

void init_cpu_data(int* A, int size, int stride){
	for (int i = 0; i < size; ++i){
		A[i]=(i + stride) % size;
   	}
}

/*
__device__ void cache_warmup(int *A, int iterations, int *B){
	
	int j = 0;
	
	for (int it =0; it < iterations; it ++){
		j = A[j];
	}
	
	B[0] = j;
}
*/

//////////min page size 4kb = 4096b = 32 * 128.
__device__ void P_chasing(int mark, int *A, int iterations, int *B, int starting_index, float clock_rate){//////////////should not hit in the tlb, and should also miss in the cache, to see the time difference.
	
	int j = starting_index;/////make them in the same page, and miss near in cache lines
	
	long long int start_time = 0;//////clock
	long long int end_time = 0;//////clock
	start_time = clock64();//////clock
	
	for (int it =0; it < iterations; it ++){
		j = A[j];
	}
	
	end_time=clock64();//////clock
	long long int total_time = end_time - start_time;//////clock
	printf("inside%d:%fms\n", mark, total_time / (float)clock_rate);//////clock
	
	B[0] = j;
}

__global__ void tlb_latency_test(int *A, int iterations, int *B, float clock_rate){	

	long long int start_time = 0;///////////clock
	long long int end_time = 0;///////////clock	
	start_time = clock64();///////////clock
	
	P_chasing(0, A, 16, B, 0 * 32, clock_rate);/////TLB warmup
	P_chasing(1, A, 16, B, 0 * 32 + 6, clock_rate);/////make them in the same page, and hit near in cache lines
	P_chasing(2, A, 16, B, 0 * 32 + 7, clock_rate);/////make them in the same page, and hit near in cache lines
	P_chasing(3, A, 16, B, 0 * 32 + 8, clock_rate);/////make them in the same page, and hit near in cache lines
	P_chasing(4, A, 16, B, 0 * 32 + 14, clock_rate);/////////////make them in the same page, and hit far in cache lines
	P_chasing(5, A, 16, B, 0 * 32 + 15, clock_rate);////////////make them in the same page, and hit far in cache lines
	P_chasing(6, A, 16, B, 0 * 32 + 16, clock_rate);////////////make them in the same page, and hit far in cache lines
	P_chasing(7, A, 16, B, 1 * 32, clock_rate);/////make them in the same page, and miss near in cache lines
	P_chasing(8, A, 16, B, 2 * 32, clock_rate);/////make them in the same page, and miss near in cache lines
	P_chasing(9, A, 16, B, 3 * 32, clock_rate);/////make them in the same page, and miss near in cache lines
	P_chasing(10, A, 16, B, 8 * 32, clock_rate);//////////////make them in the same page, and miss near in cache lines
	P_chasing(11, A, 16, B, 16 * 32, clock_rate);/////////////make them in the same page, and miss near in cache lines
	P_chasing(12, A, 16, B, 24 * 32, clock_rate);/////////////make them in the same page, and miss near in cache lines
	P_chasing(13, A, 16, B, 16 * 524288, clock_rate);//////////////TLB miss, 17th page
	P_chasing(14, A, 16, B, 32 * 524288, clock_rate);/////////////TLB miss, 33rd page
	P_chasing(15, A, 16, B, 48 * 32, clock_rate);/////////////TLB miss, 49th page
	
	end_time=clock64();///////////clock
		
	long long int total_time = end_time - start_time;///////////clock
	printf("outside1:%fms\n", total_time / (float)clock_rate);///////////clock
}

int main(int argc, char **argv)
{
	printf("\n");
	
    // set device
    hipDeviceProp_t device_prop;
    //int dev_id = findCudaDevice(argc, (const char **) argv);
	int dev_id = 0;
    checkCudaErrors(hipGetDeviceProperties(&device_prop, dev_id));
	
	int peak_clk = 1;//kHz
	checkCudaErrors(hipDeviceGetAttribute(&peak_clk, hipDeviceAttributeClockRate, dev_id));
	float clock_rate = (float) peak_clk;
	
	//printf("clock_rate_out_kernel:%f\n", clock_rate);

    if (!device_prop.managedMemory) { 
        // This samples requires being run on a device that supports Unified Memory
        fprintf(stderr, "Unified Memory not supported on this device\n");

        exit(EXIT_WAIVED);
    }

    if (device_prop.computeMode == hipComputeModeProhibited)
    {
        // This sample requires being run with a default or process exclusive mode
        fprintf(stderr, "This sample requires a device in either default or process exclusive mode\n");

        exit(EXIT_WAIVED);
    }
		
	///////////////////////////////////////////////////////////////////CPU data begin	
	int iterations = 100;
	////////size(int) = 4, 32 = 128b, 256 = 1kb, 32 * 32 = 1024 = 4kb, 262144 = 1mb, 524288 = 2mb.
	int data_stride = 524288;/////2mb. Pointing to the next page.
	//int data_size = 524288000;/////1000 * 2mb. ##### size = iteration * stride. ##### This can support 1000 iteration. The 1001st iteration starts from head again.
	int data_size = iterations * data_stride;/////size = iteration * stride = 100 2mb pages.
	
	int *CPU_data_in;	
	CPU_data_in = (int*)malloc(sizeof(int) * data_size);
	//int *CPU_data_out;
	//CPU_data_out = (int*)malloc(data_size * sizeof(int));
	
	init_cpu_data(CPU_data_in, data_size, data_stride);
	///////////////////////////////////////////////////////////////////CPU data end
	
	///////////////////////////////////////////////////////////////////GPU data begin
	int *GPU_data_in;
	//////checkCudaErrors(hipMallocManaged(&data, sizeof(int) * data_size));
	checkCudaErrors(hipMalloc(&GPU_data_in, sizeof(int) * data_size));
	
	int *GPU_data_out;
	checkCudaErrors(hipMalloc(&GPU_data_out, sizeof(int) * 1));
	
	hipMemcpy(GPU_data_in, CPU_data_in, sizeof(int) * data_size, hipMemcpyHostToDevice);
	///////////////////////////////////////////////////////////////////GPU data end				  
		
	tlb_latency_test<<<1, 1>>>(GPU_data_in, iterations, GPU_data_out, clock_rate);//////////////////////////////////////////////kernel is here
	
	//hipMemcpy(CPU_data_out, GPU_data_out, sizeof(int) * data_size, hipMemcpyDeviceToHost);
	
    hipDeviceSynchronize();	
	
    exit(EXIT_SUCCESS);
}
