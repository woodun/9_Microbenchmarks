#include "hip/hip_runtime.h"
#include <cstdio>
#include <ctime>
#include <vector>
#include <algorithm>
#include <stdlib.h>
// utilities
#include <hip/hip_runtime_api.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_cooperative_groups.h>

using namespace cooperative_groups;

/////////////////////////////L1 is enabled. "ALL_CCFLAGS += -Xptxas -dlcm=ca"
//////////////large vs small data.

void init_cpu_data(long long int* A, long long int size, double stride){
	
	for (long long int i = 0; i < size; i++){
		A[i]=1;
	}
	
	/*
	for (long long int i = 0; i < size - stride; i++){
		A[i]=(i + stride);
	}
			
	for (long long int i = size - stride; i < size; i++){
		A[i]=0;
	}
	*/
}

__global__ void gpu_initialization(long long int *A, double data_stride, long long int data_size){			

	long long int index = (blockIdx.x * blockDim.x + threadIdx.x);
	long long int thread_num =  gridDim.x * blockDim.x;
	
	for(long long int it = 0; it < data_size; it = it + thread_num){
		A[index + it]=23;
	}
}

long long unsigned time_diff(timespec start, timespec end){
	struct timespec temp;
	if ((end.tv_nsec - start.tv_nsec) < 0){
		temp.tv_sec = end.tv_sec - start.tv_sec - 1;
		temp.tv_nsec = 1000000000 + end.tv_nsec - start.tv_nsec;
	} 
	else{
		temp.tv_sec = end.tv_sec - start.tv_sec;
		temp.tv_nsec = end.tv_nsec - start.tv_nsec;
	}
	
	long long unsigned time_interval_ns = temp.tv_nsec;
	long long unsigned time_interval_s = temp.tv_sec;
	time_interval_s = time_interval_s * 1000000000;
	
	return time_interval_s + time_interval_ns;
}

//__global__ void Page_visitor(long long int *A, long long int *B, long long int data_stride, long long int clock_count){
__global__ void Page_visitor(long long int *A1, long long int *A2, long long int *B, double data_stride, long long int clock_count){////load-compute-store
			
	thread_block block = this_thread_block();
	
	long long int index = (blockIdx.x * 512 + (threadIdx.x - 32) ) * data_stride;
	long long int value1;
	long long int prefetch_index = (blockIdx.x * 512 + threadIdx.x * 16) * data_stride;
	long long int value2;
	
	if(threadIdx.x > 31){
		value1 = A1[index];
	}else{
		
		/*
		asm volatile(".reg.u64  t1;\n\t"
		".reg.u64  t2;\n\t"
		".reg.u64  t3;\n\t"
		"shl.b64  t2, %1, 3;\n\t"
		"cvta.to.global.u64  t1, %2;\n\t"
		"add.s64  t3, t2, t1;\n\t"		
		"ld.global.u64 	%0, [t3];"
		: "=l"(value2) : "l"(prefetch_index), "l"(A2));		
		*/
		
		value2 = A2[prefetch_index];
	}
	
	//block.sync();
	
	if(threadIdx.x > 31){
		//////////////////////////////////////////////loop
		long long int clock_offset = 0;
		while (clock_offset < clock_count){/////////////////what's the time overhead for addition and multiplication?
			clock_offset++;
			value1 = value1 + threadIdx.x;
		}
		
		value2 = A2[index];
	}else{
		
		/*
		asm volatile("cvta.to.global.u64  t1, %1;\n\t"
		"add.s64  t3, t2, t1;\n\t"		
		"st.global.u64 	[t3], %0;"
		: "=l"(value2) : "l"(B));
		*/
		
		B[prefetch_index] = value2;
	}	
	
	//block.sync();
	
	if(threadIdx.x > 31){
		//////////////////////////////////////////////loop
		long long int clock_offset2 = 0;
		while (clock_offset2 < clock_count){/////////////////what's the time overhead for addition and multiplication?
			clock_offset2++;
			value2 = value2 + threadIdx.x;
		}
	
		B[index] = value1 + value2;
	}
	
	/*
	if(threadIdx.x == 0){/////%tid %ntid %laneid %warpid %nwarpid %ctaid %nctaid %smid %nsmid %gridid
		int smid = 1;
		asm("mov.u32 %0, %smid;" : "=r"(smid) );
		printf("blockIdx.x: %d, smid: %d\n", blockIdx.x, smid);
		if(blockIdx.x == 55){
			int nsmid = 1;
			asm("mov.u32 %0, %smid;" : "=r"(nsmid) );
			printf("nsmid: %d\n", nsmid);
		}
	}
	*/
}

int main(int argc, char **argv)
{
	printf("\n");
	
    // set device
    hipDeviceProp_t device_prop;
    //long long int dev_id = findCudaDevice(argc, (const char **) argv);
	long long int dev_id = 0;
    checkCudaErrors(hipGetDeviceProperties(&device_prop, dev_id));
	
	int peak_clk = 1;//kHz
	checkCudaErrors(hipDeviceGetAttribute(&peak_clk, hipDeviceAttributeClockRate, dev_id));
	float clock_rate = (float) peak_clk;
	
	printf("clock_rate:%f\n", clock_rate);

    if (!device_prop.managedMemory) { 
        // This samples requires being run on a device that supports Unified Memory
        fprintf(stderr, "Unified Memory not supported on this device\n");

        exit(EXIT_WAIVED);
    }

    if (device_prop.computeMode == hipComputeModeProhibited)
    {
        // This sample requires being run with a default or process exclusive mode
        fprintf(stderr, "This sample requires a device in either default or process exclusive mode\n");

        exit(EXIT_WAIVED);
    }
	
	if (device_prop.concurrentManagedAccess == 1){
		printf("This device supports concurrent Managed Access.\n");
    }else{
		printf("This device does not support concurrent Managed Access.\n");
	}
	
	int value1 = 1;
	checkCudaErrors(hipDeviceGetAttribute(&value1, hipDeviceAttributeConcurrentManagedAccess, dev_id));
	printf("hipDeviceAttributeConcurrentManagedAccess = %d\n", value1);	
	
	//plain managed
	//when was 64k and 4k pages used?
	//how to decrease the overhead of sync?
	//printf("###################\n#########################managed\n");
	/*
	for(long long int factor = 1; factor <= 128; factor = factor * 2){
	for(double data_stride = 1 * 1 * 0.25 * factor; data_stride <= 1 * 1 * 4 * factor; data_stride = data_stride * 2){///134217728 = 1gb, 268435456 = 2gb, 536870912 = 4gb, 1073741824 = 8gb, 2147483648 = 16gb, 4294967296 = 32gb, 8589934592 = 64gb. (index)
	for(long long int clock_count = 512; clock_count <= 16384; clock_count = clock_count * 2){
	*/
	for(long long int factor = 1; factor <= 1; factor = factor * 2){
	for(double data_stride = 1 * 1 * 1 * factor; data_stride <= 1 * 1 * 1 * factor; data_stride = data_stride * 2){///134217728 = 1gb, 268435456 = 2gb, 536870912 = 4gb, 1073741824 = 8gb, 2147483648 = 16gb, 4294967296 = 32gb, 8589934592 = 64gb. (index)
	for(long long int clock_count = 4096; clock_count <= 4096; clock_count = clock_count * 2){
		///////////////////////////////////////////////////////////////////CPU data begin		
		long long int data_size = data_stride * 512;
		data_size = data_size * 8192 * 512 / factor;
		
		long long int *CPU_data_in1;
		checkCudaErrors(hipMallocManaged(&CPU_data_in1, sizeof(long long int) * data_size));/////////////using unified memory
		long long int *CPU_data_in2;
		checkCudaErrors(hipMallocManaged(&CPU_data_in2, sizeof(long long int) * data_size));/////////////using unified memory		
		///////////////////////////////////////////////////////////////////CPU data end
		
		long long int *GPU_data_out;
		checkCudaErrors(hipMallocManaged(&GPU_data_out, sizeof(long long int) * data_size));/////////////using unified memory
		///////////////////////////////////////////////////////////////////GPU data out	end
		
		if(1){
		gpu_initialization<<<8192 * 512 / factor, 512>>>(GPU_data_out, data_stride, data_size);///////////////1024 per block max
		hipDeviceSynchronize();
		gpu_initialization<<<8192 * 512 / factor, 512>>>(CPU_data_in2, data_stride, data_size);///////////////1024 per block max
		hipDeviceSynchronize();
		gpu_initialization<<<8192 * 512 / factor, 512>>>(CPU_data_in1, data_stride, data_size);///////////////1024 per block max
		hipDeviceSynchronize();
		}else{
		init_cpu_data(GPU_data_out, data_size, data_stride);
		init_cpu_data(CPU_data_in2, data_size, data_stride);
		init_cpu_data(CPU_data_in1, data_size, data_stride);
		}
		
		/////////////////////////////////time
		struct timespec ts1;
		clock_gettime(CLOCK_REALTIME, &ts1);

		////may want to use more thread to see clock_count effect
		Page_visitor<<<8192 * 512 / factor, 512>>>(CPU_data_in1, CPU_data_in2, GPU_data_out, data_stride, clock_count);///1024 per block max
		///////////////////////////////////////////////////32 * 64 * 1 * 512 * 1024 = 8gb.
		hipDeviceSynchronize();
				
		/////////////////////////////////time
		struct timespec ts2;
		clock_gettime(CLOCK_REALTIME, &ts2);
		
		//printf("###################data_stride%lld#########################clock_count:%lld\n", data_stride, clock_count);
		//printf("*\n*\n*\nruntime:  %lluns\n", time_diff(ts1, ts2));
		printf("%llu ", time_diff(ts1, ts2));
		
		checkCudaErrors(hipFree(CPU_data_in1));
		checkCudaErrors(hipFree(CPU_data_in2));
		checkCudaErrors(hipFree(GPU_data_out));
	}
	printf("\n");
	}
	printf("####################%llu\n", factor);
	}

	exit(EXIT_SUCCESS);
}