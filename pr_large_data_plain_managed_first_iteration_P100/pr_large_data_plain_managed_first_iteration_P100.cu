#include "hip/hip_runtime.h"
#include <cstdio>
#include <ctime>
#include <vector>
#include <algorithm>
#include <stdlib.h>
// utilities
#include <hip/hip_runtime_api.h>
#include <time.h>

/////////////////////////////change the data size to larger than 16 gb to test for different memories. L1 is enabled. "ALL_CCFLAGS += -Xptxas -dlcm=ca"

void init_cpu_data(long long int* A, long long int size, long long int stride, long long int mod){
	if(1){////////////normal
		for (long long int i = 0; i < size - stride; i = i + stride){
			A[i]=(i + stride);
		}
		A[size - stride]=0;
		
		long long int stride2 = 1 * 256 * 1024;////////2m
		for (long long int i = 8; i < size - stride2; i = i + stride2){
			A[i]=(i + stride2);
		}		
		A[size - stride2 + 8]=0;
	}
	
	if(0){////////////reversed
		//for (long long int i = 0; i <= size - stride; i = i + stride){
		//	A[i]=(i - stride);
		//}
		
		for (long long int i = 3; i <= size - stride + 3; i = i + stride){
			A[i]=(i - stride);
		}
		
		//A[0]=size - stride;
		A[3]=size - stride + 3;
	}
	
	/////54521859 returned page fault starting point for 2147483648.
	///////////////////2147483648 - 54521859 = 2092961789.
	///////////////////2092961789 -4096 + 3 = 1996 * 1M = 15968 MB (out of 16280 MB out of 16384 MB)
	/////2202267651 returned page fault starting point for 4294967296
	///////////////////4294967296 - 2202267651 = 2092699645.
	///////////////////2092699645 -4096 + 3 = 1995.75 * 1M = 15966 MB (out of 16280 MB out of 16384 MB)
}

long long int traverse_cpu_data(long long int *A, long long int iterations, long long int starting_index, long long int data_stride){	
	
	long long int j = starting_index;
			
	for (long long int it = 0; it < iterations; it++){
		j = A[j];
	}
	
	return j;
}

__device__ void P_chasing2(int mark, long long int *A, long long int iterations, long long int *B, long long int starting_index, float clock_rate, long long int data_stride){	
	
	__shared__ long long int s_index[1];
	
	long long int j = starting_index;/////make them in the same page, and miss near in cache lines
	
	long long int start_time = 0;//////clock
	long long int end_time = 0;//////clock
	long long int time_interval = 0;//////clock	
		
	if(true){
		if(mark){
		asm(".reg .u64 t1;\n\t"
		".reg .u64 t2;\n\t"
		".reg .u32 t3;\n\t"
		".reg .u32 t4;\n\t"
		".reg .u64 t5;\n\t"
		".reg .u32 t6;\n\t");
		}
		
		asm("cvta.to.shared.u64 	t5, %0;\n\t"
		"cvt.u32.u64 	t6, t5;\n\t"
		:: "l"(s_index));////////////////////////////////////cvta.to.global.u64 	%rd4, %rd25; needed??
		
		for (long long int it = 0; it < iterations; it++){//////////it here is limited by the size of the shared memory
			
			asm("shl.b64 	t1, %3, 3;\n\t"	
			"add.s64 	t2, t1, %4;\n\t"
			"shl.b32 	t3, %6, 3;\n\t"
			"add.s32 	t4, t3, t6;\n\t"		
			"mov.u64 	%0, %clock64;\n\t"
			"ld.global.u64 	%2, [t2];\n\t"
			"st.shared.u64 	[t4], %2;\n\t"
			"mov.u64	%1, %clock64;"
			: "=l"(start_time), "=l"(end_time), "=l"(j) : "l"(j), "l"(A), "l"(s_index), "r"(0));		
					
			time_interval = end_time - start_time;
			printf("%lld %lld\n", j, time_interval);/////printf will affect L1 cache. Also, unknown effect to TLBs because it adds latency to L2 TLB misses.
			//////////////////////////////////////We are not using it for measurement. However, it can be used to recognize different conditions.
		}
	}

	B[0] = j;
}

__global__ void tlb_latency_test(long long int *A, long long int iterations, long long int *B, float clock_rate, long long int mod, long long int data_stride){
			
	/*
	/////////////using 32gb's iteration, address stride 1 * 128 * 1024, long long int data type
	P_chasing2(1, A, iterations/4, B, 2147483648, clock_rate, data_stride);//////////////migrate the first 8gb, starting 16 gb however.
	P_chasing2(0, A, iterations/8, B, 2147483648, clock_rate, data_stride);//////////////access the first 4gb again, starting 16 gb however.
	///////////before the next migration, take a rest, and see if the additional overhead decreases.
	long long int start_clock = clock64();
    long long int clock_offset = 0;
    while (clock_offset < 1000000000)
    {
        clock_offset = clock64() - start_clock;
    }
	P_chasing2(0, A, 3 * iterations/8, B, 0, clock_rate, data_stride);///////////migrate another 12gb, however starting at 0.
	P_chasing2(0, A, iterations/4, B, 2147483648, clock_rate, data_stride);//////////////which 4gb of the first 8gb is left?
	///////////conclusion: Still this does not change the pattern, the previous conclusion holds.
	*/
	/*
	/////////////using 32gb's iteration, address stride 1 * 128 * 1024, long long int data type
	P_chasing2(1, A, iterations/4, B, 0, clock_rate, data_stride);//////////////migrate the first 8gb
	P_chasing2(0, A, iterations/8, B, 0, clock_rate, data_stride);//////////////access the first 4gb again
	P_chasing2(0, A, 3 * iterations/8, B, 2147483648, clock_rate, data_stride);///////////migrate another 12gb, however starting at 16gb.
	P_chasing2(0, A, iterations/4, B, 0, clock_rate, data_stride);//////////////which 4gb of the first 8gb is left? what's the migration latency again?
	///////////conclusion: last access of first 8gb has low latency, 
	///////////and starting at 16gb (not continue at 8gb) the latency of first 16gb migration is still increasing as the same (with similar values).
	///////////It means that there is an additional warm up latency for the 2M group initialization.
	///////////And it is relating to the memory's physical locations itself, not relating to the address of the data.
	*/
	/*
	/////////////using 32gb's iteration, address stride 1 * 128 * 1024, long long int data type
	P_chasing2(1, A, iterations/4, B, 0, clock_rate, data_stride);//////////////migrate the first 8gb
	P_chasing2(0, A, iterations/8, B, 0, clock_rate, data_stride);//////////////access the first 4gb again
	P_chasing2(0, A, 3 * iterations/8, B, 1073741824, clock_rate, data_stride);///////////migrate another 12gb
	P_chasing2(0, A, iterations/4, B, 671088640, clock_rate, data_stride);//////////////which 4gb of the first 8gb is left? starting at 5gb.
	////////////////conclusion: the latter 4gb was left, even though the first 4gb is last accessed. The LRU is for migration not for access.
	*/
	//P_chasing2(1, A, iterations, B, 0, clock_rate, data_stride);
	//P_chasing2(0, A, iterations, B, mod - data_stride + 3, clock_rate, data_stride);
	
	__syncthreads();
}

__global__ void tlb_latency_test2(long long int *A, long long int iterations, long long int *B, float clock_rate, long long int mod, long long int data_stride){
			
	P_chasing2(1, A, iterations, B, 0, clock_rate, data_stride);//////////////migrate the first 8gb	
	//P_chasing2(1, A, iterations, B, 0, clock_rate, data_stride);
	//P_chasing2(0, A, iterations, B, mod - data_stride + 3, clock_rate, data_stride);
	
	__syncthreads();
}

__global__ void tlb_latency_test3(long long int *A, long long int iterations, long long int *B, float clock_rate, long long int mod, long long int data_stride){
			
	P_chasing2(1, A, iterations, B, 2281701376, clock_rate, data_stride);//////////////starting 17gb	
	//P_chasing2(1, A, iterations, B, 0, clock_rate, data_stride);
	//P_chasing2(0, A, iterations, B, mod - data_stride + 3, clock_rate, data_stride);
	
	__syncthreads();
}

__global__ void tlb_latency_test4(long long int *A, long long int iterations, long long int *B, float clock_rate, long long int mod, long long int data_stride){
			
	P_chasing2(1, A, iterations, B, 8, clock_rate, data_stride);//////////////starting 8, with a different stride.
	//P_chasing2(1, A, iterations, B, 0, clock_rate, data_stride);
	//P_chasing2(0, A, iterations, B, mod - data_stride + 3, clock_rate, data_stride);
	
	__syncthreads();
}

__global__ void tlb_latency_test5(long long int *A, long long int iterations, long long int *B, float clock_rate, long long int mod, long long int data_stride){
			
	P_chasing2(1, A, iterations, B, 2147483648, clock_rate, data_stride);//////////////	
	//P_chasing2(1, A, iterations, B, 0, clock_rate, data_stride);
	//P_chasing2(0, A, iterations, B, mod - data_stride + 3, clock_rate, data_stride);
	
	__syncthreads();
}

int main(int argc, char **argv)
{
    // set device
    hipDeviceProp_t device_prop;
    //int dev_id = findCudaDevice(argc, (const char **) argv);
	int dev_id = 0;
    checkCudaErrors(hipGetDeviceProperties(&device_prop, dev_id));
	
	int peak_clk = 1;//kHz
	checkCudaErrors(hipDeviceGetAttribute(&peak_clk, hipDeviceAttributeClockRate, dev_id));
	float clock_rate = (float) peak_clk;
	
	printf("clock_rate_out_kernel:%f\n", clock_rate);

    if (!device_prop.managedMemory) { 
        // This samples requires being run on a device that supports Unified Memory
        fprintf(stderr, "Unified Memory not supported on this device\n");

        exit(EXIT_WAIVED);
    }

    if (device_prop.computeMode == hipComputeModeProhibited)
    {
        // This sample requires being run with a default or process exclusive mode
        fprintf(stderr, "This sample requires a device in either default or process exclusive mode\n");

        exit(EXIT_WAIVED);
    }
	
	if (device_prop.concurrentManagedAccess == 1){
		printf("This device supports concurrent Managed Access.\n");
    }else{
		printf("This device does not support concurrent Managed Access.\n");
	}
	
	int value1 = 1;
	checkCudaErrors(hipDeviceGetAttribute(&value1, hipDeviceAttributeConcurrentManagedAccess, dev_id));
	printf("hipDeviceAttributeConcurrentManagedAccess = %d\n", value1);	
	
	///////////////////////////////////////////////////////////////////GPU data out
	long long int *GPU_data_out;
	checkCudaErrors(hipMalloc(&GPU_data_out, sizeof(long long int) * 2));			
	
	int counter = 0;	
	//for(long long int data_stride = 1 * 4 * 1024; data_stride <= 1 * 64 * 1024; data_stride = data_stride * 2){
	for(long long int data_stride = 1 * 128 * 1024; data_stride <= 1 * 128 * 1024; data_stride = data_stride * 2){

	//plain managed
	printf("*\n*\n*\n plain managed\n");	
	for(long long int mod = 4294967296; mod <= 4294967296; mod = mod * 2){////134217728 = 1gb, 268435456 = 2gb, 536870912 = 4gb, 1073741824 = 8gb, 2147483648 = 16gb, 4294967296 = 32gb, 8589934592 = 64gb.
		counter++;
		///////////////////////////////////////////////////////////////////CPU data begin
		long long int data_size = mod;
		long long int iterations = mod / data_stride;////32 * 32 * 4 / 32 * 2 = 256
	
		long long int *CPU_data_in;
		//CPU_data_in = (int*)malloc(sizeof(int) * data_size);
		checkCudaErrors(hipMallocManaged(&CPU_data_in, sizeof(long long int) * data_size));/////////////using unified memory		
		init_cpu_data(CPU_data_in, data_size, data_stride, mod);		
		///////////////////////////////////////////////////////////////////CPU data end	
	
		///////////////////////////////////////////////////////////////////GPU data in	
		//int *GPU_data_in;
		//checkCudaErrors(hipMalloc(&GPU_data_in, sizeof(int) * data_size));	
		//hipMemcpy(GPU_data_in, CPU_data_in, sizeof(int) * data_size, hipMemcpyHostToDevice);		
		
		printf("###################data_stride%lld#########################\n", data_stride);
		printf("###############Mod%lld##############%lld\n", mod, iterations);		

		//tlb_latency_test<<<1, 1>>>(CPU_data_in, iterations, GPU_data_out, clock_rate, mod, data_stride);///kernel is here	
		//hipDeviceSynchronize();
		
		/*
		tlb_latency_test2<<<1, 1>>>(CPU_data_in, iterations, GPU_data_out, clock_rate, mod, data_stride);///migrate 32gb to gpu	(with warmup & no eviction & no trail) and (no warmup & with eviction & no trail)
		hipDeviceSynchronize();
		
		traverse_cpu_data(CPU_data_in, iterations/2, 2147483648, data_stride);///////migrate last 16 gb to cpu, gpu is clear
		
		printf("location1:\n");
		
		tlb_latency_test2<<<1, 1>>>(CPU_data_in, iterations, GPU_data_out, clock_rate, mod, data_stride);///migrate 32gb to gpu again (no warmup & no eviction & no trail) and (no warmup & with eviction & no trail)
		hipDeviceSynchronize();
		
		traverse_cpu_data(CPU_data_in, iterations/2, 2147483648, data_stride);///////migrate last 16 gb to cpu, gpu is clear
		
		printf("location2:\n");
		
		tlb_latency_test3<<<1, 1>>>(CPU_data_in, iterations/2, GPU_data_out, clock_rate, mod, data_stride);///migrate last 16gb (starting 17gb) to gpu again (no warmup & no eviction & with trail)
		hipDeviceSynchronize();		
		///////////conclusion: eviction overhead exists, but page migration does not evict the page group setup (trail does exist, leave a trail when page size not dynamic).
		*/
		
		//page eviction evict the whole 2M group? 1m vs 2m strides.
		tlb_latency_test5<<<1, 1>>>(CPU_data_in, iterations, GPU_data_out, clock_rate, mod, data_stride);///migrate the last 16gb
		hipDeviceSynchronize();
		
		tlb_latency_test4<<<1, 1>>>(CPU_data_in, iterations/2, GPU_data_out, clock_rate, mod, data_stride);///migrate first 16gb to gpu, offset and without covering all the previous last 16gb steps however.
		hipDeviceSynchronize();
		
		tlb_latency_test3<<<1, 1>>>(CPU_data_in, iterations, GPU_data_out, clock_rate, mod, data_stride);///migrate the last 16gb again (starting 17gb), any page hit?
		hipDeviceSynchronize();
		///////////////////conclusion: page eviction evict the whole 2M group.
		
		/*
		///////////is it migrating 64k always when not dynamic? use different stride to find out. 64 vs 128?
		tlb_latency_test5<<<1, 1>>>(CPU_data_in, iterations, GPU_data_out, clock_rate, mod, data_stride);///migrate the last 16gb
		hipDeviceSynchronize();
		
		tlb_latency_test4<<<1, 1>>>(CPU_data_in, iterations/2, GPU_data_out, clock_rate, mod, data_stride);///migrate first 16gb to gpu, offset and without covering all the previous last 16gb steps however.
		hipDeviceSynchronize();
		
		tlb_latency_test3<<<1, 1>>>(CPU_data_in, iterations, GPU_data_out, clock_rate, mod, data_stride);///migrate the last 16gb again (starting 17gb), any page hit?
		hipDeviceSynchronize();
		///////////////////conclusion: page eviction evict the whole 2M group.
		*/
				
		//checkCudaErrors(hipFree(GPU_data_in));
		checkCudaErrors(hipFree(CPU_data_in));
		//free(CPU_data_in);		
	}
	}
			
	checkCudaErrors(hipFree(GPU_data_out));	
	
    exit(EXIT_SUCCESS);
}