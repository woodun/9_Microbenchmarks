#include "hip/hip_runtime.h"
#include <cstdio>
#include <ctime>
#include <vector>
#include <algorithm>
#include <stdlib.h>
// utilities
#include <hip/hip_runtime_api.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_cooperative_groups.h>

using namespace cooperative_groups;

/////////////////////////////L1 is enabled. "ALL_CCFLAGS += -Xptxas -dlcm=ca"
//////////////large vs small data.

void init_cpu_data(long long int* A, long long int size, double stride){
	
	for (long long int i = 0; i < size; i++){
		A[i]=1;
	}
	
	/*
	for (long long int i = 0; i < size - stride; i++){
		A[i]=(i + stride);
	}
			
	for (long long int i = size - stride; i < size; i++){
		A[i]=0;
	}
	*/
}

__global__ void gpu_initialization(long long int *A, double data_stride, long long int data_size){			

	long long int index = (blockIdx.x * blockDim.x + threadIdx.x);
	long long int thread_num =  gridDim.x * blockDim.x;
	
	for(long long int it = 0; it < data_size; it = it + thread_num){
		A[index + it]=23;
	}
}

long long unsigned time_diff(timespec start, timespec end){
	struct timespec temp;
	if ((end.tv_nsec - start.tv_nsec) < 0){
		temp.tv_sec = end.tv_sec - start.tv_sec - 1;
		temp.tv_nsec = 1000000000 + end.tv_nsec - start.tv_nsec;
	} 
	else{
		temp.tv_sec = end.tv_sec - start.tv_sec;
		temp.tv_nsec = end.tv_nsec - start.tv_nsec;
	}
	
	long long unsigned time_interval_ns = temp.tv_nsec;
	long long unsigned time_interval_s = temp.tv_sec;
	time_interval_s = time_interval_s * 1000000000;
	
	return time_interval_s + time_interval_ns;
}

__global__ void baseline(long long int *A1, long long int *A2, long long int *B, double data_stride, long long int clock_count){////load-compute-store
	
	double temp = (blockIdx.x * blockDim.x + threadIdx.x) * data_stride;
	long long int index = __double2ll_rd(temp);
	
	long long int value1 = A1[index];
		
	//////////////////////////////////////////////loop
	long long int clock_offset = 0;
    while (clock_offset < clock_count){/////////////////what's the time overhead for addition and multiplication?
        clock_offset++;
		value1 = value1 + threadIdx.x;
    }
	
	long long int value2 = A2[index];
	
	//////////////////////////////////////////////loop
	long long int clock_offset2 = 0;
    while (clock_offset2 < clock_count){/////////////////what's the time overhead for addition and multiplication?
        clock_offset2++;
		value2 = value2 + threadIdx.x;
    }
	
	B[index] = value1 + value2;
}

__global__ void page_visitor(long long int *A1, long long int *A2, long long int *B, double data_stride, long long int clock_count){////load-compute-store
			
	//thread_block block = this_thread_block();	
	
	double temp = (blockIdx.x * 512 + threadIdx.x) * data_stride;
	long long int index = __double2ll_rd(temp);
	
	long long int value1;
	
	double temp2 = (blockIdx.x * 512 + threadIdx.x * 16) * data_stride;
	long long int prefetch_index = __double2ll_rd(temp2);
	long long int value2;
	
	//if(threadIdx.x < 480){
	if(threadIdx.x > 31){
	//if(0){
		value1 = A1[index];
		
	}else{
		value1 = A1[index];
		value2 = A2[index];
	}
	
	//block.sync();
		
	long long int clock_offset = 0;
    while (clock_offset < clock_count){/////////////////what's the time overhead for addition and multiplication?
        clock_offset++;
		value1 = value1 + threadIdx.x;
    }
	
	//if(threadIdx.x < 480){		
	if(threadIdx.x > 31){
	//if(0){/////////////////////////question: find out which part is causing the benefit.
		value2 = A2[index];
	}else{
		//value2 = A2[index];
		B[prefetch_index] = 0;
	}	
	
	//block.sync();
	
	long long int clock_offset2 = 0;
    while (clock_offset2 < clock_count){/////////////////what's the time overhead for addition and multiplication?
        clock_offset2++;
		value2 = value2 + threadIdx.x;
    }

	B[index] = value1 + value2;	
}

__global__ void page_visitor3(long long int *A1, long long int *A2, long long int *B1, double data_stride, long long int clock_count, long long int offset){////vertical + horizontal

	double temp = (blockIdx.x * 512 + threadIdx.x) * data_stride;
	long long int index = __double2ll_rd(temp);
	
	long long int value1;
	
	double temp2 = ( (blockIdx.x + offset) * 512 + threadIdx.x * 16) * data_stride;
	long long int prefetch_index = __double2ll_rd(temp2);
	long long int value2;
	
	//if(threadIdx.x < 480){
	if(threadIdx.x > 31){
	//if(0){
		value1 = A1[index];
		
	}else{
		value1 = A1[index];
		if(blockIdx.x < 4194304 - offset){
		value2 = A2[index];
		}
	}
			
	long long int clock_offset = 0;
    while (clock_offset < clock_count){/////////////////what's the time overhead for addition and multiplication?
        clock_offset++;
		value1 = value1 + threadIdx.x;
    }
	
	//if(threadIdx.x < 480){		
	if(threadIdx.x > 31){
	//if(0){/////////////////////////question: find out which part is causing the benefit.
		value2 = A2[index];
	}else{
		//value2 = A2[index];
		if(blockIdx.x < 4194304 - offset){
		B1[prefetch_index] = 0;
		}
	}	
	
	long long int clock_offset2 = 0;
    while (clock_offset2 < clock_count){/////////////////what's the time overhead for addition and multiplication?
        clock_offset2++;
		value2 = value2 + threadIdx.x;
    }

	B1[index] = value1 + value2;	
}

int main(int argc, char **argv)
{
	printf("\n");
	
    // set device
    hipDeviceProp_t device_prop;
    //long long int dev_id = findCudaDevice(argc, (const char **) argv);
	long long int dev_id = 0;
    checkCudaErrors(hipGetDeviceProperties(&device_prop, dev_id));
	
	//int peak_clk = 1;//kHz
	//checkCudaErrors(hipDeviceGetAttribute(&peak_clk, hipDeviceAttributeClockRate, dev_id));
	//float clock_rate = (float) peak_clk;
	//printf("clock_rate:%f\n", clock_rate);

    if (!device_prop.managedMemory) { 
        // This samples requires being run on a device that supports Unified Memory
        fprintf(stderr, "Unified Memory not supported on this device\n");

        exit(EXIT_WAIVED);
    }

    if (device_prop.computeMode == hipComputeModeProhibited)
    {
        // This sample requires being run with a default or process exclusive mode
        fprintf(stderr, "This sample requires a device in either default or process exclusive mode\n");

        exit(EXIT_WAIVED);
    }
	
	/*
	if (device_prop.concurrentManagedAccess == 1){
		printf("This device supports concurrent Managed Access.\n");
    }else{
		printf("This device does not support concurrent Managed Access.\n");
	}
	*/
	
	int value1 = 1;
	checkCudaErrors(hipDeviceGetAttribute(&value1, hipDeviceAttributeConcurrentManagedAccess, dev_id));
	//printf("hipDeviceAttributeConcurrentManagedAccess = %d\n", value1);	
	
	//changeable: block size, number of blocks(8192 * 512 max?), data size, data stride, computation length (best not too long nor too short?), ways of implementation (stand alone or incorporated? verticle or parallel prefetch?), span of prefetching (fetch for other blocks even verticlly)? additional warp with vote, different vote and prefetch locations.
	//plain managed
	//when was 64k and 4k pages used?
	//how to decrease the overhead of sync?
	//printf("###################\n#########################managed\n");
	
	/*
	for(long long int factor = 1; factor <= 128; factor = factor * 2){
	for(double data_stride = 0.25 * factor; data_stride <= 1 * 1 * 4 * factor; data_stride = data_stride * 2){///134217728 = 1gb, 268435456 = 2gb, 536870912 = 4gb, 1073741824 = 8gb, 2147483648 = 16gb, 4294967296 = 32gb, 8589934592 = 64gb. (index)
	for(long long int clock_count = 64; clock_count <= 16384; clock_count = clock_count * 2){
	*/

	printf("############approach\n");
	for(long long int offset = 64; offset <= 64; offset = offset * 2){	
	for(long long int factor = 1; factor <= 128; factor = factor * 2){
	for(double data_stride = 0.25 * factor; data_stride <= 1 * 1 * 4 * factor; data_stride = data_stride * 2){///134217728 = 1gb, 268435456 = 2gb, 536870912 = 4gb, 1073741824 = 8gb, 2147483648 = 16gb, 4294967296 = 32gb, 8589934592 = 64gb. (index)
	for(long long int clock_count = 32; clock_count <= 32; clock_count = clock_count * 2){

		///////////////////////////////////////////////////////////////////CPU data begin
		double temp = data_stride * 512;
		long long int data_size = (long long int) temp;
		data_size = data_size * 8192 * 512 / factor;
		
		long long int *CPU_data_in1;
		checkCudaErrors(hipMallocManaged(&CPU_data_in1, sizeof(long long int) * data_size));/////////////using unified memory
		long long int *CPU_data_in2;
		checkCudaErrors(hipMallocManaged(&CPU_data_in2, sizeof(long long int) * data_size));/////////////using unified memory
		///////////////////////////////////////////////////////////////////CPU data end
		
		long long int *GPU_data_out1;
		checkCudaErrors(hipMallocManaged(&GPU_data_out1, sizeof(long long int) * data_size));/////////////using unified memory
		///////////////////////////////////////////////////////////////////GPU data out	end
		
		if(1){
			double scale = 1;
			if(data_stride < 1){
				scale = data_stride;/////////make sure threadIdx is smaller than data_size in the initialization
			}
			
			gpu_initialization<<<8192 * 512 * scale / factor, 512>>>(GPU_data_out1, data_stride, data_size);////////////1024 per block max
			hipDeviceSynchronize();
			gpu_initialization<<<8192 * 512 * scale / factor, 512>>>(CPU_data_in1, data_stride, data_size);//////////1024 per block max
			hipDeviceSynchronize();
			gpu_initialization<<<8192 * 512 * scale / factor, 512>>>(CPU_data_in2, data_stride, data_size);//////////1024 per block max
			hipDeviceSynchronize();
		}else{
			init_cpu_data(GPU_data_out1, data_size, data_stride);
			init_cpu_data(CPU_data_in1, data_size, data_stride);
			init_cpu_data(CPU_data_in2, data_size, data_stride);		
		}
		
		/////////////////////////////////time
		struct timespec ts1;
		clock_gettime(CLOCK_REALTIME, &ts1);

		////may want to use more thread to see clock_count effect
		page_visitor3<<<8192 * 512 / factor, 512>>>(CPU_data_in1, CPU_data_in2, GPU_data_out1, data_stride, clock_count, offset);
		///////////////////////////////////////////////////32 * 64 * 1 * 512 * 1024 = 8gb.
		hipDeviceSynchronize();
				
		/////////////////////////////////time
		struct timespec ts2;
		clock_gettime(CLOCK_REALTIME, &ts2);
		
		//printf("###################data_stride%lld#########################clock_count:%lld\n", data_stride, clock_count);
		//printf("*\n*\n*\nruntime:  %lluns\n", time_diff(ts1, ts2));
		printf("%llu ", time_diff(ts1, ts2));
		fflush(stdout);
		
		checkCudaErrors(hipFree(CPU_data_in1));
		checkCudaErrors(hipFree(CPU_data_in2));		
		checkCudaErrors(hipFree(GPU_data_out1));
	}
	printf("\n");
	}
	printf("####################%llu\n", factor);
	}
	printf("####################%llu\n", offset);
	}
	
	printf("############baseline\n");
	for(long long int factor = 1; factor <= 128; factor = factor * 2){
	for(double data_stride = 0.25 * factor; data_stride <= 1 * 1 * 4 * factor; data_stride = data_stride * 2){///134217728 = 1gb, 268435456 = 2gb, 536870912 = 4gb, 1073741824 = 8gb, 2147483648 = 16gb, 4294967296 = 32gb, 8589934592 = 64gb. (index)
	for(long long int clock_count = 32; clock_count <= 32; clock_count = clock_count * 2){

		///////////////////////////////////////////////////////////////////CPU data begin
		double temp = data_stride * 512;
		long long int data_size = (long long int) temp;
		data_size = data_size * 8192 * 512 / factor;
		
		long long int *CPU_data_in1;
		checkCudaErrors(hipMallocManaged(&CPU_data_in1, sizeof(long long int) * data_size));/////////////using unified memory
		long long int *CPU_data_in2;
		checkCudaErrors(hipMallocManaged(&CPU_data_in2, sizeof(long long int) * data_size));/////////////using unified memory
		///////////////////////////////////////////////////////////////////CPU data end
		
		long long int *GPU_data_out1;
		checkCudaErrors(hipMallocManaged(&GPU_data_out1, sizeof(long long int) * data_size));/////////////using unified memory
		///////////////////////////////////////////////////////////////////GPU data out	end
		
		if(1){
			double scale = 1;
			if(data_stride < 1){
				scale = data_stride;/////////make sure threadIdx is smaller than data_size in the initialization
			}
			
			gpu_initialization<<<8192 * 512 * scale / factor, 512>>>(GPU_data_out1, data_stride, data_size);////////////1024 per block max
			hipDeviceSynchronize();
			gpu_initialization<<<8192 * 512 * scale / factor, 512>>>(CPU_data_in1, data_stride, data_size);//////////1024 per block max
			hipDeviceSynchronize();
			gpu_initialization<<<8192 * 512 * scale / factor, 512>>>(CPU_data_in2, data_stride, data_size);//////////1024 per block max
			hipDeviceSynchronize();
		}else{
			init_cpu_data(GPU_data_out1, data_size, data_stride);
			init_cpu_data(CPU_data_in1, data_size, data_stride);
			init_cpu_data(CPU_data_in2, data_size, data_stride);			
		}
		
		/////////////////////////////////time
		struct timespec ts1;
		clock_gettime(CLOCK_REALTIME, &ts1);

		////may want to use more thread to see clock_count effect
		baseline<<<8192 * 512 / factor, 512>>>(CPU_data_in1, CPU_data_in2, GPU_data_out1, data_stride, clock_count);
		///////////////////////////////////////////////////32 * 64 * 1 * 512 * 1024 = 8gb.
		hipDeviceSynchronize();
				
		/////////////////////////////////time
		struct timespec ts2;
		clock_gettime(CLOCK_REALTIME, &ts2);
		
		//printf("###################data_stride%lld#########################clock_count:%lld\n", data_stride, clock_count);
		//printf("*\n*\n*\nruntime:  %lluns\n", time_diff(ts1, ts2));
		printf("%llu ", time_diff(ts1, ts2));
		fflush(stdout);
		
		checkCudaErrors(hipFree(CPU_data_in1));
		checkCudaErrors(hipFree(CPU_data_in2));		
		checkCudaErrors(hipFree(GPU_data_out1));
	}
	printf("\n");
	}
	printf("####################%llu\n", factor);
	}

	exit(EXIT_SUCCESS);
}