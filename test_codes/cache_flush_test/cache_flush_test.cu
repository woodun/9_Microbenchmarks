#include "hip/hip_runtime.h"
#include <cstdio>
#include <ctime>
#include <vector>
#include <algorithm>
#include <stdlib.h>
// utilities
#include <hip/hip_runtime_api.h>
#include <time.h>

/////////////////////////////check if L1 and L2 flush between kernels.


void init_cpu_data(int* A, int size, int stride){
	for (int i = 0; i < size; ++i){
		A[i]=(i + stride) % size;
   	}
}

/*
__device__ void cache_warmup(int *A, int iterations, int *B){
	
	int j = 0;
	
	for (int it =0; it < iterations; it ++){
		j = A[j];
	}
	
	B[0] = j;
}
*/

//////////min page size 4kb = 4096b = 32 * 128.
__device__ void P_chasing(int mark, int *A, int iterations, int *B, int starting_index, float clock_rate){
	
	int j = starting_index;/////make them in the same page, and miss near in cache lines
	
	long long int start_time = 0;//////clock
	long long int end_time = 0;//////clock
	start_time = clock64();//////clock
	
	for (int it =0; it < iterations; it ++){
		j = A[j];
	}
	
	end_time=clock64();//////clock
	long long int total_time = end_time - start_time;//////clock
	printf("inside%d:%fms\n", mark, (total_time / (float)clock_rate) / (float)iterations);//////clock, average latency
	
	B[0] = j;
}

//////////////////////////////////////////////////////4 * (8) * 32 * 32 = 128kb ///////////////////48 * 128kb = 6144kb ///////////12 * 128kb = 1536kb ////////////// 16 * 64 = 1024 = 4kb
__global__ void tlb_latency_test(int *A, int iterations, int *B, float clock_rate){	

	int index = 0;
	
	long long int start_time = 0;///////////clock
	long long int end_time = 0;///////////clock	
	start_time = clock64();///////////clock
		
	P_chasing(-2, A, 7, B, 2 * 524288, clock_rate);/////warmup GPU
	
	P_chasing(-1, A, 1, B, 0 * 524288 + 0 * 16, clock_rate);/////warmup TLB
	P_chasing(0, A, 16, B, 0 * 524288 + 1 * 16, clock_rate);/////try to generate TLB hit and cache miss ///////(1)
	P_chasing(0, A, 16, B, 0 * 524288 + 1 * 16, clock_rate);/////try to generate TLB hit and cache hit ///////(1)
	P_chasing(0, A, 16, B, 0 * 524288 + 0 * 16, clock_rate);/////try to generate TLB hit and cache hit ///////(1)
	
	P_chasing(3, A, 16, B, 3 * 524288, clock_rate);/////try to generate TLB miss and cache miss
	P_chasing(3, A, 16, B, 3 * 524288 + 32 * 16, clock_rate);/////try to generate TLB hit and cache miss
	P_chasing(3, A, 16, B, 3 * 524288, clock_rate);/////try to generate TLB hit and cache hit
	P_chasing(3, A, 16, B, 3 * 524288, clock_rate);/////try to generate TLB hit and cache hit
	
	end_time=clock64();///////////clock
		
	long long int total_time = end_time - start_time;///////////clock
	//printf("outside1:%fms\n", total_time / (float)clock_rate);///////////clock
	printf("\n");
}

//////////////////////////////////////////////////////4 * (8) * 32 * 32 = 128kb ///////////////////48 * 128kb = 6144kb ///////////12 * 128kb = 1536kb
__global__ void tlb_latency_test_2(int *A, int iterations, int *B, float clock_rate){	

	int index = 0;
	
	long long int start_time = 0;///////////clock
	long long int end_time = 0;///////////clock	
	start_time = clock64();///////////clock
		
	//P_chasing(0, A, 1, B, 31 * 32, clock_rate);/////warmup TLB
	P_chasing(1, A, 16, B, 1 * 524288, clock_rate);/////warmup GPU
	P_chasing(0, A, 16, B, 0 * 524288 + 1 * 16, clock_rate);/////try to generate TLB hit and cache hit ///////(1)
	P_chasing(0, A, 16, B, 0 * 524288 + 1 * 16, clock_rate);/////try to generate TLB hit and cache hit ///////(1)
	P_chasing(0, A, 16, B, 0 * 524288 + 0 * 16, clock_rate);/////try to generate TLB hit and cache hit ///////(1)
	P_chasing(14, A, 16, B, 14 * 524288, clock_rate);/////try to generate TLB miss and cache miss
	P_chasing(15, A, 16, B, 15 * 524288 + 16 * 16, clock_rate);/////try to generate TLB hit and cache miss	
	
	end_time=clock64();///////////clock
		
	long long int total_time = end_time - start_time;///////////clock
	//printf("outside1:%fms\n", total_time / (float)clock_rate);///////////clock
	printf("\n");
}

//////////////////////////////////////////////////////4 * (8) * 32 * 32 = 128kb ///////////////////48 * 128kb = 6144kb ///////////12 * 128kb = 1536kb
__global__ void tlb_latency_test_3(int *A, int iterations, int *B, float clock_rate){	

	int index = 0;
	
	long long int start_time = 0;///////////clock
	long long int end_time = 0;///////////clock	
	start_time = clock64();///////////clock
		
	//P_chasing(0, A, 1, B, 31 * 32, clock_rate);/////warmup TLB
	P_chasing(17, A, 16, B, 17 * 524288, clock_rate);/////warmup GPU
	P_chasing(1, A, 16, B, 1 * 524288, clock_rate);/////try to generate TLB miss and cache miss
	P_chasing(0, A, 16, B, 0 * 524288, clock_rate);/////try to generate TLB hit and cache hit
	P_chasing(1, A, 16, B, 1 * 524288 + 16 * 16, clock_rate);/////try to generate TLB hit and cache miss	
	
	end_time=clock64();///////////clock
		
	long long int total_time = end_time - start_time;///////////clock
	//printf("outside1:%fms\n", total_time / (float)clock_rate);///////////clock
	printf("\n");
}

//////////////////////////////////////////////////////4 * (8) * 32 * 32 = 128kb ///////////////////48 * 128kb = 6144kb ///////////12 * 128kb = 1536kb
__global__ void tlb_latency_test_4(int *A, int iterations, int *B, float clock_rate){	

	//P_chasing(0, A, 1, B, 31 * 32, clock_rate);/////warmup TLB
	P_chasing(1, A, 16, B, 1 * 524288, clock_rate);/////warmup
	P_chasing(0, A, 16, B, 0 * 524288, clock_rate);/////try to generate TLB hit and cache hit
	P_chasing(17, A, 16, B, 17 * 524288 + 16 * 16, clock_rate);/////try to generate TLB hit and cache miss
	P_chasing(18, A, 16, B, 18 * 524288, clock_rate);/////try to generate TLB miss and cache miss
	printf("\n");
}

int main(int argc, char **argv)
{
	printf("\n");
	
    // set device
    hipDeviceProp_t device_prop;
    //int dev_id = findCudaDevice(argc, (const char **) argv);
	int dev_id = 1;
    checkCudaErrors(hipGetDeviceProperties(&device_prop, dev_id));
	
	int peak_clk = 1;//kHz
	checkCudaErrors(hipDeviceGetAttribute(&peak_clk, hipDeviceAttributeClockRate, dev_id));
	float clock_rate = (float) peak_clk;
	
	//printf("clock_rate_out_kernel:%f\n", clock_rate);

    if (!device_prop.managedMemory) { 
        // This samples requires being run on a device that supports Unified Memory
        fprintf(stderr, "Unified Memory not supported on this device\n");

        exit(EXIT_WAIVED);
    }

    if (device_prop.computeMode == hipComputeModeProhibited)
    {
        // This sample requires being run with a default or process exclusive mode
        fprintf(stderr, "This sample requires a device in either default or process exclusive mode\n");

        exit(EXIT_WAIVED);
    }
		
	///////////////////////////////////////////////////////////////////CPU data begin
	int iterations = 2 * 16384 * 100;
	////////size(int) = 4, 32 = 128b, 256 = 1kb, 16 * 64 = 32 * 32 = 1024 = 4kb, 262144 = 1mb, 16384 * 32 = 512 * 1024 = 524288 = 2mb.
	int data_stride = 16;/////64b.
	//int data_size = 524288000;/////1000 * 2mb. ##### size = iteration * stride. ##### This can support 1000 iteration. The 1001st iteration starts from head again.
	int data_size = iterations * data_stride;/////size = iteration * stride = 100 2mb pages.
	
	int *CPU_data_in;	
	CPU_data_in = (int*)malloc(sizeof(int) * data_size);
	//int *CPU_data_out;
	//CPU_data_out = (int*)malloc(data_size * sizeof(int));
	
	init_cpu_data(CPU_data_in, data_size, data_stride);
	///////////////////////////////////////////////////////////////////CPU data end
	
	///////////////////////////////////////////////////////////////////GPU data begin
	int *GPU_data_in;
	//////checkCudaErrors(hipMallocManaged(&data, sizeof(int) * data_size));
	checkCudaErrors(hipMalloc(&GPU_data_in, sizeof(int) * data_size));	
	checkCudaErrors(hipMemcpy(GPU_data_in, CPU_data_in, sizeof(int) * data_size, hipMemcpyHostToDevice));
	
	int *GPU_data_out;
	checkCudaErrors(hipMalloc(&GPU_data_out, sizeof(int) * 1));	
	///////////////////////////////////////////////////////////////////GPU data end				  
		
	tlb_latency_test<<<1, 1>>>(GPU_data_in, iterations, GPU_data_out, clock_rate);//////////////////////////////////////////////kernel is here
	
	//hipMemcpy(CPU_data_out, GPU_data_out, sizeof(int) * data_size, hipMemcpyDeviceToHost);	
    hipDeviceSynchronize();
	
	tlb_latency_test_2<<<1, 1>>>(GPU_data_in, iterations, GPU_data_out, clock_rate);//////////////////////////////////////////////kernel is here
	hipDeviceSynchronize();
	tlb_latency_test_2<<<1, 1>>>(GPU_data_in, iterations, GPU_data_out, clock_rate);//////////////////////////////////////////////kernel is here
	hipDeviceSynchronize();
	tlb_latency_test_2<<<1, 1>>>(GPU_data_in, iterations, GPU_data_out, clock_rate);//////////////////////////////////////////////kernel is here
	hipDeviceSynchronize();
	tlb_latency_test_3<<<1, 1>>>(GPU_data_in, iterations, GPU_data_out, clock_rate);//////////////////////////////////////////////kernel is here
	hipDeviceSynchronize();
	tlb_latency_test_3<<<1, 1>>>(GPU_data_in, iterations, GPU_data_out, clock_rate);//////////////////////////////////////////////kernel is here
	hipDeviceSynchronize();
	tlb_latency_test_3<<<1, 1>>>(GPU_data_in, iterations, GPU_data_out, clock_rate);//////////////////////////////////////////////kernel is here
	hipDeviceSynchronize();	
		
	
	tlb_latency_test_4<<<1, 1>>>(GPU_data_in, iterations, GPU_data_out, clock_rate);//////////////////////////////////////////////kernel is here
	hipDeviceSynchronize();
	/////////////check between copies
	int *GPU_data_in_2;	
	checkCudaErrors(hipMalloc(&GPU_data_in_2, sizeof(int) * data_size));
	checkCudaErrors(hipMemcpy(GPU_data_in_2, CPU_data_in, sizeof(int) * data_size, hipMemcpyHostToDevice));
	
	tlb_latency_test_4<<<1, 1>>>(GPU_data_in, iterations, GPU_data_out, clock_rate);//////////////////////////////////////////////kernel is here
	hipDeviceSynchronize();
	
	tlb_latency_test_4<<<1, 1>>>(GPU_data_in_2, iterations, GPU_data_out, clock_rate);//////////////////////////////////////////////kernel is here
	hipDeviceSynchronize();
	
	checkCudaErrors(hipFree(GPU_data_in_2));
	/////////////check between copies
	
	checkCudaErrors(hipFree(GPU_data_in));
	checkCudaErrors(hipFree(GPU_data_out));
	free(CPU_data_in);
	//free(CPU_data_out);
		
    exit(EXIT_SUCCESS);
}
