#include "hip/hip_runtime.h"
#include <cstdio>
#include <ctime>
#include <vector>
#include <algorithm>
#include <stdlib.h>
// utilities
#include <hip/hip_runtime_api.h>
#include <time.h>

/////////////////access different range of data to check the cacheline size. L1 is likely saturated in this example. L1 can also be disabled to check if it is from L1 or L2.

void init_cpu_data(int* A, int size, int stride){
	for (int i = 0; i < size; ++i){
		A[i]=(i + stride) % size;
   	}
}

/*
__device__ void cache_warmup(int *A, int iterations, int *B){
	
	int j = 0;
	
	for (int it =0; it < iterations; it ++){
		j = A[j];
	}
	
	B[0] = j;
}
*/

//////////min page size 4kb = 4096b = 32 * 128.
__device__ void P_chasing(int mark, int *A, int iterations, int *B, int starting_index, float clock_rate){//////////////should not hit in the tlb, and should also miss in the cache, to see the time difference.
	
	int j = starting_index;/////make them in the same page, and miss near in cache lines
	
	long long int start_time = 0;//////clock
	long long int end_time = 0;//////clock
	start_time = clock64();//////clock
	
	for (int it =0; it < iterations; it ++){
		j = A[j];
	}
	
	end_time=clock64();//////clock
	long long int total_time = end_time - start_time;//////clock
	printf("inside%d:%fms\n", mark, ((total_time / (float)clock_rate))/ (float) iterations);//////clock
	
	B[0] = j;
}

__global__ void tlb_latency_test(int *A, int iterations, int *B, float clock_rate, int iter){	
	
	printf("iter:%d\n", iter);
	
	long long int start_time = 0;///////////clock
	long long int end_time = 0;///////////clock	
	start_time = clock64();///////////clock
	
	P_chasing(0, A, 7, B, 192 * 524288, clock_rate);/////GPU warmup
	P_chasing(0, A, iter, B, 0 * 32, clock_rate);/////TLB warmup
	P_chasing(1, A, iter, B, 0 * 32 + 1, clock_rate);/////make them in the same page, and hit near in cache lines
	P_chasing(4, A, iter, B, 0 * 32 + 4, clock_rate);/////make them in the same page, and hit near in cache lines	
	P_chasing(7, A, iter, B, 0 * 32 + 7, clock_rate);/////////////make them in the same page, and hit far in cache lines
	P_chasing(8, A, iter, B, 0 * 32 + 8, clock_rate);/////////////make them in the same page, and hit far in cache lines
	P_chasing(9, A, iter, B, 0 * 32 + 9, clock_rate);/////////////make them in the same page, and hit far in cache lines
	P_chasing(15, A, iter, B, 0 * 32 + 15, clock_rate);////////////make them in the same page, and hit far in cache lines
	P_chasing(16, A, iter, B, 0 * 32 + 16, clock_rate);////////////make them in the same page, and hit far in cache lines
	P_chasing(24, A, iter, B, 0 * 32 + 24, clock_rate);////////////make them in the same page, and hit far in cache lines
	P_chasing(1, A, iter, B, 1 * 32, clock_rate);/////make them in the same page, and miss near in cache lines
	P_chasing(4, A, iter, B, 4 * 32, clock_rate);/////make them in the same page, and miss near in cache lines
	P_chasing(8, A, iter, B, 8 * 32, clock_rate);//////////////make them in the same page, and miss far in cache lines
	P_chasing(16, A, iter, B, 16 * 32, clock_rate);/////////////make them in the same page, and miss far in cache lines
	P_chasing(24, A, iter, B, 24 * 32, clock_rate);/////////////make them in the same page, and miss far in cache lines
	
	end_time=clock64();///////////clock
		
	long long int total_time = end_time - start_time;///////////clock
	printf("outside1:%fms\n", total_time / (float)clock_rate);///////////clock
}

int main(int argc, char **argv)
{
	printf("\n");
	
    // set device
    hipDeviceProp_t device_prop;
    //int dev_id = findCudaDevice(argc, (const char **) argv);
	int dev_id = 0;
    checkCudaErrors(hipGetDeviceProperties(&device_prop, dev_id));
	
	int peak_clk = 1;//kHz
	checkCudaErrors(hipDeviceGetAttribute(&peak_clk, hipDeviceAttributeClockRate, dev_id));
	float clock_rate = (float) peak_clk;
	
	//printf("clock_rate_out_kernel:%f\n", clock_rate);

    if (!device_prop.managedMemory) { 
        // This samples requires being run on a device that supports Unified Memory
        fprintf(stderr, "Unified Memory not supported on this device\n");

        exit(EXIT_WAIVED);
    }

    if (device_prop.computeMode == hipComputeModeProhibited)
    {
        // This sample requires being run with a default or process exclusive mode
        fprintf(stderr, "This sample requires a device in either default or process exclusive mode\n");

        exit(EXIT_WAIVED);
    }
		
	///////////////////////////////////////////////////////////////////CPU data begin
	////////size(int) = 4, 256 = 1kb, 262144 = 1mb, 524288 = 2mb.
	int iterations = 200;
	int data_stride = 524288;/////2mb. Pointing to the next page.
	//int data_size = 524288000;/////1000 * 2mb. ##### size = iteration * stride. ##### This can support 1000 iteration. The 1001st iteration starts from head again.
	int data_size = iterations * data_stride;/////size = iteration * stride = 200 pages.
	
	int *CPU_data_in;	
	CPU_data_in = (int*)malloc(sizeof(int) * data_size);	
	init_cpu_data(CPU_data_in, data_size, data_stride);
	///////////////////////////////////////////////////////////////////CPU data end
	
	///////////////////////////////////////////////////////////////////GPU data out
	int *GPU_data_out;
	checkCudaErrors(hipMalloc(&GPU_data_out, sizeof(int) * 1));
	
	////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////iter128
	///////////////////////////////////////////////////////////////////GPU data in
	int *GPU_data_in128;	
	checkCudaErrors(hipMalloc(&GPU_data_in128, sizeof(int) * data_size));
	hipMemcpy(GPU_data_in128, CPU_data_in, sizeof(int) * data_size, hipMemcpyHostToDevice);
			
	tlb_latency_test<<<1, 1>>>(GPU_data_in128, iterations, GPU_data_out, clock_rate, 128);//////////////////////////////////////////////kernel is here			
    hipDeviceSynchronize();		
	
	checkCudaErrors(hipFree(GPU_data_in128));
	////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////iter128
	
	////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////iter64
	///////////////////////////////////////////////////////////////////GPU data in
	int *GPU_data_in64;	
	checkCudaErrors(hipMalloc(&GPU_data_in64, sizeof(int) * data_size));
	hipMemcpy(GPU_data_in64, CPU_data_in, sizeof(int) * data_size, hipMemcpyHostToDevice);
			
	tlb_latency_test<<<1, 1>>>(GPU_data_in64, iterations, GPU_data_out, clock_rate, 64);//////////////////////////////////////////////kernel is here			
    hipDeviceSynchronize();		
	
	checkCudaErrors(hipFree(GPU_data_in64));
	////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////iter64
	
	////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////iter32
	///////////////////////////////////////////////////////////////////GPU data in
	int *GPU_data_in32;	
	checkCudaErrors(hipMalloc(&GPU_data_in32, sizeof(int) * data_size));
	hipMemcpy(GPU_data_in32, CPU_data_in, sizeof(int) * data_size, hipMemcpyHostToDevice);
			
	tlb_latency_test<<<1, 1>>>(GPU_data_in32, iterations, GPU_data_out, clock_rate, 32);//////////////////////////////////////////////kernel is here			
    hipDeviceSynchronize();		
	
	checkCudaErrors(hipFree(GPU_data_in32));
	////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////iter32
	
	////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////iter16
	///////////////////////////////////////////////////////////////////GPU data in
	int *GPU_data_in16;	
	checkCudaErrors(hipMalloc(&GPU_data_in16, sizeof(int) * data_size));
	hipMemcpy(GPU_data_in16, CPU_data_in, sizeof(int) * data_size, hipMemcpyHostToDevice);
			
	tlb_latency_test<<<1, 1>>>(GPU_data_in16, iterations, GPU_data_out, clock_rate, 16);//////////////////////////////////////////////kernel is here			
    hipDeviceSynchronize();		
	
	checkCudaErrors(hipFree(GPU_data_in16));
	////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////iter16
	
		
	////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////iter8
	///////////////////////////////////////////////////////////////////GPU data in
	int *GPU_data_in8;	
	checkCudaErrors(hipMalloc(&GPU_data_in8, sizeof(int) * data_size));
	hipMemcpy(GPU_data_in8, CPU_data_in, sizeof(int) * data_size, hipMemcpyHostToDevice);
			
	tlb_latency_test<<<1, 1>>>(GPU_data_in8, iterations, GPU_data_out, clock_rate, 8);//////////////////////////////////////////////kernel is here			
    hipDeviceSynchronize();		
	
	checkCudaErrors(hipFree(GPU_data_in8));
	////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////iter8
	
		
	////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////iter4
	///////////////////////////////////////////////////////////////////GPU data in
	int *GPU_data_in4;	
	checkCudaErrors(hipMalloc(&GPU_data_in4, sizeof(int) * data_size));
	hipMemcpy(GPU_data_in4, CPU_data_in, sizeof(int) * data_size, hipMemcpyHostToDevice);
			
	tlb_latency_test<<<1, 1>>>(GPU_data_in4, iterations, GPU_data_out, clock_rate, 4);//////////////////////////////////////////////kernel is here			
    hipDeviceSynchronize();		
	
	checkCudaErrors(hipFree(GPU_data_in4));
	////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////iter4
	
	////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////iter2
	///////////////////////////////////////////////////////////////////GPU data in
	int *GPU_data_in2;	
	checkCudaErrors(hipMalloc(&GPU_data_in2, sizeof(int) * data_size));
	hipMemcpy(GPU_data_in4, CPU_data_in, sizeof(int) * data_size, hipMemcpyHostToDevice);
			
	tlb_latency_test<<<1, 1>>>(GPU_data_in2, iterations, GPU_data_out, clock_rate, 2);//////////////////////////////////////////////kernel is here			
    hipDeviceSynchronize();		
	
	checkCudaErrors(hipFree(GPU_data_in2));
	////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////iter2
	
	
	checkCudaErrors(hipFree(GPU_data_out));	
	free(CPU_data_in);
	
    exit(EXIT_SUCCESS);
}
