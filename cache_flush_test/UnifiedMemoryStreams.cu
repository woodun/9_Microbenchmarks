#include "hip/hip_runtime.h"
#include <cstdio>
#include <ctime>
#include <vector>
#include <algorithm>
#include <stdlib.h>
// utilities
#include <hip/hip_runtime_api.h>
#include <time.h>

//////////cache flush test: can I test multiple kernels in the same run? will they cause cache hits? then I can launch different strides to figure out if the tlb and cache miss or not.

void init_cpu_data(int* A, int size, int stride){
	for (int i = 0; i < size; ++i){
		A[i]=(i + stride) % size;
   	}
}

//////////min page size 4kb = 4096b = 32 * 128.
__device__ void tlb_warmup(int *A, int iterations, int *B, float clock_rate){
	
	long long int start_time = 0;
	long long int end_time = 0;
	
	start_time = clock64();
	
	//iterations = 8;///////should not saturate the tlb
	
	int j = 31;/////make them in the same page, but far in cache lines
	
	for (int it =0; it < iterations; it ++){
		j = A[j];
	}	
	
	B[0] = j;
	
	end_time=clock64();
	long long int total_time = end_time - start_time;
	printf("inside:%lld\n", total_time);
	printf("inside:%fms\n", total_time / (float)clock_rate);

}

/*
__device__ void cache_warmup(int *A, int iterations, int *B){
	
	int j = 0;
	
	for (int it =0; it < iterations; it ++){
		j = A[j];
	}
	
	B[0] = j;
}
*/

//////////min page size 4kb = 4096b = 32 * 128.
__device__ void cache_miss_1(int *A, int iterations, int *B){//////////////should hit in the tlb, but miss in the cache, to prove tlb hit exists.
	
	//iterations = 8;///////should not saturate the tlb
	
	int j = 0;/////make them in the same page, but far in cache lines
	
	for (int it =0; it < iterations; it ++){
		j = A[j];
	}
	
	B[0] = j;
}

//////////min page size 4kb = 4096b = 32 * 128.
__device__ void cache_miss_2(int *A, int iterations, int *B){//////////////should hit in the tlb, but miss in the cache, to prove tlb hit exists.
	
	//iterations = 8;///////should not saturate the tlb
	
	int j = 8;/////make them in the same page, but far in cache lines
	
	for (int it =0; it < iterations; it ++){
		j = A[j];
	}
	
	B[0] = j;
}

//////////min page size 4kb = 4096b = 32 * 128.
__device__ void cache_miss_3(int *A, int iterations, int *B){//////////////should hit in the tlb, but miss in the cache, to prove tlb hit exists.
	
	//iterations = 8;///////should not saturate the tlb
	
	int j = 16;/////make them in the same page, but far in cache lines
	
	for (int it =0; it < iterations; it ++){
		j = A[j];
	}
	
	B[0] = j;
}


//////////min page size 4kb = 4096b = 32 * 128.
__device__ void cache_miss_4(int *A, int iterations, int *B){//////////////should not hit in the tlb, and should also miss in the cache, to see the time difference.
	
	//iterations = 8;///////should not saturate the tlb
	
	int j = 8388608;/////make them in the different page, 524288 * 16 = 8388608. 2m * 16. The 17th page.
	
	for (int it =0; it < iterations; it ++){
		j = A[j];
	}
	
	B[0] = j;
}

//////////min page size 4kb = 4096b = 32 * 128.
__device__ void cache_miss_5(int *A, int iterations, int *B){//////////////should not hit in the tlb, and should also miss in the cache, to see the time difference.
	
	//iterations = 8;///////should not saturate the tlb
	
	int j = 16777216;/////make them in the different page, 524288 * 32 = 8388608. 2m * 32. The 33rd page.
	
	for (int it =0; it < iterations; it ++){
		j = A[j];
	}
	
	B[0] = j;
}

//////////min page size 4kb = 4096b = 32 * 128.
__device__ void cache_miss_6(int *A, int iterations, int *B){//////////////should not hit in the tlb, and should also miss in the cache, to see the time difference.
	
	//iterations = 8;///////should not saturate the tlb
	
	int j = 16777216;/////make them in the different page, 524288 * 48 = 25165824. 2m * 48. The 49th page.
	
	for (int it =0; it < iterations; it ++){
		j = A[j];
	}
	
	B[0] = j;
}


__global__ void tlb_latency_test(int *A, int iterations, int *B, float clock_rate){
	
	//int j = 0;	
	//for (int it =0; it < iterations; it ++){
	//	j = A[j];
	//}	
	//B[0] = j;

	long long int start_time = 0;
	long long int end_time = 0;
	
	long long int end_time2 = 0;
	
	start_time = clock64();
		
	tlb_warmup(A, 8, B, clock_rate);	
	
	end_time2=clock64();
		
	cache_miss_1(A, 8, B);
	cache_miss_2(A, 8, B);
	cache_miss_3(A, 8, B);
	cache_miss_4(A, 8, B);
	cache_miss_5(A, 8, B);
	cache_miss_6(A, 8, B);
	
	end_time=clock64();	
	
	long long int total_time2 = end_time2 - start_time;	
	printf("outside2:%fms\n", total_time2 / (float)clock_rate);
	
	long long int total_time = end_time - start_time;
	printf("outside1:%fms\n", total_time / (float)clock_rate);
}

int main(int argc, char **argv)
{
	
    // set device
    hipDeviceProp_t device_prop;
    //int dev_id = findCudaDevice(argc, (const char **) argv);
	int dev_id = 0;
    checkCudaErrors(hipGetDeviceProperties(&device_prop, dev_id));
	
	int peak_clk = 1;//kHz
	checkCudaErrors(hipDeviceGetAttribute(&peak_clk, hipDeviceAttributeClockRate, dev_id));
	float clock_rate = (float) peak_clk;
	
	printf("clock_rate_out_kernel:%f\n", clock_rate);

    if (!device_prop.managedMemory) { 
        // This samples requires being run on a device that supports Unified Memory
        fprintf(stderr, "Unified Memory not supported on this device\n");

        exit(EXIT_WAIVED);
    }

    if (device_prop.computeMode == hipComputeModeProhibited)
    {
        // This sample requires being run with a default or process exclusive mode
        fprintf(stderr, "This sample requires a device in either default or process exclusive mode\n");

        exit(EXIT_WAIVED);
    }
	
	//printf("%d\n", sizeof(int));//////////size of int is 4.
	//exit(0);
	
	//////////////CPU data begin
	////////size(int) = 4, 256 = 1kb, 262144 = 1mb, 524288 = 2mb.
	int iterations = 1000;
	int data_stride = 524288;/////2mb.
	//int data_size = 524288000;/////1000 * 2mb. ##### size = iteration * stride. ##### This can support 1000 iteration. The 1001st iteration starts from head again.
	int data_size = iterations * data_stride;/////size = iteration * stride.
	
	int *CPU_data_in;	
	CPU_data_in = (int*)malloc(sizeof(int) * data_size);
	//int *CPU_data_out;
	//CPU_data_out = (int*)malloc(data_size * sizeof(int));
	
	init_cpu_data(CPU_data_in, data_size, data_stride);
	//////////////CPU data end
	
	//////////////GPU data begin
	int *GPU_data_in;
	//////checkCudaErrors(hipMallocManaged(&data, sizeof(int) * data_size));
	checkCudaErrors(hipMalloc(&GPU_data_in, sizeof(int) * data_size));
	
	int *GPU_data_out;
	checkCudaErrors(hipMalloc(&GPU_data_out, sizeof(int) * 1));
	
	hipMemcpy(GPU_data_in, CPU_data_in, sizeof(int) * data_size, hipMemcpyHostToDevice);
	//////////////GPU data end
				
    hipEvent_t start, stop;////////events timer is not accurate.
	hipEventCreate(&start);
	hipEventCreate(&stop);	

	//////////////kernel begin
	hipEventRecord(start);	
	hipEventSynchronize(start);
	
	///////////CPU timer
	struct timespec ts_start, ts_end;
	clock_gettime(CLOCK_REALTIME, ts_start);///////////CPU timer
	
	tlb_latency_test<<<1, 1>>>(GPU_data_in, iterations, GPU_data_out, clock_rate);
	
	///////////CPU timer
	clock_gettime(CLOCK_REALTIME, ts_end);///////////CPU timer
	
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	//////////////kernel end
	
	//hipMemcpy(CPU_data_out, GPU_data_out, sizeof(int) * data_size, hipMemcpyDeviceToHost);
	
    hipDeviceSynchronize();
	
	///////////CPU timer
	printf("CPU clock: %lu\n", ts_end.tv_nsec - ts_start.tv_nsec);///////////CPU timer
	
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
    printf("out kernel:%f %f\n", milliseconds, milliseconds / iterations);
	
    exit(EXIT_SUCCESS);
}
