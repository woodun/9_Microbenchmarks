#include "hip/hip_runtime.h"
#include <cstdio>
#include <ctime>
#include <vector>
#include <algorithm>
#include <stdlib.h>
// utilities
#include <hip/hip_runtime_api.h>
#include <time.h>

/////////////////////////////prefetch test for dense accesses.


void init_cpu_data(int* A, int size, int stride){
	for (int i = 0; i < size; ++i){
		A[i]=(i + stride) % size;
   	}
}

/*
__device__ void cache_warmup(int *A, int iterations, int *B){
	
	int j = 0;
	
	for (int it =0; it < iterations; it ++){
		j = A[j];
	}
	
	B[0] = j;
}
*/

//////////min page size 4kb = 4096b = 32 * 128.
__device__ void P_chasing(int mark, int *A, int iterations, int *B, int starting_index, float clock_rate){
	
	int j = starting_index;/////make them in the same page, and miss near in cache lines
	
	long long int start_time = 0;//////clock
	long long int end_time = 0;//////clock
	start_time = clock64();//////clock
	
	for (int it =0; it < iterations; it ++){
		j = A[j];
	}
	
	end_time=clock64();//////clock
	long long int total_time = end_time - start_time;//////clock
	printf("inside%d:%fms\n", mark, (total_time / (float)clock_rate) / (float)iterations);//////clock, average latency
	
	B[0] = j;
}

__global__ void tlb_latency_test(int *A, int iterations, int *B, float clock_rate){	

	int index = 0;
	
	long long int start_time = 0;///////////clock
	long long int end_time = 0;///////////clock	
	start_time = clock64();///////////clock
		
	//////////////////////////////////////////////////////32 * 32 * 32 = 128kb
	for(index = 4 * 32 * 32 - 256; index <= 2 * 4 * 32 * 32; index++){
		P_chasing(index, A, index, B, 0, clock_rate);/////warmup cache and TLB
		P_chasing(index, A, index, B, 0, clock_rate);/////try to generate hits	
	}
	
	end_time=clock64();///////////clock
		
	long long int total_time = end_time - start_time;///////////clock
	printf("outside1:%fms\n", total_time / (float)clock_rate);///////////clock
}

int main(int argc, char **argv)
{
	printf("\n");
	
    // set device
    hipDeviceProp_t device_prop;
    //int dev_id = findCudaDevice(argc, (const char **) argv);
	int dev_id = 0;
    checkCudaErrors(hipGetDeviceProperties(&device_prop, dev_id));
	
	int peak_clk = 1;//kHz
	checkCudaErrors(hipDeviceGetAttribute(&peak_clk, hipDeviceAttributeClockRate, dev_id));
	float clock_rate = (float) peak_clk;
	
	//printf("clock_rate_out_kernel:%f\n", clock_rate);

    if (!device_prop.managedMemory) { 
        // This samples requires being run on a device that supports Unified Memory
        fprintf(stderr, "Unified Memory not supported on this device\n");

        exit(EXIT_WAIVED);
    }

    if (device_prop.computeMode == hipComputeModeProhibited)
    {
        // This sample requires being run with a default or process exclusive mode
        fprintf(stderr, "This sample requires a device in either default or process exclusive mode\n");

        exit(EXIT_WAIVED);
    }
		
	///////////////////////////////////////////////////////////////////CPU data begin
	int iterations = 4 * 16384 * 100;
	////////size(int) = 4, 32 = 128b, 256 = 1kb, 32 * 32 = 1024 = 4kb, 262144 = 1mb, 16384 * 32 = 512 * 1024 = 524288 = 2mb.
	int data_stride = 8;/////128b. Pointing to the next cacheline.
	//int data_size = 524288000;/////1000 * 2mb. ##### size = iteration * stride. ##### This can support 1000 iteration. The 1001st iteration starts from head again.
	int data_size = iterations * data_stride;/////size = iteration * stride = 100 2mb pages.
	
	int *CPU_data_in;	
	CPU_data_in = (int*)malloc(sizeof(int) * data_size);
	//int *CPU_data_out;
	//CPU_data_out = (int*)malloc(data_size * sizeof(int));
	
	init_cpu_data(CPU_data_in, data_size, data_stride);
	///////////////////////////////////////////////////////////////////CPU data end
	
	///////////////////////////////////////////////////////////////////GPU data begin
	int *GPU_data_in;
	//////checkCudaErrors(hipMallocManaged(&data, sizeof(int) * data_size));
	checkCudaErrors(hipMalloc(&GPU_data_in, sizeof(int) * data_size));
	
	int *GPU_data_out;
	checkCudaErrors(hipMalloc(&GPU_data_out, sizeof(int) * 1));
	
	hipMemcpy(GPU_data_in, CPU_data_in, sizeof(int) * data_size, hipMemcpyHostToDevice);
	///////////////////////////////////////////////////////////////////GPU data end				  
		
	tlb_latency_test<<<1, 1>>>(GPU_data_in, iterations, GPU_data_out, clock_rate);//////////////////////////////////////////////kernel is here
	
	//hipMemcpy(CPU_data_out, GPU_data_out, sizeof(int) * data_size, hipMemcpyDeviceToHost);
	
    hipDeviceSynchronize();	
	
    exit(EXIT_SUCCESS);
}
