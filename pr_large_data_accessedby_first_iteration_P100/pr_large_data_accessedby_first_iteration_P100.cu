#include "hip/hip_runtime.h"
#include <cstdio>
#include <ctime>
#include <vector>
#include <algorithm>
#include <stdlib.h>
// utilities
#include <hip/hip_runtime_api.h>
#include <time.h>

/////////////////////////////change the data size to larger than 16 gb to test for different memories. L1 is enabled. "ALL_CCFLAGS += -Xptxas -dlcm=ca"

void init_cpu_data(long long int* A, long long int size, long long int stride, long long int mod){
	if(1){////////////normal
		for (long long int i = 0; i < size - stride; i = i + stride){
			A[i]=(i + stride);
		}
		
		//for (long long int i = 3; i < size - stride; i = i + stride){
		//	A[i]=(i + stride);
		//}
				
		A[size - stride]=0;
		//A[size - stride + 3]=0;
	}
	
	if(1){////////////reversed
		//for (long long int i = 0; i <= size - stride; i = i + stride){
		//	A[i]=(i - stride);
		//}
		
		for (long long int i = 3; i <= size - stride + 3; i = i + stride){
			A[i]=(i - stride);
		}
		
		//A[0]=size - stride;
		A[3]=size - stride + 3;
	}
	
	/////54521859 returned page fault starting point for 2147483648.
	///////////////////2147483648 - 54521859 = 2092961789.
	///////////////////2092961789 -4096 + 3 = 1996 * 1M = 15968 MB (out of 16280 MB out of 16384 MB)
	/////2202267651 returned page fault starting point for 4294967296
	///////////////////4294967296 - 2202267651 = 2092699645.
	///////////////////2092699645 -4096 + 3 = 1995.75 * 1M = 15966 MB (out of 16280 MB out of 16384 MB)
}

__device__ void P_chasing2(int mark, long long int *A, long long int iterations, long long int *B, long long int starting_index, float clock_rate, long long int data_stride){	
	
	__shared__ long long int s_index[1];
	
	long long int j = starting_index;/////make them in the same page, and miss near in cache lines
	
	long long int start_time = 0;//////clock
	long long int end_time = 0;//////clock
	long long int time_interval = 0;//////clock	
		
	if(true){
		if(mark){
		asm(".reg .u64 t1;\n\t"
		".reg .u64 t2;\n\t"
		".reg .u32 t3;\n\t"
		".reg .u32 t4;\n\t"
		".reg .u64 t5;\n\t"
		".reg .u32 t6;\n\t");
		}
		
		asm("cvta.to.shared.u64 	t5, %0;\n\t"
		"cvt.u32.u64 	t6, t5;\n\t"
		:: "l"(s_index));////////////////////////////////////cvta.to.global.u64 	%rd4, %rd25; needed??
		
		for (long long int it = 0; it < iterations; it++){//////////it here is limited by the size of the shared memory
			
			asm("shl.b64 	t1, %3, 3;\n\t"	
			"add.s64 	t2, t1, %4;\n\t"
			"shl.b32 	t3, %6, 3;\n\t"
			"add.s32 	t4, t3, t6;\n\t"		
			"mov.u64 	%0, %clock64;\n\t"
			"ld.global.u64 	%2, [t2];\n\t"
			"st.shared.u64 	[t4], %2;\n\t"
			"mov.u64	%1, %clock64;"
			: "=l"(start_time), "=l"(end_time), "=l"(j) : "l"(j), "l"(A), "l"(s_index), "r"(0));		
					
			time_interval = end_time - start_time;
			printf("%lld %lld\n", j, time_interval);/////printf will affect L1 cache. Also, unknown effect to TLBs because it adds latency to L2 TLB misses.
			//////////////////////////////////////We are not using it for measurement. However, it can be used to recognize different conditions.
		}
	}

	B[0] = j;
}

__global__ void tlb_latency_test(long long int *A, long long int iterations, long long int *B, float clock_rate, long long int mod, long long int data_stride){
			
	P_chasing2(1, A, iterations * 2, B, 0, clock_rate, data_stride);
	//P_chasing2(1, A, iterations, B, 0, clock_rate, data_stride);
	//P_chasing2(0, A, iterations, B, mod - data_stride + 3, clock_rate, data_stride);
	
	 __syncthreads();
}

int main(int argc, char **argv)
{
    // set device
    hipDeviceProp_t device_prop;
    //int dev_id = findCudaDevice(argc, (const char **) argv);
	int dev_id = 0;
    checkCudaErrors(hipGetDeviceProperties(&device_prop, dev_id));
	
	int peak_clk = 1;//kHz
	checkCudaErrors(hipDeviceGetAttribute(&peak_clk, hipDeviceAttributeClockRate, dev_id));
	float clock_rate = (float) peak_clk;
	
	printf("clock_rate_out_kernel:%f\n", clock_rate);

    if (!device_prop.managedMemory) { 
        // This samples requires being run on a device that supports Unified Memory
        fprintf(stderr, "Unified Memory not supported on this device\n");

        exit(EXIT_WAIVED);
    }

    if (device_prop.computeMode == hipComputeModeProhibited)
    {
        // This sample requires being run with a default or process exclusive mode
        fprintf(stderr, "This sample requires a device in either default or process exclusive mode\n");

        exit(EXIT_WAIVED);
    }
	
	if (device_prop.concurrentManagedAccess == 1){
		printf("This device supports concurrent Managed Access.\n");
    }else{
		printf("This device does not support concurrent Managed Access.\n");
	}
	
	int value1 = 1;
	checkCudaErrors(hipDeviceGetAttribute(&value1, hipDeviceAttributeConcurrentManagedAccess, dev_id));
	printf("hipDeviceAttributeConcurrentManagedAccess = %d\n", value1);	
	
	///////////////////////////////////////////////////////////////////GPU data out
	long long int *GPU_data_out;
	checkCudaErrors(hipMalloc(&GPU_data_out, sizeof(long long int) * 2));			
	
	int counter = 0;	
	for(long long int data_stride = 1 * 4 * 1024; data_stride <= 1 * 4 * 1024; data_stride = data_stride * 2){/////////32mb stride

	//plain managed
	printf("*\n*\n*\n plain managed\n");	
	for(long long int mod = 1073741824; mod <= 4294967296; mod = mod * 2){////268435456 = 2gb, 536870912 = 4gb, 1073741824 = 8gb, 2147483648 = 16gb, 4294967296 = 32gb, 8589934592 = 64gb.
		counter++;
		///////////////////////////////////////////////////////////////////CPU data begin
		long long int data_size = mod;
		long long int iterations = mod / data_stride;////32 * 32 * 4 / 32 * 2 = 256
	
		long long int *CPU_data_in;
		//CPU_data_in = (int*)malloc(sizeof(int) * data_size);
		checkCudaErrors(hipMallocManaged(&CPU_data_in, sizeof(long long int) * data_size));/////////////using unified memory
checkCudaErrors(hipMemAdvise(CPU_data_in, sizeof(long long int) * data_size, hipMemAdviseSetAccessedBy, hipCpuDeviceId));//////////using hint		
		init_cpu_data(CPU_data_in, data_size, data_stride, mod);		
		///////////////////////////////////////////////////////////////////CPU data end	
	
		///////////////////////////////////////////////////////////////////GPU data in	
		//int *GPU_data_in;
		//checkCudaErrors(hipMalloc(&GPU_data_in, sizeof(int) * data_size));	
		//hipMemcpy(GPU_data_in, CPU_data_in, sizeof(int) * data_size, hipMemcpyHostToDevice);		
		
		printf("###################data_stride%lld#########################\n", data_stride);
		printf("###############Mod%lld##############%lld\n", mod, iterations);		
		
		tlb_latency_test<<<1, 1>>>(CPU_data_in, iterations, GPU_data_out, clock_rate, mod, data_stride);///kernel is here	
		hipDeviceSynchronize();
		
		//checkCudaErrors(hipFree(GPU_data_in));
		checkCudaErrors(hipFree(CPU_data_in));
		//free(CPU_data_in);		
	}
	}
			
	checkCudaErrors(hipFree(GPU_data_out));	
	
    exit(EXIT_SUCCESS);
}