#include "hip/hip_runtime.h"
#include <cstdio>
#include <ctime>
#include <vector>
#include <algorithm>
#include <stdlib.h>
// utilities
#include <hip/hip_runtime_api.h>
#include <time.h>
#include <sys/time.h>

/////////////////////////////L1 is enabled. "ALL_CCFLAGS += -Xptxas -dlcm=ca"

void init_cpu_data(int* A, int size, int stride){
	
	for (int i = 0; i < size; i++){
		A[i]=1;
	}
	
	/*
	for (long long int i = 0; i < size - stride; i++){
		A[i]=(i + stride);
	}
			
	for (long long int i = size - stride; i < size; i++){
		A[i]=0;
	}
	*/
}

long long unsigned time_diff(timespec start, timespec end){
	struct timespec temp;
	if ((end.tv_nsec - start.tv_nsec) < 0){
		temp.tv_sec = end.tv_sec - start.tv_sec - 1;
		temp.tv_nsec = 1000000000 + end.tv_nsec - start.tv_nsec;
	} 
	else{
		temp.tv_sec = end.tv_sec - start.tv_sec;
		temp.tv_nsec = end.tv_nsec - start.tv_nsec;
	}
	
	long long unsigned time_interval_ns = temp.tv_nsec;
	long long unsigned time_interval_s = temp.tv_sec;
	time_interval_s = time_interval_s * 1000000000;
	
	return time_interval_s + time_interval_ns;
}

__global__ void Page_visitor(int *A, int *B, int data_stride, long long int clock_count){
		
	/*
	int index = threadIdx.x;
	
	/////////////////////////////////time
	long long int start_time = 0;//////clock
	long long int end_time = 0;//////clock
	long long int time_interval = 0;//////clock
	
	if(index = 0){
		start_time= clock64();
	}
	__syncthreads();
	*/
	
	int index = threadIdx.x * data_stride;
	
	int value = A[index];
	
	//////////////////////////////////////////////sleep
	long long int start_clock = clock64();
    long long int clock_offset = 0;
    while (clock_offset < clock_count)
    {
        clock_offset = clock64() - start_clock;
    }
    //d_o[0] = clock_offset;
	//////////////////////////////////////////////sleep
	
	B[index] = value;
	
	/*
	__syncthreads();
	/////////////////////////////////time
	if(index = 0){
		start_time= clock64();
		time_interval = end_time - start_time;//////clock
	}	
	//B[0] = time_interval;
	*/
}

int main(int argc, char **argv)
{
	printf("\n");
	
    // set device
    hipDeviceProp_t device_prop;
    //int dev_id = findCudaDevice(argc, (const char **) argv);
	int dev_id = 0;
    checkCudaErrors(hipGetDeviceProperties(&device_prop, dev_id));
	
	int peak_clk = 1;//kHz
	checkCudaErrors(hipDeviceGetAttribute(&peak_clk, hipDeviceAttributeClockRate, dev_id));
	float clock_rate = (float) peak_clk;
	
	printf("clock_rate:%f\n", clock_rate);

    if (!device_prop.managedMemory) { 
        // This samples requires being run on a device that supports Unified Memory
        fprintf(stderr, "Unified Memory not supported on this device\n");

        exit(EXIT_WAIVED);
    }

    if (device_prop.computeMode == hipComputeModeProhibited)
    {
        // This sample requires being run with a default or process exclusive mode
        fprintf(stderr, "This sample requires a device in either default or process exclusive mode\n");

        exit(EXIT_WAIVED);
    }
	
	if (device_prop.concurrentManagedAccess == 1){
		printf("This device supports concurrent Managed Access.\n");
    }else{
		printf("This device does not support concurrent Managed Access.\n");
	}
	
	int value1 = 1;
	checkCudaErrors(hipDeviceGetAttribute(&value1, hipDeviceAttributeConcurrentManagedAccess, dev_id));
	printf("hipDeviceAttributeConcurrentManagedAccess = %d\n", value1);	
	
	//plain managed
	printf("*\n*\n*\n plain managed\n");
	for(int data_stride = 2 * 256 * 1024; data_stride <= 2 * 256 * 1024; data_stride = data_stride * 2){
	for(int mod = 8589934592; mod <= 8589934592; mod = mod * 2){////268435456 = 1gb, 536870912 = 2gb, 1073741824 = 4gb, 2147483648 = 8gb, 4294967296 = 16gb, 8589934592 = 32gb.	
	for(long long int clock_count = 1000; clock_count <= 1000; clock_count = clock_count * 2){
		///////////////////////////////////////////////////////////////////CPU data begin		
		int data_size = mod;
		//int iterations = mod / data_stride;////32 * 32 * 4 / 32 * 2 = 256
	
		int *CPU_data_in;
		//CPU_data_in = (int*)malloc(sizeof(int) * data_size);
		checkCudaErrors(hipMallocManaged(&CPU_data_in, sizeof(int) * data_size));/////////////using unified memory		
		init_cpu_data(CPU_data_in, data_size, data_stride);				
		///////////////////////////////////////////////////////////////////CPU data end	
	
		///////////////////////////////////////////////////////////////////GPU data in	
		//int *GPU_data_in;
		//checkCudaErrors(hipMalloc(&GPU_data_in, sizeof(int) * data_size));	
		//hipMemcpy(GPU_data_in, CPU_data_in, sizeof(int) * data_size, hipMemcpyHostToDevice);
		
		///////////////////////////////////////////////////////////////////GPU data out
		int *GPU_data_out;
		//checkCudaErrors(hipMalloc(&GPU_data_out, sizeof(int) * 2));
		checkCudaErrors(hipMallocManaged(&GPU_data_out, sizeof(int) * data_size));/////////////using unified memory		
				
		/////////////////////////////////time
		struct timespec ts1;
		clock_gettime(CLOCK_REALTIME, &ts1);
  
		Page_visitor<<<8, 2048>>>(CPU_data_in, GPU_data_out, data_stride, clock_count);///////////////kernel is here	
		hipDeviceSynchronize();
				
		/////////////////////////////////time
		struct timespec ts2;
		clock_gettime(CLOCK_REALTIME, &ts2);
		
		printf("###################data_stride%d#########################clock_count:%lld\n", data_stride, clock_count);
		printf("*\n*\n*\nruntime:  %lluns\n", time_diff(ts1, ts2));	
		
		//checkCudaErrors(hipFree(GPU_data_in));
		checkCudaErrors(hipFree(CPU_data_in));
		//free(CPU_data_in);
		checkCudaErrors(hipFree(GPU_data_out));
	}
	}
	}
	
	/*
	//memcopy
	printf("*\n*\n*\n memcopy\n");
	for(int data_stride = 1 * 128 * 1024; data_stride <= 2 * 256 * 1024; data_stride = data_stride * 2){
	for(int mod = 536870912; mod <= 536870912; mod = mod * 2){////268435456 = 1gb, 536870912 = 2gb, 1073741824 = 4gb, 2147483648 = 8gb, 4294967296 = 16gb, 8589934592 = 32gb.	
	for(long long int clock_count = 1000; clock_count <= 1000; clock_count = clock_count * 2){

		///////////////////////////////////////////////////////////////////CPU data begin		
		int data_size = mod;
		//int iterations = mod / data_stride;////32 * 32 * 4 / 32 * 2 = 256
	
		int *CPU_data_in;
		CPU_data_in = (int*)malloc(sizeof(int) * data_size);		
		init_cpu_data(CPU_data_in, data_size, data_stride);				
		///////////////////////////////////////////////////////////////////CPU data end	
	
		///////////////////////////////////////////////////////////////////GPU data in	
		int *GPU_data_in;
		checkCudaErrors(hipMalloc(&GPU_data_in, sizeof(int) * data_size));	
		
		///////////////////////////////////////////////////////////////////GPU data out
		int *GPU_data_out;
		checkCudaErrors(hipMalloc(&GPU_data_out, sizeof(int) * data_size));
				
		/////////////////////////////////time
		struct timespec ts1;
		clock_gettime(CLOCK_REALTIME, &ts1);

		hipMemcpy(GPU_data_in, CPU_data_in, sizeof(int) * data_size, hipMemcpyHostToDevice);
		Page_visitor<<<1, 512>>>(GPU_data_in, GPU_data_out, data_stride, clock_count);///////////////kernel is here	
		hipDeviceSynchronize();
				
		/////////////////////////////////time
		struct timespec ts2;
		clock_gettime(CLOCK_REALTIME, &ts2);
		
		printf("###################data_stride%d#########################clock_count:%lld\n", data_stride, clock_count);
		printf("*\n*\n*\nruntime:  %lluns\n", time_diff(ts1, ts2));
		
		checkCudaErrors(hipFree(GPU_data_in));
		//checkCudaErrors(hipFree(CPU_data_in));
		free(CPU_data_in);
		checkCudaErrors(hipFree(GPU_data_out));
	}
	}
	}
	*/
	
    exit(EXIT_SUCCESS);
}