#include "hip/hip_runtime.h"
#include <cstdio>
#include <ctime>
#include <vector>
#include <algorithm>
#include <stdlib.h>
// utilities
#include <hip/hip_runtime_api.h>
#include <time.h>
#include <sys/time.h>

/////////////////////////////L1 is enabled. "ALL_CCFLAGS += -Xptxas -dlcm=ca"

//typedef unsigned char byte;

void init_cpu_data(int* A, int size, int stride){
	
	for (int i = 0; i < size; i++){
		A[i]=1;
	}
	
	/*
	for (long long int i = 0; i < size - stride; i++){
		A[i]=(i + stride);
	}
			
	for (long long int i = size - stride; i < size; i++){
		A[i]=0;
	}
	*/
}

__global__ void Page_visitor(int *A, int *B, int data_stride, long long int clock_count){
		
	/*
	int index = threadIdx.x;
	
	/////////////////////////////////time
	long long int start_time = 0;//////clock
	long long int end_time = 0;//////clock
	long long int time_interval = 0;//////clock
	
	if(index = 0){
		start_time= clock64();
	}
	__syncthreads();
	*/
	
	int index = threadIdx.x * data_stride;
	
	int value = A[index];
	
	//////////////////////////////////////////////sleep
	long long int start_clock = clock64();
    long long int clock_offset = 0;
    while (clock_offset < clock_count)
    {
        clock_offset = clock64() - start_clock;
    }
    //d_o[0] = clock_offset;
	//////////////////////////////////////////////sleep
	
	B[index] = value;
	
	/*
	__syncthreads();
	/////////////////////////////////time
	if(index = 0){
		start_time= clock64();
		time_interval = end_time - start_time;//////clock
	}	
	//B[0] = time_interval;
	*/
}

int main(int argc, char **argv)
{
	printf("\n");
	
    // set device
    hipDeviceProp_t device_prop;
    //int dev_id = findCudaDevice(argc, (const char **) argv);
	int dev_id = 0;
    checkCudaErrors(hipGetDeviceProperties(&device_prop, dev_id));
	
	int peak_clk = 1;//kHz
	checkCudaErrors(hipDeviceGetAttribute(&peak_clk, hipDeviceAttributeClockRate, dev_id));
	float clock_rate = (float) peak_clk;
	
	printf("clock_rate_out_kernel:%f\n", clock_rate);

    if (!device_prop.managedMemory) { 
        // This samples requires being run on a device that supports Unified Memory
        fprintf(stderr, "Unified Memory not supported on this device\n");

        exit(EXIT_WAIVED);
    }

    if (device_prop.computeMode == hipComputeModeProhibited)
    {
        // This sample requires being run with a default or process exclusive mode
        fprintf(stderr, "This sample requires a device in either default or process exclusive mode\n");

        exit(EXIT_WAIVED);
    }
	
	if (device_prop.concurrentManagedAccess == 1){
		printf("This device supports concurrent Managed Access.\n");
    }else{
		printf("This device does not support concurrent Managed Access.\n");
	}
	
	int value1 = 1;
	checkCudaErrors(hipDeviceGetAttribute(&value1, hipDeviceAttributeConcurrentManagedAccess, dev_id));
	printf("hipDeviceAttributeConcurrentManagedAccess = %d\n", value1);	
		
	FILE * pFile;
    pFile = fopen ("output.txt","w");		
	
	int counter = 0;
	
	//plain managed
	printf("*\n*\n*\n plain managed\n");
	for(int data_stride = 1 * 128 * 1024; data_stride <= 2 * 256 * 1024; data_stride = data_stride * 2){

	for(int mod = 268435456; mod <= 268435456; mod = mod * 2){////268435456 = 1gb, 536870912 = 2gb, 1073741824 = 4gb, 2147483648 = 8gb, 4294967296 = 16gb, 8589934592 = 32gb.
	
	for(long long int clock_count = 1000; clock_count <= 1000; clock_count = clock_count * 2){
		counter++;
		///////////////////////////////////////////////////////////////////CPU data begin		
		int data_size = mod;
		if(data_size < 4194304){//////////data size at least 16mb to prevent L2 prefetch
			data_size = 4194304;
		}		
		//int iterations = mod / data_stride;////32 * 32 * 4 / 32 * 2 = 256
	
		int *CPU_data_in;
		//CPU_data_in = (int*)malloc(sizeof(int) * data_size);
		checkCudaErrors(hipMallocManaged(&CPU_data_in, sizeof(int) * data_size));/////////////using unified memory		
		init_cpu_data(CPU_data_in, data_size, data_stride);				
		///////////////////////////////////////////////////////////////////CPU data end	
	
		///////////////////////////////////////////////////////////////////GPU data in	
		//int *GPU_data_in;
		//checkCudaErrors(hipMalloc(&GPU_data_in, sizeof(int) * data_size));	
		//hipMemcpy(GPU_data_in, CPU_data_in, sizeof(int) * data_size, hipMemcpyHostToDevice);
		
		///////////////////////////////////////////////////////////////////GPU data out
		int *GPU_data_out;
		//checkCudaErrors(hipMalloc(&GPU_data_out, sizeof(int) * 2));
		checkCudaErrors(hipMallocManaged(&GPU_data_out, sizeof(int) * data_size));/////////////using unified memory		
				
		/////////////////////////////////time
		struct timespec ts1;
		clock_gettime(CLOCK_REALTIME, &ts1);

		//printf("s:  %lu\n", ts1.tv_sec);
		//printf("ns: %lu\n", ts1.tv_nsec);
  
		Page_visitor<<<1, 512>>>(CPU_data_in, GPU_data_out, data_stride, clock_count);///////////////kernel is here	
		hipDeviceSynchronize();
				
		/////////////////////////////////time
		struct timespec ts2;
		clock_gettime(CLOCK_REALTIME, &ts2);

		//printf("s:  %lu\n", ts2.tv_sec);
		//printf("ns: %lu\n", ts2.tv_nsec);
		//printf("s:  %lu\n", ts2.tv_sec - ts1.tv_sec);
		
		printf("###################data_stride%d#########################clock_count:%lld\n", data_stride, clock_count);
		printf("runtime:  %luns\n", ts2.tv_nsec - ts1.tv_nsec);		
		
		//checkCudaErrors(hipFree(GPU_data_in));
		checkCudaErrors(hipFree(CPU_data_in));
		//free(CPU_data_in);
		checkCudaErrors(hipFree(GPU_data_out));
	}
	}
	}
	
	/*
	//preferredlocation
	fprintf(pFile,"*\n*\n*\n preferredlocation\n");
	fflush(pFile);
	for(long long int mod2 = 1073741824; mod2 <= 4294967296; mod2 = mod2 * 2){////268435456 = 1gb, 536870912 = 2gb, 1073741824 = 4gb, 2147483648 = 8gb, 4294967296 = 16gb, 8589934592 = 32gb.
		counter++;
		///////////////////////////////////////////////////////////////////CPU data begin
		//int data_size = 2 * 256 * 1024 * 32;/////size = iteration * stride = 32 2mb pages.
		long long int mod = mod2;

		long long int data_size = mod;
		if(data_size < 4194304){//////////data size at least 16mb to prevent L2 prefetch
			data_size = 4194304;
		}	
		//int iterations = data_size / data_stride;
		//int iterations = 1024 * 256 * 8;
		long long int iterations = mod / data_stride;////32 * 32 * 4 / 32 * 2 = 256
	
		long long int *CPU_data_in;
		//CPU_data_in = (int*)malloc(sizeof(int) * data_size);
		checkCudaErrors(hipMallocManaged(&CPU_data_in, sizeof(long long int) * data_size));/////////////using unified memory
		checkCudaErrors(hipMemAdvise(CPU_data_in, sizeof(long long int) * data_size, hipMemAdviseSetPreferredLocation, hipCpuDeviceId));////////using hint		
		init_cpu_data(CPU_data_in, data_size, data_stride, mod);
		
		long long int reduced_iter = iterations;
		if(reduced_iter > 2048){
			reduced_iter = 2048;
		}else if(reduced_iter < 16){
			reduced_iter = 16;
		}
		
		long long int *CPU_data_out_index;
		CPU_data_out_index = (long long int*)malloc(sizeof(long long int) * reduced_iter);
		long long int *CPU_data_out_time;
		CPU_data_out_time = (long long int*)malloc(sizeof(long long int) * reduced_iter);
		///////////////////////////////////////////////////////////////////CPU data end	
	
		///////////////////////////////////////////////////////////////////GPU data in	
		//int *GPU_data_in;
		//checkCudaErrors(hipMalloc(&GPU_data_in, sizeof(int) * data_size));	
		//hipMemcpy(GPU_data_in, CPU_data_in, sizeof(int) * data_size, hipMemcpyHostToDevice);
		
		///////////////////////////////////////////////////////////////////GPU data out
		long long int *GPU_data_out_index;
		checkCudaErrors(hipMalloc(&GPU_data_out_index, sizeof(long long int) * reduced_iter));
		long long int *GPU_data_out_time;
		checkCudaErrors(hipMalloc(&GPU_data_out_time, sizeof(long long int) * reduced_iter));
		
		tlb_latency_test<<<1, 1>>>(CPU_data_in, iterations, GPU_data_out, GPU_data_out_index, GPU_data_out_time, clock_rate, mod, data_stride);///////////////kernel is here	
		hipDeviceSynchronize();
				
		hipMemcpy(CPU_data_out_index, GPU_data_out_index, sizeof(long long int) * reduced_iter, hipMemcpyDeviceToHost);
		hipMemcpy(CPU_data_out_time, GPU_data_out_time, sizeof(long long int) * reduced_iter, hipMemcpyDeviceToHost);
				

		fprintf(pFile, "###################data_stride%d#########################\n", data_stride);
		fprintf (pFile, "###############Mod%lld##############%lld\n", mod, iterations);
		for (long long int it = 0; it < reduced_iter; it++){		
			fprintf (pFile, "%lld %fms %lldcycles\n", CPU_data_out_index[it], CPU_data_out_time[it] / (float)clock_rate, CPU_data_out_time[it]);
			//fprintf (pFile, "%d %fms\n", it, CPU_data_out_time[it] / (float)clock_rate);
			//printf ("%d %fms\n", CPU_data_out_index[it], CPU_data_out_time[it] / (float)clock_rate);
		}
		fflush(pFile);
		
		checkCudaErrors(hipFree(GPU_data_out_index));
		checkCudaErrors(hipFree(GPU_data_out_time));
		//checkCudaErrors(hipFree(GPU_data_in));
		checkCudaErrors(hipFree(CPU_data_in));
		//free(CPU_data_in);
		free(CPU_data_out_index);
		free(CPU_data_out_time);
	}
	
	//accessedby
	fprintf(pFile,"*\n*\n*\n accessedby\n");
	fflush(pFile);
	for(long long int mod2 = 1073741824; mod2 <= 4294967296; mod2 = mod2 * 2){////268435456 = 1gb, 536870912 = 2gb, 1073741824 = 4gb, 2147483648 = 8gb, 4294967296 = 16gb, 8589934592 = 32gb.
		counter++;
		///////////////////////////////////////////////////////////////////CPU data begin
		//int data_size = 2 * 256 * 1024 * 32;/////size = iteration * stride = 32 2mb pages.
		long long int mod = mod2;

		long long int data_size = mod;
		if(data_size < 4194304){//////////data size at least 16mb to prevent L2 prefetch
			data_size = 4194304;
		}	
		//int iterations = data_size / data_stride;
		//int iterations = 1024 * 256 * 8;
		long long int iterations = mod / data_stride;////32 * 32 * 4 / 32 * 2 = 256
	
		long long int *CPU_data_in;
		//CPU_data_in = (int*)malloc(sizeof(int) * data_size);
		checkCudaErrors(hipMallocManaged(&CPU_data_in, sizeof(long long int) * data_size));/////////////using unified memory
		checkCudaErrors(hipMemAdvise(CPU_data_in, sizeof(long long int) * data_size, hipMemAdviseSetAccessedBy, dev_id));//////////using hint	
		init_cpu_data(CPU_data_in, data_size, data_stride, mod);
		
		long long int reduced_iter = iterations;
		if(reduced_iter > 2048){
			reduced_iter = 2048;
		}else if(reduced_iter < 16){
			reduced_iter = 16;
		}
		
		long long int *CPU_data_out_index;
		CPU_data_out_index = (long long int*)malloc(sizeof(long long int) * reduced_iter);
		long long int *CPU_data_out_time;
		CPU_data_out_time = (long long int*)malloc(sizeof(long long int) * reduced_iter);
		///////////////////////////////////////////////////////////////////CPU data end	
	
		///////////////////////////////////////////////////////////////////GPU data in	
		//int *GPU_data_in;
		//checkCudaErrors(hipMalloc(&GPU_data_in, sizeof(int) * data_size));
		//hipMemcpy(GPU_data_in, CPU_data_in, sizeof(int) * data_size, hipMemcpyHostToDevice);
		
		///////////////////////////////////////////////////////////////////GPU data out
		long long int *GPU_data_out_index;
		checkCudaErrors(hipMalloc(&GPU_data_out_index, sizeof(long long int) * reduced_iter));
		long long int *GPU_data_out_time;
		checkCudaErrors(hipMalloc(&GPU_data_out_time, sizeof(long long int) * reduced_iter));
		
		tlb_latency_test<<<1, 1>>>(CPU_data_in, iterations, GPU_data_out, GPU_data_out_index, GPU_data_out_time, clock_rate, mod, data_stride);///////////////kernel is here	
		hipDeviceSynchronize();
				
		hipMemcpy(CPU_data_out_index, GPU_data_out_index, sizeof(long long int) * reduced_iter, hipMemcpyDeviceToHost);
		hipMemcpy(CPU_data_out_time, GPU_data_out_time, sizeof(long long int) * reduced_iter, hipMemcpyDeviceToHost);
				

		fprintf(pFile, "###################data_stride%d#########################\n", data_stride);
		fprintf (pFile, "###############Mod%lld##############%lld\n", mod, iterations);
		for (long long int it = 0; it < reduced_iter; it++){
			fprintf (pFile, "%lld %fms %lldcycles\n", CPU_data_out_index[it], CPU_data_out_time[it] / (float)clock_rate, CPU_data_out_time[it]);
			//fprintf (pFile, "%d %fms\n", it, CPU_data_out_time[it] / (float)clock_rate);
			//printf ("%d %fms\n", CPU_data_out_index[it], CPU_data_out_time[it] / (float)clock_rate);
		}
		fflush(pFile);
		
		checkCudaErrors(hipFree(GPU_data_out_index));
		checkCudaErrors(hipFree(GPU_data_out_time));
		//checkCudaErrors(hipFree(GPU_data_in));
		checkCudaErrors(hipFree(CPU_data_in));
		//free(CPU_data_in);
		free(CPU_data_out_index);
		free(CPU_data_out_time);
	}
	
	//pinned
	fprintf(pFile,"*\n*\n*\n pinned\n");
	fflush(pFile);
	for(long long int mod2 = 1073741824; mod2 <= 4294967296; mod2 = mod2 * 2){////268435456 = 1gb, 536870912 = 2gb, 1073741824 = 4gb, 2147483648 = 8gb, 4294967296 = 16gb, 8589934592 = 32gb.
		counter++;
		///////////////////////////////////////////////////////////////////CPU data begin
		//int data_size = 2 * 256 * 1024 * 32;/////size = iteration * stride = 32 2mb pages.
		long long int mod = mod2;

		long long int data_size = mod;
		if(data_size < 4194304){//////////data size at least 16mb to prevent L2 prefetch
			data_size = 4194304;
		}	
		//int iterations = data_size / data_stride;
		//int iterations = 1024 * 256 * 8;
		long long int iterations = mod / data_stride;////32 * 32 * 4 / 32 * 2 = 256
	
		long long int *CPU_data_in;
		//CPU_data_in = (int*)malloc(sizeof(int) * data_size);
		checkCudaErrors(hipHostAlloc((void**)&CPU_data_in, sizeof(long long int) * data_size, hipHostMallocDefault));//////////using pinned memory	
		init_cpu_data(CPU_data_in, data_size, data_stride, mod);
		
		long long int reduced_iter = iterations;
		if(reduced_iter > 2048){
			reduced_iter = 2048;
		}else if(reduced_iter < 16){
			reduced_iter = 16;
		}
		
		long long int *CPU_data_out_index;
		CPU_data_out_index = (long long int*)malloc(sizeof(long long int) * reduced_iter);
		long long int *CPU_data_out_time;
		CPU_data_out_time = (long long int*)malloc(sizeof(long long int) * reduced_iter);
		///////////////////////////////////////////////////////////////////CPU data end	
	
		///////////////////////////////////////////////////////////////////GPU data in	
		//int *GPU_data_in;
		//checkCudaErrors(hipMalloc(&GPU_data_in, sizeof(int) * data_size));	
		//hipMemcpy(GPU_data_in, CPU_data_in, sizeof(int) * data_size, hipMemcpyHostToDevice);
		
		///////////////////////////////////////////////////////////////////GPU data out
		long long int *GPU_data_out_index;
		checkCudaErrors(hipMalloc(&GPU_data_out_index, sizeof(long long int) * reduced_iter));
		long long int *GPU_data_out_time;
		checkCudaErrors(hipMalloc(&GPU_data_out_time, sizeof(long long int) * reduced_iter));
		
		tlb_latency_test<<<1, 1>>>(CPU_data_in, iterations, GPU_data_out, GPU_data_out_index, GPU_data_out_time, clock_rate, mod, data_stride);///////////////kernel is here	
		hipDeviceSynchronize();
				
		hipMemcpy(CPU_data_out_index, GPU_data_out_index, sizeof(long long int) * reduced_iter, hipMemcpyDeviceToHost);
		hipMemcpy(CPU_data_out_time, GPU_data_out_time, sizeof(long long int) * reduced_iter, hipMemcpyDeviceToHost);
				

		fprintf(pFile, "###################data_stride%d#########################\n", data_stride);
		fprintf (pFile, "###############Mod%lld##############%lld\n", mod, iterations);
		for (long long int it = 0; it < reduced_iter; it++){		
			fprintf (pFile, "%lld %fms %lldcycles\n", CPU_data_out_index[it], CPU_data_out_time[it] / (float)clock_rate, CPU_data_out_time[it]);
			//fprintf (pFile, "%d %fms\n", it, CPU_data_out_time[it] / (float)clock_rate);
			//printf ("%d %fms\n", CPU_data_out_index[it], CPU_data_out_time[it] / (float)clock_rate);
		}
		fflush(pFile);
		
		checkCudaErrors(hipFree(GPU_data_out_index));
		checkCudaErrors(hipFree(GPU_data_out_time));
		//checkCudaErrors(hipFree(GPU_data_in));
		//checkCudaErrors(hipFree(CPU_data_in));
		checkCudaErrors(hipHostFree(CPU_data_in));//////using pinned memory
		//free(CPU_data_in);
		free(CPU_data_out_index);
		free(CPU_data_out_time);
	}
	
	//memcopy
	fprintf(pFile,"*\n*\n*\n memcopy\n");
	fflush(pFile);
	for(long long int mod2 = 1073741824; mod2 <= 4294967296; mod2 = mod2 * 2){////268435456 = 1gb, 536870912 = 2gb, 1073741824 = 4gb, 2147483648 = 8gb, 4294967296 = 16gb, 8589934592 = 32gb.
		counter++;
		///////////////////////////////////////////////////////////////////CPU data begin
		//int data_size = 2 * 256 * 1024 * 32;/////size = iteration * stride = 32 2mb pages.
		long long int mod = mod2;

		long long int data_size = mod;
		if(data_size < 4194304){//////////data size at least 16mb to prevent L2 prefetch
			data_size = 4194304;
		}	
		//int iterations = data_size / data_stride;
		//int iterations = 1024 * 256 * 8;
		long long int iterations = mod / data_stride;////32 * 32 * 4 / 32 * 2 = 256
	
		long long int *CPU_data_in;
		CPU_data_in = (long long int*)malloc(sizeof(long long int) * data_size);
		init_cpu_data(CPU_data_in, data_size, data_stride, mod);
		
		long long int reduced_iter = iterations;
		if(reduced_iter > 2048){
			reduced_iter = 2048;
		}else if(reduced_iter < 16){
			reduced_iter = 16;
		}
		
		long long int *CPU_data_out_index;
		CPU_data_out_index = (long long int*)malloc(sizeof(long long int) * reduced_iter);
		long long int *CPU_data_out_time;
		CPU_data_out_time = (long long int*)malloc(sizeof(long long int) * reduced_iter);
		///////////////////////////////////////////////////////////////////CPU data end	
	
		///////////////////////////////////////////////////////////////////GPU data in	
		long long int *GPU_data_in;
		checkCudaErrors(hipMalloc(&GPU_data_in, sizeof(long long int) * data_size));	
		hipMemcpy(GPU_data_in, CPU_data_in, sizeof(long long int) * data_size, hipMemcpyHostToDevice);
		
		///////////////////////////////////////////////////////////////////GPU data out
		long long int *GPU_data_out_index;
		checkCudaErrors(hipMalloc(&GPU_data_out_index, sizeof(long long int) * reduced_iter));
		long long int *GPU_data_out_time;
		checkCudaErrors(hipMalloc(&GPU_data_out_time, sizeof(long long int) * reduced_iter));
		
		tlb_latency_test<<<1, 1>>>(GPU_data_in, iterations, GPU_data_out, GPU_data_out_index, GPU_data_out_time, clock_rate, mod, data_stride);///////////////kernel is here	
		hipDeviceSynchronize();
				
		hipMemcpy(CPU_data_out_index, GPU_data_out_index, sizeof(long long int) * reduced_iter, hipMemcpyDeviceToHost);
		hipMemcpy(CPU_data_out_time, GPU_data_out_time, sizeof(long long int) * reduced_iter, hipMemcpyDeviceToHost);
				

		fprintf(pFile, "###################data_stride%d#########################\n", data_stride);
		fprintf (pFile, "###############Mod%lld##############%lld\n", mod, iterations);
		for (long long int it = 0; it < reduced_iter; it++){		
			fprintf (pFile, "%lld %fms %lldcycles\n", CPU_data_out_index[it], CPU_data_out_time[it] / (float)clock_rate, CPU_data_out_time[it]);
			//fprintf (pFile, "%d %fms\n", it, CPU_data_out_time[it] / (float)clock_rate);
			//printf ("%d %fms\n", CPU_data_out_index[it], CPU_data_out_time[it] / (float)clock_rate);
		}
		fflush(pFile);
		
		checkCudaErrors(hipFree(GPU_data_out_index));
		checkCudaErrors(hipFree(GPU_data_out_time));
		checkCudaErrors(hipFree(GPU_data_in));	
		free(CPU_data_in);
		free(CPU_data_out_index);
		free(CPU_data_out_time);
	}
	*/

	fclose (pFile);
	
    exit(EXIT_SUCCESS);
}