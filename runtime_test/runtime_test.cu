#include "hip/hip_runtime.h"
#include <cstdio>
#include <ctime>
#include <vector>
#include <algorithm>
#include <stdlib.h>
// utilities
#include <hip/hip_runtime_api.h>
#include <time.h>
#include <sys/time.h>

/////////////////////////////L1 is enabled. "ALL_CCFLAGS += -Xptxas -dlcm=ca"

void init_cpu_data(long long int* A, long long int size, long long int stride){
	
	for (long long int i = 0; i < size; i++){
		A[i]=1;
	}
	
	/*
	for (long long int i = 0; i < size - stride; i++){
		A[i]=(i + stride);
	}
			
	for (long long int i = size - stride; i < size; i++){
		A[i]=0;
	}
	*/
}

long long unsigned time_diff(timespec start, timespec end){
	struct timespec temp;
	if ((end.tv_nsec - start.tv_nsec) < 0){
		temp.tv_sec = end.tv_sec - start.tv_sec - 1;
		temp.tv_nsec = 1000000000 + end.tv_nsec - start.tv_nsec;
	} 
	else{
		temp.tv_sec = end.tv_sec - start.tv_sec;
		temp.tv_nsec = end.tv_nsec - start.tv_nsec;
	}
	
	long long unsigned time_long long interval_ns = temp.tv_nsec;
	long long unsigned time_long long interval_s = temp.tv_sec;
	time_long long interval_s = time_long long interval_s * 1000000000;
	
	return time_long long interval_s + time_long long interval_ns;
}

__global__ void Page_visitor(long long int *A, long long int *B, long long int data_stride, long long int clock_count){
		
	/*
	long long int index = threadIdx.x;
	
	/////////////////////////////////time
	long long int start_time = 0;//////clock
	long long int end_time = 0;//////clock
	long long int time_long long interval = 0;//////clock
	
	if(index = 0){
		start_time= clock64();
	}
	__syncthreads();
	*/
	
	long long int index = threadIdx.x * data_stride;
	
	long long int value = A[index];
	
	//////////////////////////////////////////////sleep
	long long int start_clock = clock64();
    long long int clock_offset = 0;
    while (clock_offset < clock_count)
    {
        clock_offset = clock64() - start_clock;
    }
    //d_o[0] = clock_offset;
	//////////////////////////////////////////////sleep
	
	B[index] = value;
	
	/*
	__syncthreads();
	/////////////////////////////////time
	if(index = 0){
		start_time= clock64();
		time_long long interval = end_time - start_time;//////clock
	}	
	//B[0] = time_long long interval;
	*/
}

long long int main(long long int argc, char **argv)
{
	prlong long intf("\n");
	
    // set device
    hipDeviceProp_t device_prop;
    //long long int dev_id = findCudaDevice(argc, (const char **) argv);
	long long int dev_id = 0;
    checkCudaErrors(hipGetDeviceProperties(&device_prop, dev_id));
	
	long long int peak_clk = 1;//kHz
	checkCudaErrors(hipDeviceGetAttribute(&peak_clk, hipDeviceAttributeClockRate, dev_id));
	float clock_rate = (float) peak_clk;
	
	prlong long intf("clock_rate:%f\n", clock_rate);

    if (!device_prop.managedMemory) { 
        // This samples requires being run on a device that supports Unified Memory
        fprlong long intf(stderr, "Unified Memory not supported on this device\n");

        exit(EXIT_WAIVED);
    }

    if (device_prop.computeMode == hipComputeModeProhibited)
    {
        // This sample requires being run with a default or process exclusive mode
        fprlong long intf(stderr, "This sample requires a device in either default or process exclusive mode\n");

        exit(EXIT_WAIVED);
    }
	
	if (device_prop.concurrentManagedAccess == 1){
		prlong long intf("This device supports concurrent Managed Access.\n");
    }else{
		prlong long intf("This device does not support concurrent Managed Access.\n");
	}
	
	long long int value1 = 1;
	checkCudaErrors(hipDeviceGetAttribute(&value1, hipDeviceAttributeConcurrentManagedAccess, dev_id));
	prlong long intf("hipDeviceAttributeConcurrentManagedAccess = %d\n", value1);	
	
	//plain managed
	prlong long intf("*\n*\n*\n plain managed\n");
	for(long long int data_stride = 1 * 256 * 1024; data_stride <= 1 * 256 * 1024; data_stride = data_stride * 2){
	for(long long int mod = 4294967296; mod <= 4294967296; mod = mod * 2){////134217728 = 1gb, 268435456 = 2gb, 536870912 = 4gb, 1073741824 = 8gb, 2147483648 = 16gb, 4294967296 = 32gb, 8589934592 = 64gb. (index)
	for(long long int clock_count = 1000; clock_count <= 1000; clock_count = clock_count * 2){
		///////////////////////////////////////////////////////////////////CPU data begin		
		long long int data_size = mod;
		//long long int iterations = mod / data_stride;////32 * 32 * 4 / 32 * 2 = 256
	
		long long int *CPU_data_in;
		//CPU_data_in = (long long int*)malloc(sizeof(long long int) * data_size);
		checkCudaErrors(hipMallocManaged(&CPU_data_in, sizeof(long long int) * data_size));/////////////using unified memory		
		init_cpu_data(CPU_data_in, data_size, data_stride);				
		///////////////////////////////////////////////////////////////////CPU data end	
	
		///////////////////////////////////////////////////////////////////GPU data in	
		//long long int *GPU_data_in;
		//checkCudaErrors(hipMalloc(&GPU_data_in, sizeof(long long int) * data_size));	
		//hipMemcpy(GPU_data_in, CPU_data_in, sizeof(long long int) * data_size, hipMemcpyHostToDevice);
		
		///////////////////////////////////////////////////////////////////GPU data out
		long long int *GPU_data_out;
		//checkCudaErrors(hipMalloc(&GPU_data_out, sizeof(long long int) * 2));
		checkCudaErrors(hipMallocManaged(&GPU_data_out, sizeof(long long int) * data_size));/////////////using unified memory		
				
		/////////////////////////////////time
		struct timespec ts1;
		clock_gettime(CLOCK_REALTIME, &ts1);
  
		Page_visitor<<<8, 2048>>>(CPU_data_in, GPU_data_out, data_stride, clock_count);///////////////kernel is here	
		hipDeviceSynchronize();
				
		/////////////////////////////////time
		struct timespec ts2;
		clock_gettime(CLOCK_REALTIME, &ts2);
		
		prlong long intf("###################data_stride%d#########################clock_count:%lld\n", data_stride, clock_count);
		prlong long intf("*\n*\n*\nruntime:  %lluns\n", time_diff(ts1, ts2));	
		
		//checkCudaErrors(hipFree(GPU_data_in));
		checkCudaErrors(hipFree(CPU_data_in));
		//free(CPU_data_in);
		checkCudaErrors(hipFree(GPU_data_out));
	}
	}
	}
	
	/*
	//memcopy
	prlong long intf("*\n*\n*\n memcopy\n");
	for(long long int data_stride = 1 * 128 * 1024; data_stride <= 2 * 256 * 1024; data_stride = data_stride * 2){
	for(long long int mod = 536870912; mod <= 536870912; mod = mod * 2){////268435456 = 1gb, 536870912 = 2gb, 1073741824 = 4gb, 2147483648 = 8gb, 4294967296 = 16gb, 8589934592 = 32gb.	
	for(long long int clock_count = 1000; clock_count <= 1000; clock_count = clock_count * 2){

		///////////////////////////////////////////////////////////////////CPU data begin		
		long long int data_size = mod;
		//long long int iterations = mod / data_stride;////32 * 32 * 4 / 32 * 2 = 256
	
		long long int *CPU_data_in;
		CPU_data_in = (long long int*)malloc(sizeof(long long int) * data_size);		
		init_cpu_data(CPU_data_in, data_size, data_stride);				
		///////////////////////////////////////////////////////////////////CPU data end	
	
		///////////////////////////////////////////////////////////////////GPU data in	
		long long int *GPU_data_in;
		checkCudaErrors(hipMalloc(&GPU_data_in, sizeof(long long int) * data_size));	
		
		///////////////////////////////////////////////////////////////////GPU data out
		long long int *GPU_data_out;
		checkCudaErrors(hipMalloc(&GPU_data_out, sizeof(long long int) * data_size));
				
		/////////////////////////////////time
		struct timespec ts1;
		clock_gettime(CLOCK_REALTIME, &ts1);

		hipMemcpy(GPU_data_in, CPU_data_in, sizeof(long long int) * data_size, hipMemcpyHostToDevice);
		Page_visitor<<<1, 512>>>(GPU_data_in, GPU_data_out, data_stride, clock_count);///////////////kernel is here	
		hipDeviceSynchronize();
				
		/////////////////////////////////time
		struct timespec ts2;
		clock_gettime(CLOCK_REALTIME, &ts2);
		
		prlong long intf("###################data_stride%d#########################clock_count:%lld\n", data_stride, clock_count);
		prlong long intf("*\n*\n*\nruntime:  %lluns\n", time_diff(ts1, ts2));
		
		checkCudaErrors(hipFree(GPU_data_in));
		//checkCudaErrors(hipFree(CPU_data_in));
		free(CPU_data_in);
		checkCudaErrors(hipFree(GPU_data_out));
	}
	}
	}
	*/
	
    exit(EXIT_SUCCESS);
}