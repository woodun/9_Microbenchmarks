#include "hip/hip_runtime.h"
#include <cstdio>
#include <ctime>
#include <vector>
#include <algorithm>
#include <stdlib.h>
// utilities
#include <hip/hip_runtime_api.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_cooperative_groups.h>

using namespace cooperative_groups;

/////////////////////////////L1 is enabled. "ALL_CCFLAGS += -Xptxas -dlcm=ca"
//////////////large vs small data.
//////test the intra-warp coalescing. also create larger intervals between block, E.g., sm0 and sm 32? & remote address

void init_cpu_data(long long int* A, long long int size, double stride){
	
	for (long long int i = 0; i < size; i++){
		A[i]=1;
	}
	
	/*
	for (long long int i = 0; i < size - stride; i++){
		A[i]=(i + stride);
	}
			
	for (long long int i = size - stride; i < size; i++){
		A[i]=0;
	}
	*/
}

__global__ void gpu_initialization(long long int *A, double data_stride, long long int data_size){			

	long long int index = (blockIdx.x * blockDim.x + threadIdx.x);
	long long int thread_num =  gridDim.x * blockDim.x;
	
	for(long long int it = 0; it < data_size; it = it + thread_num){
		A[index + it]=23;
	}
}

long long unsigned time_diff(timespec start, timespec end){
	struct timespec temp;
	if ((end.tv_nsec - start.tv_nsec) < 0){
		temp.tv_sec = end.tv_sec - start.tv_sec - 1;
		temp.tv_nsec = 1000000000 + end.tv_nsec - start.tv_nsec;
	} 
	else{
		temp.tv_sec = end.tv_sec - start.tv_sec;
		temp.tv_nsec = end.tv_nsec - start.tv_nsec;
	}
	
	long long unsigned time_interval_ns = temp.tv_nsec;
	long long unsigned time_interval_s = temp.tv_sec;
	time_interval_s = time_interval_s * 1000000000;
	
	return time_interval_s + time_interval_ns;
}

#define stride 16

///////////////262144 (2m), 4194304 (32m), 8388608 (64m), 
__global__ void page_visitor(long long int *A1, long long int *B1, double data_stride, long long int clock_count){////long
			
	long long int warp_id = (threadIdx.x + blockIdx.x * blockDim.x) >> 5;
	double temp = (warp_id * 32 + (threadIdx.x % 32) ) * stride;
	if(warp_id == 27){
		temp = (512 * 32 + (threadIdx.x % 32) ) * stride;
	}
	
	//double temp = (blockIdx.x * blockDim.x + threadIdx.x) * stride;
	//double temp = ((blockIdx.x * blockDim.x + threadIdx.x) % 32) * 2 + blockIdx.x * 1;
	long long int index = __double2ll_rd(temp);
	long long int value1;

	if(warp_id == 0 || warp_id == 27){
		if(threadIdx.x % 32 <= clock_count){
			value1 = A1[index];
		
			B1[index] = value1;	
		}
	}
}


///////////long 0 - 31 same core
///////////long 0 - 64 same core
///////////long 0 - 64 different core
///////////mixed 0 - 64 same core
///////////mixed 0 - 64 different core

int main(int argc, char **argv)
{
	printf("\n");
	
    // set device
    hipDeviceProp_t device_prop;
    //long long int dev_id = findCudaDevice(argc, (const char **) argv);
	long long int dev_id = 0;
    checkCudaErrors(hipGetDeviceProperties(&device_prop, dev_id));
	
	//int peak_clk = 1;//kHz
	//checkCudaErrors(hipDeviceGetAttribute(&peak_clk, hipDeviceAttributeClockRate, dev_id));
	//float clock_rate = (float) peak_clk;
	//printf("clock_rate:%f\n", clock_rate);

    if (!device_prop.managedMemory) { 
        // This samples requires being run on a device that supports Unified Memory
        fprintf(stderr, "Unified Memory not supported on this device\n");

        exit(EXIT_WAIVED);
    }

    if (device_prop.computeMode == hipComputeModeProhibited)
    {
        // This sample requires being run with a default or process exclusive mode
        fprintf(stderr, "This sample requires a device in either default or process exclusive mode\n");

        exit(EXIT_WAIVED);
    }
	
	/*
	if (device_prop.concurrentManagedAccess == 1){
		printf("This device supports concurrent Managed Access.\n");
    }else{
		printf("This device does not support concurrent Managed Access.\n");
	}
	*/
	
	int value1 = 1;
	checkCudaErrors(hipDeviceGetAttribute(&value1, hipDeviceAttributeConcurrentManagedAccess, dev_id));
	//printf("hipDeviceAttributeConcurrentManagedAccess = %d\n", value1);	
	
	
	///*
	//printf("############approach\n");
	for(long long int time = 0; time <= 0; time = time + 1){
	//printf("\n####################time: %llu\n", time);
	
	//long long int coverage2 = 0;
	for(long long int coverage = 1; coverage <= 1; coverage = coverage * 2){///////////////8192 is 2m.
		//coverage2++;
		//if(coverage2 == 2){
		//	coverage = 1;
		//}
		//printf("############coverage: %llu\n", coverage);
		
	for(long long int rate = 1; rate <= 1; rate = rate * 2){
		//printf("############rate: %llu\n", rate);
		
	//long long int offset2 = 0;
	//for(long long int offset = 0; offset <= 0; offset = offset * 2){///////8
	for(long long int offset = 0; offset <= 0; offset = offset + 8){
		//offset2++;
		//if(offset2 == 2){
		//	offset = 1;
		//}
	//printf("############offset: %llu\n", offset);
	
	for(long long int factor = 1; factor <= 1; factor = factor * 2){/////////////16384 (128k) max
	//printf("####################factor: %llu\n", factor);
	
	for(double data_stride = 1 * 1 * 1 * factor; data_stride <= 1 * 1 * 1 * factor; data_stride = data_stride * 2){///134217728 = 1gb, 268435456 = 2gb, 536870912 = 4gb, 1073741824 = 8gb, 2147483648 = 16gb, 4294967296 = 32gb, 8589934592 = 64gb. (index)
	//printf("\n");

	for(long long int clock_count = 0; clock_count <= 31; clock_count = clock_count + 1){
		
	///long long int time2 = time;
	//if(time2 > clock_count){
	//	time2 = clock_count;
	//}

		///////////////////////////////////////////////////////////////////CPU data begin
		double temp = data_stride * 512;
		long long int data_size = (long long int) temp;
		//data_size = data_size * 8192 * 512 / factor;
		data_size = data_size * 8192 * 128 / factor;
		
		long long int *CPU_data_in1;
		checkCudaErrors(hipMallocManaged(&CPU_data_in1, sizeof(long long int) * data_size));/////////////using unified memory
		///////////////////////////////////////////////////////////////////CPU data end
		
		long long int *GPU_data_out1;
		checkCudaErrors(hipMallocManaged(&GPU_data_out1, sizeof(long long int) * data_size));/////////////using unified memory
		///////////////////////////////////////////////////////////////////GPU data out	end
		
		if(1){
			double scale = 1;
			if(data_stride < 1){
				scale = data_stride;/////////make sure threadIdx is smaller than data_size in the initialization
			}
			
			gpu_initialization<<<8192 * 128 * scale / factor, 512>>>(GPU_data_out1, data_stride, data_size);///1024 per block max
			hipDeviceSynchronize();
			if(0){
			gpu_initialization<<<8192 * 128 * scale / factor, 512>>>(CPU_data_in1, data_stride, data_size);///1024 per block max
			hipDeviceSynchronize();
			}else{
			init_cpu_data(CPU_data_in1, data_size, data_stride);
			}
		}else{
			init_cpu_data(GPU_data_out1, data_size, data_stride);
			init_cpu_data(CPU_data_in1, data_size, data_stride);		
		}
		
		/////////////////////////////////time
		struct timespec ts1;
		clock_gettime(CLOCK_REALTIME, &ts1);

		int block_num = 1;

		page_visitor<<<block_num, 32>>>(CPU_data_in1, GPU_data_out1, data_stride, clock_count);/////long 
	
		hipDeviceSynchronize();
				
		/////////////////////////////////time
		struct timespec ts2;
		clock_gettime(CLOCK_REALTIME, &ts2);
		
		//printf("###################data_stride%lld#########################clock_count:%lld\n", data_stride, clock_count);
		//printf("*\n*\n*\nruntime:  %lluns\n", time_diff(ts1, ts2));
		printf("%llu ", time_diff(ts1, ts2));
		fflush(stdout);
		
		checkCudaErrors(hipFree(CPU_data_in1));		
		checkCudaErrors(hipFree(GPU_data_out1));
	}
	}
	}
	}
	}
	}
	}
	printf("\n");	
	
	
	for(long long int time = 0; time <= 0; time = time + 1){
	//printf("\n####################time: %llu\n", time);
	
	//long long int coverage2 = 0;
	for(long long int coverage = 1; coverage <= 1; coverage = coverage * 2){///////////////8192 is 2m.
		//coverage2++;
		//if(coverage2 == 2){
		//	coverage = 1;
		//}
		//printf("############coverage: %llu\n", coverage);
		
	for(long long int rate = 1; rate <= 1; rate = rate * 2){
		//printf("############rate: %llu\n", rate);
		
	//long long int offset2 = 0;
	//for(long long int offset = 0; offset <= 0; offset = offset * 2){///////8
	for(long long int offset = 0; offset <= 0; offset = offset + 8){
		//offset2++;
		//if(offset2 == 2){
		//	offset = 1;
		//}
	//printf("############offset: %llu\n", offset);
	
	for(long long int factor = 1; factor <= 1; factor = factor * 2){/////////////16384 (128k) max
	//printf("####################factor: %llu\n", factor);
	
	for(double data_stride = 1 * 1 * 1 * factor; data_stride <= 1 * 1 * 1 * factor; data_stride = data_stride * 2){///134217728 = 1gb, 268435456 = 2gb, 536870912 = 4gb, 1073741824 = 8gb, 2147483648 = 16gb, 4294967296 = 32gb, 8589934592 = 64gb. (index)
	//printf("\n");

	for(long long int clock_count = 0; clock_count <= 31; clock_count = clock_count + 1){
		
	///long long int time2 = time;
	//if(time2 > clock_count){
	//	time2 = clock_count;
	//}

		///////////////////////////////////////////////////////////////////CPU data begin
		double temp = data_stride * 512;
		long long int data_size = (long long int) temp;
		//data_size = data_size * 8192 * 512 / factor;
		data_size = data_size * 8192 * 128 / factor;
		
		long long int *CPU_data_in1;
		checkCudaErrors(hipMallocManaged(&CPU_data_in1, sizeof(long long int) * data_size));/////////////using unified memory
		///////////////////////////////////////////////////////////////////CPU data end
		
		long long int *GPU_data_out1;
		checkCudaErrors(hipMallocManaged(&GPU_data_out1, sizeof(long long int) * data_size));/////////////using unified memory
		///////////////////////////////////////////////////////////////////GPU data out	end
		
		if(1){
			double scale = 1;
			if(data_stride < 1){
				scale = data_stride;/////////make sure threadIdx is smaller than data_size in the initialization
			}
			
			gpu_initialization<<<8192 * 128 * scale / factor, 512>>>(GPU_data_out1, data_stride, data_size);///1024 per block max
			hipDeviceSynchronize();
			if(0){
			gpu_initialization<<<8192 * 128 * scale / factor, 512>>>(CPU_data_in1, data_stride, data_size);///1024 per block max
			hipDeviceSynchronize();
			}else{
			init_cpu_data(CPU_data_in1, data_size, data_stride);
			}
		}else{
			init_cpu_data(GPU_data_out1, data_size, data_stride);
			init_cpu_data(CPU_data_in1, data_size, data_stride);		
		}
		
		/////////////////////////////////time
		struct timespec ts1;
		clock_gettime(CLOCK_REALTIME, &ts1);

		int block_num = 1;

		page_visitor<<<block_num, 1024>>>(CPU_data_in1, GPU_data_out1, data_stride, clock_count);/////long 
	
		hipDeviceSynchronize();
				
		/////////////////////////////////time
		struct timespec ts2;
		clock_gettime(CLOCK_REALTIME, &ts2);
		
		//printf("###################data_stride%lld#########################clock_count:%lld\n", data_stride, clock_count);
		//printf("*\n*\n*\nruntime:  %lluns\n", time_diff(ts1, ts2));
		printf("%llu ", time_diff(ts1, ts2));
		fflush(stdout);
		
		checkCudaErrors(hipFree(CPU_data_in1));		
		checkCudaErrors(hipFree(GPU_data_out1));
	}
	}
	}
	}
	}
	}
	}
	printf("\n");
	
	
	for(long long int time = 0; time <= 0; time = time + 1){
	//printf("\n####################time: %llu\n", time);
	
	//long long int coverage2 = 0;
	for(long long int coverage = 1; coverage <= 1; coverage = coverage * 2){///////////////8192 is 2m.
		//coverage2++;
		//if(coverage2 == 2){
		//	coverage = 1;
		//}
		//printf("############coverage: %llu\n", coverage);
		
	for(long long int rate = 1; rate <= 1; rate = rate * 2){
		//printf("############rate: %llu\n", rate);
		
	//long long int offset2 = 0;
	//for(long long int offset = 0; offset <= 0; offset = offset * 2){///////8
	for(long long int offset = 0; offset <= 0; offset = offset + 8){
		//offset2++;
		//if(offset2 == 2){
		//	offset = 1;
		//}
	//printf("############offset: %llu\n", offset);
	
	for(long long int factor = 1; factor <= 1; factor = factor * 2){/////////////16384 (128k) max
	//printf("####################factor: %llu\n", factor);
	
	for(double data_stride = 1 * 1 * 1 * factor; data_stride <= 1 * 1 * 1 * factor; data_stride = data_stride * 2){///134217728 = 1gb, 268435456 = 2gb, 536870912 = 4gb, 1073741824 = 8gb, 2147483648 = 16gb, 4294967296 = 32gb, 8589934592 = 64gb. (index)
	//printf("\n");

	for(long long int clock_count = 0; clock_count <= 31; clock_count = clock_count + 1){
		
	///long long int time2 = time;
	//if(time2 > clock_count){
	//	time2 = clock_count;
	//}

		///////////////////////////////////////////////////////////////////CPU data begin
		double temp = data_stride * 512;
		long long int data_size = (long long int) temp;
		//data_size = data_size * 8192 * 512 / factor;
		data_size = data_size * 8192 * 128 / factor;
		
		long long int *CPU_data_in1;
		checkCudaErrors(hipMallocManaged(&CPU_data_in1, sizeof(long long int) * data_size));/////////////using unified memory
		///////////////////////////////////////////////////////////////////CPU data end
		
		long long int *GPU_data_out1;
		checkCudaErrors(hipMallocManaged(&GPU_data_out1, sizeof(long long int) * data_size));/////////////using unified memory
		///////////////////////////////////////////////////////////////////GPU data out	end
		
		if(1){
			double scale = 1;
			if(data_stride < 1){
				scale = data_stride;/////////make sure threadIdx is smaller than data_size in the initialization
			}
			
			gpu_initialization<<<8192 * 128 * scale / factor, 512>>>(GPU_data_out1, data_stride, data_size);///1024 per block max
			hipDeviceSynchronize();
			if(0){
			gpu_initialization<<<8192 * 128 * scale / factor, 512>>>(CPU_data_in1, data_stride, data_size);///1024 per block max
			hipDeviceSynchronize();
			}else{
			init_cpu_data(CPU_data_in1, data_size, data_stride);
			}
		}else{
			init_cpu_data(GPU_data_out1, data_size, data_stride);
			init_cpu_data(CPU_data_in1, data_size, data_stride);		
		}
		
		/////////////////////////////////time
		struct timespec ts1;
		clock_gettime(CLOCK_REALTIME, &ts1);

		int block_num = 32;

		page_visitor<<<block_num, 32>>>(CPU_data_in1, GPU_data_out1, data_stride, clock_count);/////long 
	
		hipDeviceSynchronize();
				
		/////////////////////////////////time
		struct timespec ts2;
		clock_gettime(CLOCK_REALTIME, &ts2);
		
		//printf("###################data_stride%lld#########################clock_count:%lld\n", data_stride, clock_count);
		//printf("*\n*\n*\nruntime:  %lluns\n", time_diff(ts1, ts2));
		printf("%llu ", time_diff(ts1, ts2));
		fflush(stdout);
		
		checkCudaErrors(hipFree(CPU_data_in1));		
		checkCudaErrors(hipFree(GPU_data_out1));
	}
	}
	}
	}
	}
	}
	}
	printf("\n");
	
	
	exit(EXIT_SUCCESS);
}