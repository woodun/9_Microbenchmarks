#include "hip/hip_runtime.h"
#include <cstdio>
#include <ctime>
#include <vector>
#include <algorithm>
#include <stdlib.h>
// utilities
#include <hip/hip_runtime_api.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_cooperative_groups.h>

using namespace cooperative_groups;

/////////////////////////////L1 is enabled. "ALL_CCFLAGS += -Xptxas -dlcm=ca"
//////////////large vs small data.
//////creating 2 blocks doing exactly the same thing?

void init_cpu_data(long long int* A, long long int size, double stride){
	
	for (long long int i = 0; i < size; i++){
		A[i]=1;
	}
	
	/*
	for (long long int i = 0; i < size - stride; i++){
		A[i]=(i + stride);
	}
			
	for (long long int i = size - stride; i < size; i++){
		A[i]=0;
	}
	*/
}

__global__ void gpu_initialization(long long int *A, double data_stride, long long int data_size){			

	long long int index = (blockIdx.x * blockDim.x + threadIdx.x);
	long long int thread_num =  gridDim.x * blockDim.x;
	
	for(long long int it = 0; it < data_size; it = it + thread_num){
		A[index + it]=23;
	}
}

long long unsigned time_diff(timespec start, timespec end){
	struct timespec temp;
	if ((end.tv_nsec - start.tv_nsec) < 0){
		temp.tv_sec = end.tv_sec - start.tv_sec - 1;
		temp.tv_nsec = 1000000000 + end.tv_nsec - start.tv_nsec;
	} 
	else{
		temp.tv_sec = end.tv_sec - start.tv_sec;
		temp.tv_nsec = end.tv_nsec - start.tv_nsec;
	}
	
	long long unsigned time_interval_ns = temp.tv_nsec;
	long long unsigned time_interval_s = temp.tv_sec;
	time_interval_s = time_interval_s * 1000000000;
	
	return time_interval_s + time_interval_ns;
}

///////////////262144 (2m), 4194304 (32m), 8388608 (64m), 
__global__ void page_visitor(long long int *A1, long long int *B1, double data_stride, long long int clock_count){////vertical
			
	//thread_block block = this_thread_block();	
	
	//double temp = (blockIdx.x * blockDim.x + threadIdx.x) * 1;
	
	unsigned warpid; 
    asm("mov.u32 %0, %warpid;" : "=r"(warpid));
    
	//double temp = (blockIdx.x * blockDim.x + threadIdx.x) * 8388608;
	double temp = warpid * 4096 * 32 + (threadIdx.x % 16) * 4096;
	//double temp = (threadIdx.x % 32) * 262144;
	long long int index = __double2ll_rd(temp);
	long long int value1;

	value1 = A1[index];

	long long int clock_offset = 0;
    while (clock_offset < clock_count){/////////////////what's the time overhead for addition and multiplication?
        clock_offset++;
		value1 = value1 + threadIdx.x;
    }

	B1[index] = value1;	
} 

int main(int argc, char **argv)
{
	printf("\n");
	
    // set device
    hipDeviceProp_t device_prop;
    //long long int dev_id = findCudaDevice(argc, (const char **) argv);
	long long int dev_id = 0;
    checkCudaErrors(hipGetDeviceProperties(&device_prop, dev_id));
	
	//int peak_clk = 1;//kHz
	//checkCudaErrors(hipDeviceGetAttribute(&peak_clk, hipDeviceAttributeClockRate, dev_id));
	//float clock_rate = (float) peak_clk;
	//printf("clock_rate:%f\n", clock_rate);

    if (!device_prop.managedMemory) { 
        // This samples requires being run on a device that supports Unified Memory
        fprintf(stderr, "Unified Memory not supported on this device\n");

        exit(EXIT_WAIVED);
    }

    if (device_prop.computeMode == hipComputeModeProhibited)
    {
        // This sample requires being run with a default or process exclusive mode
        fprintf(stderr, "This sample requires a device in either default or process exclusive mode\n");

        exit(EXIT_WAIVED);
    }
	
	/*
	if (device_prop.concurrentManagedAccess == 1){
		printf("This device supports concurrent Managed Access.\n");
    }else{
		printf("This device does not support concurrent Managed Access.\n");
	}
	*/
	
	int value1 = 1;
	checkCudaErrors(hipDeviceGetAttribute(&value1, hipDeviceAttributeConcurrentManagedAccess, dev_id));
	//printf("hipDeviceAttributeConcurrentManagedAccess = %d\n", value1);	

	//printf("############approach\n");
	for(long long int time = 32; time <= 32; time = time * 2){
	//printf("\n####################time: %llu\n", time);
	
	//long long int coverage2 = 0;
	for(long long int coverage = 1; coverage <= 1; coverage = coverage * 2){///////////////8192 is 2m.
		//coverage2++;
		//if(coverage2 == 2){
		//	coverage = 1;
		//}
		//printf("############coverage: %llu\n", coverage);
		
	for(long long int rate = 1; rate <= 1; rate = rate * 2){
		//printf("############rate: %llu\n", rate);
		
	//long long int offset2 = 0;
	//for(long long int offset = 0; offset <= 0; offset = offset * 2){///////8
	for(long long int offset = 0; offset <= 0; offset = offset + 8){
		//offset2++;
		//if(offset2 == 2){
		//	offset = 1;
		//}
	//printf("############offset: %llu\n", offset);
	
	for(long long int factor = 8; factor <= 8; factor = factor * 2){/////////////16384 (128k) max
	//printf("####################factor: %llu\n", factor);
	
	for(double data_stride = 1 * 1 * 1 * factor; data_stride <= 1 * 1 * 1 * factor; data_stride = data_stride * 2){///134217728 = 1gb, 268435456 = 2gb, 536870912 = 4gb, 1073741824 = 8gb, 2147483648 = 16gb, 4294967296 = 32gb, 8589934592 = 64gb. (index)
	//printf("\n");

	for(long long int clock_count = 8192; clock_count <= 8192; clock_count = clock_count * 2){
		
	///long long int time2 = time;
	//if(time2 > clock_count){
	//	time2 = clock_count;
	//}

		///////////////////////////////////////////////////////////////////CPU data begin
		double temp = data_stride * 512;
		long long int data_size = (long long int) temp;
		//data_size = data_size * 8192 * 512 / factor;
		data_size = data_size * 8192 * 128 / factor;
		
		long long int *CPU_data_in1;
		checkCudaErrors(hipMallocManaged(&CPU_data_in1, sizeof(long long int) * data_size));/////////////using unified memory
		///////////////////////////////////////////////////////////////////CPU data end
		
		long long int *GPU_data_out1;
		checkCudaErrors(hipMallocManaged(&GPU_data_out1, sizeof(long long int) * data_size));/////////////using unified memory
		///////////////////////////////////////////////////////////////////GPU data out	end
		
		if(1){
			double scale = 1;
			if(data_stride < 1){
				scale = data_stride;/////////make sure threadIdx is smaller than data_size in the initialization
			}
			
			gpu_initialization<<<8192 * 128 * scale / factor, 512>>>(GPU_data_out1, data_stride, data_size);///1024 per block max
			hipDeviceSynchronize();
			//gpu_initialization<<<8192 * 128 * scale / factor, 512>>>(CPU_data_in1, data_stride, data_size);///1024 per block max
			//hipDeviceSynchronize();
			init_cpu_data(CPU_data_in1, data_size, data_stride);
		}else{
			init_cpu_data(GPU_data_out1, data_size, data_stride);
			init_cpu_data(CPU_data_in1, data_size, data_stride);		
		}
		
		/////////////////////////////////time
		struct timespec ts1;
		clock_gettime(CLOCK_REALTIME, &ts1);

		////may want to use more thread to see clock_count effect		
		//page_visitor<<<8192 * 512 / factor, 512>>>(CPU_data_in1, GPU_data_out1, data_stride, clock_count);
		page_visitor<<<1, 64>>>(CPU_data_in1, GPU_data_out1, data_stride, clock_count);		
		hipDeviceSynchronize();
				
		/////////////////////////////////time
		struct timespec ts2;
		clock_gettime(CLOCK_REALTIME, &ts2);
		
		//printf("###################data_stride%lld#########################clock_count:%lld\n", data_stride, clock_count);
		//printf("*\n*\n*\nruntime:  %lluns\n", time_diff(ts1, ts2));
		printf("%llu ", time_diff(ts1, ts2));
		fflush(stdout);
		
		checkCudaErrors(hipFree(CPU_data_in1));		
		checkCudaErrors(hipFree(GPU_data_out1));
	}
	}
	}
	}
	}
	}
	}
	
	exit(EXIT_SUCCESS);
}