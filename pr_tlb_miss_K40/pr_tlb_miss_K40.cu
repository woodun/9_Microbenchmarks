#include "hip/hip_runtime.h"
#include <cstdio>
#include <ctime>
#include <vector>
#include <algorithm>
#include <stdlib.h>
// utilities
#include <hip/hip_runtime_api.h>
#include <time.h>

///////////per request timing. L1 enabled. 

//typedef unsigned char byte;

void init_cpu_data(int* A, long int size, int stride, long int mod){
	for (long int i = 0; i < size; i = i + stride){
		A[i]=(i + stride) % mod;
   	}
}

__device__ void P_chasing0(int mark, int *A, int iterations, int *B, int *C, long long int *D, int starting_index, float clock_rate, int data_stride){	
	
	int j = starting_index;/////make them in the same page, and miss near in cache lines
			
	for (int it = 0; it < iterations; it++){	
		j = A[j];		
	}	
		
	B[0] = j;
}

//////////min page size 4kb = 4096b = 32 * 128.
__device__ void P_chasing1(int mark, int *A, int iterations, int *B, int *C, long long int *D, int starting_index, float clock_rate, int data_stride){	
	
	int j = starting_index;/////make them in the same page, and miss near in cache lines
	
	//long long int start_time = 0;//////clock
	//long long int end_time = 0;//////clock
	//start_time = clock64();//////clock
			
	for (int it = 0; it < iterations; it++){
		j = A[j];
	}
	
	//end_time=clock64();//////clock
	//long long int total_time = end_time - start_time;//////clock
	//printf("inside%d:%fms\n", mark, (total_time / (float)clock_rate) / ((float)iterations));//////clock, average latency //////////the print will flush the L1?! (
	
	B[0] = j;
	//B[1] = (int) total_time;
}

//////////min page size 4kb = 4096b = 32 * 128.
__device__ void P_chasing2(int mark, int *A, long int iterations, int *B, int *C, long long int *D, int starting_index, float clock_rate, int data_stride){//////what is the effect of warmup outside vs inside?
	
	//////shared memory: 0xc000 max (49152 Bytes = 48KB)
	__shared__ long long int s_tvalue[1024 * 4];/////must be enough to contain the number of iterations.
	__shared__ int s_index[1024 * 4];
	//__shared__ int s_index[1];
	
	int j = starting_index;/////make them in the same page, and miss near in cache lines
	//int j = B[0];
	
	long long int start_time = 0;//////clock
	long long int end_time = 0;//////clock
	long long int time_interval = 0;//////clock
	//long long int total_time = end_time - start_time;//////clock
	
	/*		
	for (int it = 0; it < iterations; it++){
		
		start_time = clock64();//////clock		
		j = A[j];
		//s_index[it] = j;
		end_time=clock64();//////clock		
		s_tvalue[it] = end_time - start_time;
	}
	*/
	
		asm(".reg .u64 t1;\n\t"
		".reg .u64 t2;\n\t");
	
	for (long int it = 0; it < iterations; it++){
		
		/*
		asm("mul.wide.u32 	t1, %3, %5;\n\t"	
		"add.u64 	t2, t1, %4;\n\t"		
		"mov.u64 	%0, %clock64;\n\t"		
		"ld.global.u32 	%2, [t2];\n\t"
		"mov.u64 	%1, %clock64;"
		: "=l"(start_time), "=l"(end_time), "=r"(j) : "r"(j), "l"(A), "r"(4));
		*/

		asm("mul.wide.u32 	t1, %2, %4;\n\t"	
		"add.u64 	t2, t1, %3;\n\t"		
		"mov.u64 	%0, %clock64;\n\t"		
		"ld.global.u32 	%1, [t2];\n\t"		
		: "=l"(start_time), "=r"(j) : "r"(j), "l"(A), "r"(4));
		
		s_index[it] = j;////what if without this? ///Then it is not accurate and cannot get the access time at all, due to the ILP. (another way is to use average time, but inevitably containing other instructions:setp, add).
		
		asm volatile ("mov.u64 %0, %clock64;": "=l"(end_time));
		
		time_interval = end_time - start_time;
		//if(it >= 4 * 1024){
		s_tvalue[it] = time_interval;
		//}
	}
	
	//printf("inside%d:%fms\n", mark, (total_time / (float)clock_rate) / ((float)iterations));//////clock, average latency
	
	B[0] = j;
	
	for (long int it = 0; it < iterations; it++){		
		C[it] = s_index[it];
		D[it] = s_tvalue[it];
	}
}

__global__ void tlb_latency_test(int *A, long int iterations, int *B, int *C, long long int *D, float clock_rate, long int mod, int data_stride){
	
	///////////kepler L2 has 48 * 1024 = 49152 cache lines. But we only have 1024 * 4 slots in shared memory.
	//P_chasing1(0, A, iterations + 0, B, C, D, 0, clock_rate, data_stride);////////saturate the L2
	P_chasing2(0, A, iterations, B, C, D, 0, clock_rate, data_stride);////////partially print the data
	
	 __syncthreads();
}

int main(int argc, char **argv)
{
	printf("\n");
	
    // set device
    hipDeviceProp_t device_prop;
    //int dev_id = findCudaDevice(argc, (const char **) argv);
	int dev_id = 0;
    checkCudaErrors(hipGetDeviceProperties(&device_prop, dev_id));
	
	int peak_clk = 1;//kHz
	checkCudaErrors(hipDeviceGetAttribute(&peak_clk, hipDeviceAttributeClockRate, dev_id));
	float clock_rate = (float) peak_clk;
	
	//printf("clock_rate_out_kernel:%f\n", clock_rate);

    if (!device_prop.managedMemory) { 
        // This samples requires being run on a device that supports Unified Memory
        fprintf(stderr, "Unified Memory not supported on this device\n");

        exit(EXIT_WAIVED);
    }

    if (device_prop.computeMode == hipComputeModeProhibited)
    {
        // This sample requires being run with a default or process exclusive mode
        fprintf(stderr, "This sample requires a device in either default or process exclusive mode\n");

        exit(EXIT_WAIVED);
    }
	
	///////////////////////////////////////////////////////////////////GPU data out
	int *GPU_data_out;
	checkCudaErrors(hipMalloc(&GPU_data_out, sizeof(int) * 2));			
	
	FILE * pFile;
    pFile = fopen ("output.txt","w");		
	
	int counter = 0;
	for(int data_stride = 2 * 256 * 1024; data_stride <= 2 * 256 * 1024; data_stride = data_stride + 1){/////////2mb stride
		//printf("###################data_stride%d#########################\n", data_stride);
	//for(int mod = 1024 * 256 * 2; mod > 0; mod = mod - 32 * 1024){/////kepler L2 1.5m
	for(long int mod = 2 * 256 * 1024 * 32; mod <= 2 * 256 * 1024 * 1024 * 8; mod = mod * 2){/////kepler L2 1.5m /////kepler L1 16KB ////////saturate the L1 not L2
		counter++;
		///////////////////////////////////////////////////////////////////CPU data begin
		//int data_size = 2 * 256 * 1024 * 32;/////size = iteration * stride = 32 2mb pages.
		long int data_size = mod;
		//int iterations = data_size / data_stride;
		//int iterations = 1024 * 256 * 8;
		long int iterations = mod / data_stride * 2;////32 * 32 * 4 / 32 * 2 = 256
	
		int *CPU_data_in;
		CPU_data_in = (int*)malloc(sizeof(int) * data_size);	
		init_cpu_data(CPU_data_in, data_size, data_stride, mod);
		
		int *CPU_data_out_index;
		CPU_data_out_index = (int*)malloc(sizeof(int) * iterations);
		long long int *CPU_data_out_time;
		CPU_data_out_time = (long long int*)malloc(sizeof(long long int) * iterations);
		///////////////////////////////////////////////////////////////////CPU data end	
	
		///////////////////////////////////////////////////////////////////GPU data in	
		int *GPU_data_in;
		checkCudaErrors(hipMalloc(&GPU_data_in, sizeof(int) * data_size));	
		hipMemcpy(GPU_data_in, CPU_data_in, sizeof(int) * data_size, hipMemcpyHostToDevice);
		
		///////////////////////////////////////////////////////////////////GPU data out
		int *GPU_data_out_index;
		checkCudaErrors(hipMalloc(&GPU_data_out_index, sizeof(int) * iterations));
		long long int *GPU_data_out_time;
		checkCudaErrors(hipMalloc(&GPU_data_out_time, sizeof(long long int) * iterations));
		
		tlb_latency_test<<<1, 1>>>(GPU_data_in, iterations, GPU_data_out, GPU_data_out_index, GPU_data_out_time, clock_rate, mod, data_stride);///////////////kernel is here	
		hipDeviceSynchronize();
				
		hipMemcpy(CPU_data_out_index, GPU_data_out_index, sizeof(int) * iterations, hipMemcpyDeviceToHost);
		hipMemcpy(CPU_data_out_time, GPU_data_out_time, sizeof(long long int) * iterations, hipMemcpyDeviceToHost);
				
		fprintf(pFile, "###################data_stride%d#########################\n", data_stride);
		fprintf (pFile, "###############Mod%d##############%d\n", mod, mod / (2 * 256 * 1024));
		for (long int it = 0; it < iterations; it++){			
			fprintf (pFile, "%d %fms %lldcycles\n", CPU_data_out_index[it], CPU_data_out_time[it] / (float)clock_rate, CPU_data_out_time[it]);
			//fprintf (pFile, "%d %fms\n", it, CPU_data_out_time[it] / (float)clock_rate);
			//printf ("%d %fms\n", CPU_data_out_index[it], CPU_data_out_time[it] / (float)clock_rate);
		}
		
		checkCudaErrors(hipFree(GPU_data_out_index));
		checkCudaErrors(hipFree(GPU_data_out_time));
		checkCudaErrors(hipFree(GPU_data_in));
		free(CPU_data_in);
		free(CPU_data_out_index);
		free(CPU_data_out_time);
	}
		//printf("############################################\n\n");
	}
			
	checkCudaErrors(hipFree(GPU_data_out));	
	//free(CPU_data_out);
	fclose (pFile);
	
    exit(EXIT_SUCCESS);
}
