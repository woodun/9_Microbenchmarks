#include "hip/hip_runtime.h"
#include <cstdio>
#include <ctime>
#include <vector>
#include <algorithm>
#include <stdlib.h>
// utilities
#include <hip/hip_runtime_api.h>
#include <time.h>

/////////////////////////////saturate L2 with long consecutive data. this one use the method in the paper which initialize the data multiple times. L1 is disabled with "ALL_CCFLAGS += -Xptxas -dlcm=cg"


void init_cpu_data(int* A, int size, int stride, int mod){
	for (int i = 0; i < size; ++i){
		A[i]=(i + stride) % mod;
   	}
}

//////////min page size 4kb = 4096b = 32 * 128.
__device__ void P_chasing(int mark, int *A, int iterations, int *B, int starting_index, float clock_rate, int data_stride){
	
	int k = starting_index;/////make them in the same page, and miss near in cache lines
	for (int it = 0; it < iterations; it++){/////////////warmup
		k = A[k];
	}
	B[0] = k;
	
	int j = starting_index;/////make them in the same page, and miss near in cache lines
	
	long long int start_time = 0;//////clock
	long long int end_time = 0;//////clock
	start_time = clock64();//////clock
			
	for (int it = 0; it < iterations; it++){
		j = A[j];
	}
	
	end_time=clock64();//////clock
	long long int total_time = end_time - start_time;//////clock
	printf("inside%d:%fms\n", mark, (total_time / (float)clock_rate) / ((float)iterations));//////clock, average latency
	
	B[0] = j;
}

__global__ void tlb_latency_test(int *A, int iterations, int *B, float clock_rate, int mod, int data_stride){	
	
	P_chasing(mod, A, iterations, B, 0, clock_rate, data_stride);
	
	 __syncthreads();
}

int main(int argc, char **argv)
{
	printf("\n");
	
    // set device
    hipDeviceProp_t device_prop;
    //int dev_id = findCudaDevice(argc, (const char **) argv);
	int dev_id = 0;
    checkCudaErrors(hipGetDeviceProperties(&device_prop, dev_id));
	
	int peak_clk = 1;//kHz
	checkCudaErrors(hipDeviceGetAttribute(&peak_clk, hipDeviceAttributeClockRate, dev_id));
	float clock_rate = (float) peak_clk;
	
	//printf("clock_rate_out_kernel:%f\n", clock_rate);

    if (!device_prop.managedMemory) { 
        // This samples requires being run on a device that supports Unified Memory
        fprintf(stderr, "Unified Memory not supported on this device\n");

        exit(EXIT_WAIVED);
    }

    if (device_prop.computeMode == hipComputeModeProhibited)
    {
        // This sample requires being run with a default or process exclusive mode
        fprintf(stderr, "This sample requires a device in either default or process exclusive mode\n");

        exit(EXIT_WAIVED);
    }
	
	///////////////////////////////////////////////////////////////////GPU data out
	int *GPU_data_out;
	checkCudaErrors(hipMalloc(&GPU_data_out, sizeof(int) * 1));
	
	printf("################fixing data range, changing stride############################\n");
	//for(int mod = 1024 * 256 * 8; mod > 0; mod = mod / 2){/////volta L2 6m
	//for(int mod = 1024 * 256 * 7 ; mod >= 1024 * 256 * 6; mod = mod - 256 * 128){/////volta L2 6m
	for(int data_stride = 4; data_stride <= 128; data_stride = data_stride * 2){
		printf("###################data_stride%d#########################\n", data_stride);
	//for(int mod = 1024 * 256 * 2; mod > 0; mod = mod - 32 * 1024){/////kepler L2 1.5m
	for(int mod = 1024 * 256 * 6; mod >= 1024 * 256 * 6; mod = mod / 2){/////kepler L2 1.5m //////////////1024 * 4 * 3 /////////8 /////////// 1024 * 256 * 1.5 / 1024 * 4 * 3 / 8 = 4 sets? 
		///////////////////////////////////////////////////////////////////CPU data begin
		int data_size = 512 * 1024 * 30;/////size = iteration * stride = 30 2mb pages.		
		//int iterations = data_size / data_stride;
		int iterations = data_size;
	
		int *CPU_data_in;
		CPU_data_in = (int*)malloc(sizeof(int) * data_size);	
		init_cpu_data(CPU_data_in, data_size, data_stride, mod);
		///////////////////////////////////////////////////////////////////CPU data end	
	
		///////////////////////////////////////////////////////////////////GPU data in	
		int *GPU_data_in;
		checkCudaErrors(hipMalloc(&GPU_data_in, sizeof(int) * data_size));	
		hipMemcpy(GPU_data_in, CPU_data_in, sizeof(int) * data_size, hipMemcpyHostToDevice);
		
		tlb_latency_test<<<1, 1>>>(GPU_data_in, iterations, GPU_data_out, clock_rate, mod, data_stride);//////////////////////////////////////////////kernel is here	
		hipDeviceSynchronize();
		
		checkCudaErrors(hipFree(GPU_data_in));
		free(CPU_data_in);
	}
		printf("############################################\n\n");
	}
	
	printf("\n\n################fixing stride, changing data range############################\n\n");
	//for(int mod = 1024 * 256 * 8; mod > 0; mod = mod / 2){/////volta L2 6m
	//for(int mod = 1024 * 256 * 7 ; mod >= 1024 * 256 * 6; mod = mod - 256 * 128){/////volta L2 6m
	for(int data_stride = 4; data_stride <= 4; data_stride = data_stride * 2){
		printf("###################data_stride%d#########################\n", data_stride);
	for(int mod = 1024 * 256 * 1.5 + 32 * 1024; mod > 1024 * 256 * 1.5 - 8 * 1024; mod = mod - 8 * 1024){/////kepler L2 1.5m
	//for(int mod = 1024 * 256 * 6; mod > 0; mod = mod / 2){/////kepler L2 1.5m //////////////1024 * 256 * 6 / 128 = 1024 * 2 * 6 ///////8 /////// 1024 * 256 * 1.5 / 1024 * 2 * 6 / 8 = 4 sets? 
		///////////////////////////////////////////////////////////////////CPU data begin
		int data_size = 1024 * 512 * 30;/////size = iteration * stride = 30 2mb pages.		
		//int iterations = data_size / data_stride;
		int iterations = data_size;
	
		int *CPU_data_in;
		CPU_data_in = (int*)malloc(sizeof(int) * data_size);	
		init_cpu_data(CPU_data_in, data_size, data_stride, mod);
		///////////////////////////////////////////////////////////////////CPU data end	
	
		///////////////////////////////////////////////////////////////////GPU data in	
		int *GPU_data_in;
		checkCudaErrors(hipMalloc(&GPU_data_in, sizeof(int) * data_size));	
		hipMemcpy(GPU_data_in, CPU_data_in, sizeof(int) * data_size, hipMemcpyHostToDevice);
		
		tlb_latency_test<<<1, 1>>>(GPU_data_in, iterations, GPU_data_out, clock_rate, mod, data_stride);//////////////////////////////////////////////kernel is here	
		hipDeviceSynchronize();
		
		checkCudaErrors(hipFree(GPU_data_in));
		free(CPU_data_in);
	}
		printf("############################################\n\n");
	}
			
	checkCudaErrors(hipFree(GPU_data_out));	
	//free(CPU_data_out);
	
    exit(EXIT_SUCCESS);
}
