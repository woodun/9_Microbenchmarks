#include "hip/hip_runtime.h"
#include <cstdio>
#include <ctime>
#include <vector>
#include <algorithm>
#include <stdlib.h>
// utilities
#include <hip/hip_runtime_api.h>

void init_cpu_data(int* A, int size, int stride){
	for (int i = 0; i < size; ++i){
		A[i]=(i + stride) % size;
   	}
}

__global__ void tlb_latency_test(int *A, int iterations, int *B){
	
	int j = 0;
	
	for (int it =0; it < iterations; it ++){
		j = A[j];
	}	
	
	B[0] = j;
}

int main(int argc, char **argv)
{
    // set device
    hipDeviceProp_t device_prop;
    int dev_id = findCudaDevice(argc, (const char **) argv);
    checkCudaErrors(hipGetDeviceProperties(&device_prop, dev_id));

    if (!device_prop.managedMemory) { 
        // This samples requires being run on a device that supports Unified Memory
        fprintf(stderr, "Unified Memory not supported on this device\n");

        exit(EXIT_WAIVED);
    }

    if (device_prop.computeMode == hipComputeModeProhibited)
    {
        // This sample requires being run with a default or process exclusive mode
        fprintf(stderr, "This sample requires a device in either default or process exclusive mode\n");

        exit(EXIT_WAIVED);
    }
	
	//printf("%d\n", sizeof(int));//////////size of int is 4.
	//exit(0);
	
	//////////////CPU data begin
	////////256 = 1kb, 262144 = 1mb, 524288 = 2mb.
	int iterations = 1000;
	int data_stride = 524288;/////2mb.
	//int data_size = 524288000;/////1000 * 2mb. ##### size = iteration * stride. ##### This can support 1000 iteration. The 1001st iteration starts from head again.
	int data_size = iterations * data_stride;/////size = iteration * stride.
	
	int *CPU_data_in;	
	CPU_data_in = (int*)malloc(sizeof(int) * data_size);
	//int *CPU_data_out;
	//CPU_data_out = (int*)malloc(data_size * sizeof(int));
	
	init_cpu_data(CPU_data_in, data_size, data_stride);
	//////////////CPU data end
	
	//////////////GPU data begin
	int *GPU_data_in;
	//////checkCudaErrors(hipMallocManaged(&data, sizeof(int) * data_size));
	checkCudaErrors(hipMalloc(&GPU_data_in, sizeof(int) * data_size));
	
	int *GPU_data_out;
	checkCudaErrors(hipMalloc(&GPU_data_out, sizeof(int) * 1));
	
	hipMemcpy(GPU_data_in, CPU_data_in, sizeof(int) * data_size, hipMemcpyHostToDevice);
	//////////////GPU data end
		
		
    hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);	

	//////////////kernel begin
	hipEventRecord(start);	
	tlb_latency_test<<<1, 1>>>(GPU_data_in, iterations, GPU_data_out);
	
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	//////////////kernel end
	
	
	//hipMemcpy(CPU_data_out, GPU_data_out, sizeof(int) * data_size, hipMemcpyDeviceToHost);
	
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

    hipDeviceSynchronize();
	
	checkCudaErrors(hipFree(GPU_data_in));
	checkCudaErrors(hipFree(GPU_data_out));
	
	free(CPU_data_in);
	//free(CPU_data_out);

    printf("%f %f\n", milliseconds, milliseconds / 1000);
    exit(EXIT_SUCCESS);
}
