#include "hip/hip_runtime.h"
#include <cstdio>
#include <ctime>
#include <vector>
#include <algorithm>
#include <stdlib.h>
// utilities
#include <hip/hip_runtime_api.h>
#include <time.h>

///////////per request timing. L1 enabled. 
///////////For P100, when using managed memory, L2 always prefetches, and L1 doesn't.

//typedef unsigned char byte;

void init_cpu_data(int* A, int size, int stride, int mod){
	for (int i = 0; i < size; ++i){
		A[i]=(i + stride) % mod;
   	}
}

__device__ void P_chasing0(int mark, int *A, int iterations, int *B, int *C, long long int *D, int starting_index, float clock_rate, int data_stride){	
	
	int j = starting_index;/////make them in the same page, and miss near in cache lines
			
	for (int it = 0; it < iterations; it++){	
		j = A[j];		
	}	
		
	B[0] = j;
}

//////////min page size 4kb = 4096b = 32 * 128.
__device__ void P_chasing1(int mark, int *A, int iterations, int *B, int *C, long long int *D, int starting_index, float clock_rate, int data_stride){	
	
	int j = starting_index;/////make them in the same page, and miss near in cache lines
	
	//long long int start_time = 0;//////clock
	//long long int end_time = 0;//////clock
	//start_time = clock64();//////clock
			
	for (int it = 0; it < iterations; it++){
		j = A[j];
	}
	
	//end_time=clock64();//////clock
	//long long int total_time = end_time - start_time;//////clock
	//printf("inside%d:%fms\n", mark, (total_time / (float)clock_rate) / ((float)iterations));//////clock, average latency //////////the print will flush the L1?! (
	
	B[0] = j;
	//B[1] = (int) total_time;
}

//////////min page size 4kb = 4096b = 32 * 128.
__device__ void P_chasing2(int mark, int *A, long long int iterations, int *B, int *C, long long int *D, int starting_index, float clock_rate, int data_stride){//////what is the effect of warmup outside vs inside?
	
	//////shared memory: 0xc000 max (49152 Bytes = 48KB)
	__shared__ long long int s_tvalue[1024 * 4];/////must be enough to contain the number of iterations.
	__shared__ int s_index[1024 * 4];
	//__shared__ int s_index[1];
	
	int j = starting_index;/////make them in the same page, and miss near in cache lines
	//int j = B[0];
	
	long long int start_time = 0;//////clock
	long long int end_time = 0;//////clock
	long long int time_interval = 0;//////clock
	//long long int total_time = end_time - start_time;//////clock
	
	/*		
	for (int it = 0; it < iterations; it++){
		
		start_time = clock64();//////clock		
		j = A[j];
		//s_index[it] = j;
		end_time=clock64();//////clock		
		s_tvalue[it] = end_time - start_time;
	}
	*/	
	
	asm(".reg .u32 t1;\n\t"
	".reg .u64 t2;\n\t"
	".reg .u32 t3;\n\t"
	".reg .u32 t4;\n\t"
	".reg .u64 t5;\n\t"
	".reg .u32 t6;\n\t"
	".reg .u64 t7;\n\t"
	"cvta.to.shared.u64 	t5, %0;\n\t"
	"cvt.u32.u64 	t6, t5;\n\t"
	:: "l"(s_index));////////////////////////////////////cvta.to.global.u64 	%rd4, %rd25; needed??
	
	for (int it = 0; it < iterations; it++){//////////it here is limited by the size of the shared memory
		
		asm("shl.b32 	t1, %3, 2;\n\t"
		"cvt.u64.u32 	t7, t1;\n\t"
		"add.s64 	t2, t7, %4;\n\t"
		"shl.b32 	t3, %6, 2;\n\t"
		"add.s32 	t4, t3, t6;\n\t"		
		"mov.u64 	%0, %clock64;\n\t"
		"ld.global.u32 	%2, [t2];\n\t"
		"st.shared.u32 	[t4], %2;\n\t"
		"mov.u64	%1, %clock64;"
		: "=l"(start_time), "=l"(end_time), "=r"(j) : "r"(j), "l"(A), "l"(s_index), "r"(it));		
				
		time_interval = end_time - start_time;
		//if(it >= 4 * 1024){
		s_tvalue[it] = time_interval;
		//}
	}
	
	//printf("inside%d:%fms\n", mark, (total_time / (float)clock_rate) / ((float)iterations));//////clock, average latency
	
	B[0] = j;
	
	for (int it = 0; it < iterations; it++){		
		C[it] = s_index[it];
		D[it] = s_tvalue[it];
	}
}

__global__ void tlb_latency_test(int *A, int iterations, int *B, int *C, long long int *D, float clock_rate, int mod, int data_stride){
	
	///////////kepler L2 has 48 * 1024 = 49152 cache lines. But we only have 1024 * 4 slots in shared memory.
	//P_chasing1(0, A, iterations + 0, B, C, D, 0, clock_rate, data_stride);////////saturate the L2
	P_chasing2(0, A, iterations, B, C, D, 0, clock_rate, data_stride);////////partially print the data
	
	 __syncthreads();
}

int main(int argc, char **argv)
{
	printf("\n");
	
    // set device
    hipDeviceProp_t device_prop;
    //int dev_id = findCudaDevice(argc, (const char **) argv);
	int dev_id = 0;
    checkCudaErrors(hipGetDeviceProperties(&device_prop, dev_id));
	
	int peak_clk = 1;//kHz
	checkCudaErrors(hipDeviceGetAttribute(&peak_clk, hipDeviceAttributeClockRate, dev_id));
	float clock_rate = (float) peak_clk;
	
	//printf("clock_rate_out_kernel:%f\n", clock_rate);

    if (!device_prop.managedMemory) { 
        // This samples requires being run on a device that supports Unified Memory
        fprintf(stderr, "Unified Memory not supported on this device\n");

        exit(EXIT_WAIVED);
    }

    if (device_prop.computeMode == hipComputeModeProhibited)
    {
        // This sample requires being run with a default or process exclusive mode
        fprintf(stderr, "This sample requires a device in either default or process exclusive mode\n");

        exit(EXIT_WAIVED);
    }
	
	///////////////////////////////////////////////////////////////////GPU data out
	int *GPU_data_out;
	checkCudaErrors(hipMalloc(&GPU_data_out, sizeof(int) * 2));			
	
	FILE * pFile;
    pFile = fopen ("output.txt","w");		
	
	for(int data_stride = 32; data_stride <= 32; data_stride = data_stride + 1){/////////stride shall be L1 cache line size.
	
	for(int mod = 1024 * 2; mod <= 1024 * 2; mod = mod + 32){/////kepler L2 1.5m /////kepler L1 16KB ////////saturate the L1 not L2
		///////////////////////////////////////////////////////////////////CPU data begin
		int data_size = 1024 * 256 * 4;/////4mb.		
		//int iterations = data_size / data_stride;
		//int iterations = 1024 * 256 * 8;
		int iterations = mod / data_stride;////32 * 32 * 4 / 32 * 2 = 256 ////////we only need to see the first time if it is prefetched or not.
	
		int *CPU_data_in;
		//CPU_data_in = (int*)malloc(sizeof(int) * data_size);
		checkCudaErrors(hipMallocManaged(&CPU_data_in, sizeof(int) * data_size));/////////////using unified memory		
		init_cpu_data(CPU_data_in, data_size, data_stride, mod);
		
		int *CPU_data_out_index;
		CPU_data_out_index = (int*)malloc(sizeof(int) * iterations);
		long long int *CPU_data_out_time;
		CPU_data_out_time = (long long int*)malloc(sizeof(long long int) * iterations);
		///////////////////////////////////////////////////////////////////CPU data end	
	
		///////////////////////////////////////////////////////////////////GPU data in	
		//int *GPU_data_in;
		//checkCudaErrors(hipMalloc(&GPU_data_in, sizeof(int) * data_size));	
		//hipMemcpy(GPU_data_in, CPU_data_in, sizeof(int) * data_size, hipMemcpyHostToDevice);
		
		///////////////////////////////////////////////////////////////////GPU data out
		int *GPU_data_out_index;
		checkCudaErrors(hipMalloc(&GPU_data_out_index, sizeof(int) * iterations));
		long long int *GPU_data_out_time;
		checkCudaErrors(hipMalloc(&GPU_data_out_time, sizeof(long long int) * iterations));
		
		tlb_latency_test<<<1, 1>>>(CPU_data_in, iterations, GPU_data_out, GPU_data_out_index, GPU_data_out_time, clock_rate, mod, data_stride);///////////////kernel is here	
		hipDeviceSynchronize();
				
		hipMemcpy(CPU_data_out_index, GPU_data_out_index, sizeof(int) * iterations, hipMemcpyDeviceToHost);
		hipMemcpy(CPU_data_out_time, GPU_data_out_time, sizeof(long long int) * iterations, hipMemcpyDeviceToHost);
				
		fprintf(pFile, "############data_size%d#########################\n", data_size);
		fprintf(pFile, "###################data_stride%d#########################\n", data_stride);
		fprintf (pFile, "###############Mod%d##############%d\n", mod, (mod - 1024 * 4) / 32);
		for (int it = 0; it < iterations; it++){			
			fprintf (pFile, "%d %fms %lldcycles\n", CPU_data_out_index[it], CPU_data_out_time[it] / (float)clock_rate, CPU_data_out_time[it]);
			//fprintf (pFile, "%d %fms\n", it, CPU_data_out_time[it] / (float)clock_rate);
			//printf ("%d %fms\n", CPU_data_out_index[it], CPU_data_out_time[it] / (float)clock_rate);
		}
		
		checkCudaErrors(hipFree(GPU_data_out_index));
		checkCudaErrors(hipFree(GPU_data_out_time));
		//checkCudaErrors(hipFree(GPU_data_in));
		checkCudaErrors(hipFree(CPU_data_in));
		//free(CPU_data_in);
		free(CPU_data_out_index);
		free(CPU_data_out_time);
	}
	
	for(int mod = 1024 * 2; mod <= 1024 * 2; mod = mod + 32){/////kepler L2 1.5m /////kepler L1 16KB ////////saturate the L1 not L2
		///////////////////////////////////////////////////////////////////CPU data begin
		int data_size = 1024 * 256 * 2;/////2mb.		
		//int iterations = data_size / data_stride;
		//int iterations = 1024 * 256 * 8;
		int iterations = mod / data_stride;////32 * 32 * 4 / 32 * 2 = 256 ////////we only need to see the first time if it is prefetched or not.
	
		int *CPU_data_in;
		//CPU_data_in = (int*)malloc(sizeof(int) * data_size);
		checkCudaErrors(hipMallocManaged(&CPU_data_in, sizeof(int) * data_size));/////////////using unified memory		
		init_cpu_data(CPU_data_in, data_size, data_stride, mod);
		
		int *CPU_data_out_index;
		CPU_data_out_index = (int*)malloc(sizeof(int) * iterations);
		long long int *CPU_data_out_time;
		CPU_data_out_time = (long long int*)malloc(sizeof(long long int) * iterations);
		///////////////////////////////////////////////////////////////////CPU data end	
	
		///////////////////////////////////////////////////////////////////GPU data in	
		//int *GPU_data_in;
		//checkCudaErrors(hipMalloc(&GPU_data_in, sizeof(int) * data_size));	
		//hipMemcpy(GPU_data_in, CPU_data_in, sizeof(int) * data_size, hipMemcpyHostToDevice);
		
		///////////////////////////////////////////////////////////////////GPU data out
		int *GPU_data_out_index;
		checkCudaErrors(hipMalloc(&GPU_data_out_index, sizeof(int) * iterations));
		long long int *GPU_data_out_time;
		checkCudaErrors(hipMalloc(&GPU_data_out_time, sizeof(long long int) * iterations));
		
		tlb_latency_test<<<1, 1>>>(CPU_data_in, iterations, GPU_data_out, GPU_data_out_index, GPU_data_out_time, clock_rate, mod, data_stride);///////////////kernel is here	
		hipDeviceSynchronize();
				
		hipMemcpy(CPU_data_out_index, GPU_data_out_index, sizeof(int) * iterations, hipMemcpyDeviceToHost);
		hipMemcpy(CPU_data_out_time, GPU_data_out_time, sizeof(long long int) * iterations, hipMemcpyDeviceToHost);
				
		fprintf(pFile, "############data_size%d#########################\n", data_size);
		fprintf(pFile, "###################data_stride%d#########################\n", data_stride);
		fprintf (pFile, "###############Mod%d##############%d\n", mod, (mod - 1024 * 4) / 32);
		for (int it = 0; it < iterations; it++){			
			fprintf (pFile, "%d %fms %lldcycles\n", CPU_data_out_index[it], CPU_data_out_time[it] / (float)clock_rate, CPU_data_out_time[it]);
			//fprintf (pFile, "%d %fms\n", it, CPU_data_out_time[it] / (float)clock_rate);
			//printf ("%d %fms\n", CPU_data_out_index[it], CPU_data_out_time[it] / (float)clock_rate);
		}
		
		checkCudaErrors(hipFree(GPU_data_out_index));
		checkCudaErrors(hipFree(GPU_data_out_time));
		//checkCudaErrors(hipFree(GPU_data_in));
		checkCudaErrors(hipFree(CPU_data_in));
		//free(CPU_data_in);
		free(CPU_data_out_index);
		free(CPU_data_out_time);
	}
	
	for(int mod = 1024 * 2; mod <= 1024 * 2; mod = mod + 32){/////kepler L2 1.5m /////kepler L1 16KB ////////saturate the L1 not L2
		///////////////////////////////////////////////////////////////////CPU data begin
		int data_size = 1024 * 256 * 1.5;/////1.5mb.		
		//int iterations = data_size / data_stride;
		//int iterations = 1024 * 256 * 8;
		int iterations = mod / data_stride;////32 * 32 * 4 / 32 * 2 = 256 ////////we only need to see the first time if it is prefetched or not.
	
		int *CPU_data_in;
		//CPU_data_in = (int*)malloc(sizeof(int) * data_size);
		checkCudaErrors(hipMallocManaged(&CPU_data_in, sizeof(int) * data_size));/////////////using unified memory		
		init_cpu_data(CPU_data_in, data_size, data_stride, mod);
		
		int *CPU_data_out_index;
		CPU_data_out_index = (int*)malloc(sizeof(int) * iterations);
		long long int *CPU_data_out_time;
		CPU_data_out_time = (long long int*)malloc(sizeof(long long int) * iterations);
		///////////////////////////////////////////////////////////////////CPU data end	
	
		///////////////////////////////////////////////////////////////////GPU data in	
		//int *GPU_data_in;
		//checkCudaErrors(hipMalloc(&GPU_data_in, sizeof(int) * data_size));	
		//hipMemcpy(GPU_data_in, CPU_data_in, sizeof(int) * data_size, hipMemcpyHostToDevice);
		
		///////////////////////////////////////////////////////////////////GPU data out
		int *GPU_data_out_index;
		checkCudaErrors(hipMalloc(&GPU_data_out_index, sizeof(int) * iterations));
		long long int *GPU_data_out_time;
		checkCudaErrors(hipMalloc(&GPU_data_out_time, sizeof(long long int) * iterations));
		
		tlb_latency_test<<<1, 1>>>(CPU_data_in, iterations, GPU_data_out, GPU_data_out_index, GPU_data_out_time, clock_rate, mod, data_stride);///////////////kernel is here	
		hipDeviceSynchronize();
				
		hipMemcpy(CPU_data_out_index, GPU_data_out_index, sizeof(int) * iterations, hipMemcpyDeviceToHost);
		hipMemcpy(CPU_data_out_time, GPU_data_out_time, sizeof(long long int) * iterations, hipMemcpyDeviceToHost);
				
		fprintf(pFile, "############data_size%d#########################\n", data_size);
		fprintf(pFile, "###################data_stride%d#########################\n", data_stride);
		fprintf (pFile, "###############Mod%d##############%d\n", mod, (mod - 1024 * 4) / 32);
		for (int it = 0; it < iterations; it++){			
			fprintf (pFile, "%d %fms %lldcycles\n", CPU_data_out_index[it], CPU_data_out_time[it] / (float)clock_rate, CPU_data_out_time[it]);
			//fprintf (pFile, "%d %fms\n", it, CPU_data_out_time[it] / (float)clock_rate);
			//printf ("%d %fms\n", CPU_data_out_index[it], CPU_data_out_time[it] / (float)clock_rate);
		}
		
		checkCudaErrors(hipFree(GPU_data_out_index));
		checkCudaErrors(hipFree(GPU_data_out_time));
		//checkCudaErrors(hipFree(GPU_data_in));
		checkCudaErrors(hipFree(CPU_data_in));
		//free(CPU_data_in);
		free(CPU_data_out_index);
		free(CPU_data_out_time);
	}
	
	for(int mod = 1024 * 2; mod <= 1024 * 2; mod = mod + 32){/////kepler L2 1.5m /////kepler L1 16KB ////////saturate the L1 not L2
		///////////////////////////////////////////////////////////////////CPU data begin
		int data_size = 1024 * 256;/////1mb.		
		//int iterations = data_size / data_stride;
		//int iterations = 1024 * 256 * 8;
		int iterations = mod / data_stride;////32 * 32 * 4 / 32 * 2 = 256 ////////we only need to see the first time if it is prefetched or not.
	
		int *CPU_data_in;
		//CPU_data_in = (int*)malloc(sizeof(int) * data_size);
		checkCudaErrors(hipMallocManaged(&CPU_data_in, sizeof(int) * data_size));/////////////using unified memory		
		init_cpu_data(CPU_data_in, data_size, data_stride, mod);
		
		int *CPU_data_out_index;
		CPU_data_out_index = (int*)malloc(sizeof(int) * iterations);
		long long int *CPU_data_out_time;
		CPU_data_out_time = (long long int*)malloc(sizeof(long long int) * iterations);
		///////////////////////////////////////////////////////////////////CPU data end	
	
		///////////////////////////////////////////////////////////////////GPU data in	
		//int *GPU_data_in;
		//checkCudaErrors(hipMalloc(&GPU_data_in, sizeof(int) * data_size));	
		//hipMemcpy(GPU_data_in, CPU_data_in, sizeof(int) * data_size, hipMemcpyHostToDevice);
		
		///////////////////////////////////////////////////////////////////GPU data out
		int *GPU_data_out_index;
		checkCudaErrors(hipMalloc(&GPU_data_out_index, sizeof(int) * iterations));
		long long int *GPU_data_out_time;
		checkCudaErrors(hipMalloc(&GPU_data_out_time, sizeof(long long int) * iterations));
		
		tlb_latency_test<<<1, 1>>>(CPU_data_in, iterations, GPU_data_out, GPU_data_out_index, GPU_data_out_time, clock_rate, mod, data_stride);///////////////kernel is here	
		hipDeviceSynchronize();
				
		hipMemcpy(CPU_data_out_index, GPU_data_out_index, sizeof(int) * iterations, hipMemcpyDeviceToHost);
		hipMemcpy(CPU_data_out_time, GPU_data_out_time, sizeof(long long int) * iterations, hipMemcpyDeviceToHost);
				
		fprintf(pFile, "############data_size%d#########################\n", data_size);
		fprintf(pFile, "###################data_stride%d#########################\n", data_stride);
		fprintf (pFile, "###############Mod%d##############%d\n", mod, (mod - 1024 * 4) / 32);
		for (int it = 0; it < iterations; it++){			
			fprintf (pFile, "%d %fms %lldcycles\n", CPU_data_out_index[it], CPU_data_out_time[it] / (float)clock_rate, CPU_data_out_time[it]);
			//fprintf (pFile, "%d %fms\n", it, CPU_data_out_time[it] / (float)clock_rate);
			//printf ("%d %fms\n", CPU_data_out_index[it], CPU_data_out_time[it] / (float)clock_rate);
		}
		
		checkCudaErrors(hipFree(GPU_data_out_index));
		checkCudaErrors(hipFree(GPU_data_out_time));
		//checkCudaErrors(hipFree(GPU_data_in));
		checkCudaErrors(hipFree(CPU_data_in));
		//free(CPU_data_in);
		free(CPU_data_out_index);
		free(CPU_data_out_time);
	}
	
	for(int mod = 1024 * 2; mod <= 1024 * 2; mod = mod + 32){/////kepler L2 1.5m /////kepler L1 16KB ////////saturate the L1 not L2
		///////////////////////////////////////////////////////////////////CPU data begin
		int data_size = 1024 * 8;/////size = iteration * stride = 30 2mb pages.		
		//int iterations = data_size / data_stride;
		//int iterations = 1024 * 256 * 8;
		int iterations = mod / data_stride;////32 * 32 * 4 / 32 * 2 = 256 ////////we only need to see the first time if it is prefetched or not.
	
		int *CPU_data_in;
		//CPU_data_in = (int*)malloc(sizeof(int) * data_size);
		checkCudaErrors(hipMallocManaged(&CPU_data_in, sizeof(int) * data_size));/////////////using unified memory		
		init_cpu_data(CPU_data_in, data_size, data_stride, mod);
		
		int *CPU_data_out_index;
		CPU_data_out_index = (int*)malloc(sizeof(int) * iterations);
		long long int *CPU_data_out_time;
		CPU_data_out_time = (long long int*)malloc(sizeof(long long int) * iterations);
		///////////////////////////////////////////////////////////////////CPU data end	
	
		///////////////////////////////////////////////////////////////////GPU data in	
		//int *GPU_data_in;
		//checkCudaErrors(hipMalloc(&GPU_data_in, sizeof(int) * data_size));	
		//hipMemcpy(GPU_data_in, CPU_data_in, sizeof(int) * data_size, hipMemcpyHostToDevice);
		
		///////////////////////////////////////////////////////////////////GPU data out
		int *GPU_data_out_index;
		checkCudaErrors(hipMalloc(&GPU_data_out_index, sizeof(int) * iterations));
		long long int *GPU_data_out_time;
		checkCudaErrors(hipMalloc(&GPU_data_out_time, sizeof(long long int) * iterations));
		
		tlb_latency_test<<<1, 1>>>(CPU_data_in, iterations, GPU_data_out, GPU_data_out_index, GPU_data_out_time, clock_rate, mod, data_stride);///////////////kernel is here	
		hipDeviceSynchronize();
				
		hipMemcpy(CPU_data_out_index, GPU_data_out_index, sizeof(int) * iterations, hipMemcpyDeviceToHost);
		hipMemcpy(CPU_data_out_time, GPU_data_out_time, sizeof(long long int) * iterations, hipMemcpyDeviceToHost);
				
		fprintf(pFile, "############data_size%d#########################\n", data_size);
		fprintf(pFile, "###################data_stride%d#########################\n", data_stride);
		fprintf (pFile, "###############Mod%d##############%d\n", mod, (mod - 1024 * 4) / 32);
		for (int it = 0; it < iterations; it++){			
			fprintf (pFile, "%d %fms %lldcycles\n", CPU_data_out_index[it], CPU_data_out_time[it] / (float)clock_rate, CPU_data_out_time[it]);
			//fprintf (pFile, "%d %fms\n", it, CPU_data_out_time[it] / (float)clock_rate);
			//printf ("%d %fms\n", CPU_data_out_index[it], CPU_data_out_time[it] / (float)clock_rate);
		}
		
		checkCudaErrors(hipFree(GPU_data_out_index));
		checkCudaErrors(hipFree(GPU_data_out_time));
		//checkCudaErrors(hipFree(GPU_data_in));
		checkCudaErrors(hipFree(CPU_data_in));
		//free(CPU_data_in);
		free(CPU_data_out_index);
		free(CPU_data_out_time);
	}
	
	for(int mod = 1024 * 2; mod <= 1024 * 2; mod = mod + 32){/////kepler L2 1.5m /////kepler L1 16KB ////////saturate the L1 not L2
		///////////////////////////////////////////////////////////////////CPU data begin
		int data_size = 1024 * 4;/////size = iteration * stride = 30 2mb pages.		
		//int iterations = data_size / data_stride;
		//int iterations = 1024 * 256 * 8;
		int iterations = mod / data_stride;////32 * 32 * 4 / 32 * 2 = 256 ////////we only need to see the first time if it is prefetched or not.
	
		int *CPU_data_in;
		//CPU_data_in = (int*)malloc(sizeof(int) * data_size);
		checkCudaErrors(hipMallocManaged(&CPU_data_in, sizeof(int) * data_size));/////////////using unified memory		
		init_cpu_data(CPU_data_in, data_size, data_stride, mod);
		
		int *CPU_data_out_index;
		CPU_data_out_index = (int*)malloc(sizeof(int) * iterations);
		long long int *CPU_data_out_time;
		CPU_data_out_time = (long long int*)malloc(sizeof(long long int) * iterations);
		///////////////////////////////////////////////////////////////////CPU data end	
	
		///////////////////////////////////////////////////////////////////GPU data in	
		//int *GPU_data_in;
		//checkCudaErrors(hipMalloc(&GPU_data_in, sizeof(int) * data_size));	
		//hipMemcpy(GPU_data_in, CPU_data_in, sizeof(int) * data_size, hipMemcpyHostToDevice);
		
		///////////////////////////////////////////////////////////////////GPU data out
		int *GPU_data_out_index;
		checkCudaErrors(hipMalloc(&GPU_data_out_index, sizeof(int) * iterations));
		long long int *GPU_data_out_time;
		checkCudaErrors(hipMalloc(&GPU_data_out_time, sizeof(long long int) * iterations));
		
		tlb_latency_test<<<1, 1>>>(CPU_data_in, iterations, GPU_data_out, GPU_data_out_index, GPU_data_out_time, clock_rate, mod, data_stride);///////////////kernel is here	
		hipDeviceSynchronize();
				
		hipMemcpy(CPU_data_out_index, GPU_data_out_index, sizeof(int) * iterations, hipMemcpyDeviceToHost);
		hipMemcpy(CPU_data_out_time, GPU_data_out_time, sizeof(long long int) * iterations, hipMemcpyDeviceToHost);
				
		fprintf(pFile, "############data_size%d#########################\n", data_size);
		fprintf(pFile, "###################data_stride%d#########################\n", data_stride);
		fprintf (pFile, "###############Mod%d##############%d\n", mod, (mod - 1024 * 4) / 32);
		for (int it = 0; it < iterations; it++){			
			fprintf (pFile, "%d %fms %lldcycles\n", CPU_data_out_index[it], CPU_data_out_time[it] / (float)clock_rate, CPU_data_out_time[it]);
			//fprintf (pFile, "%d %fms\n", it, CPU_data_out_time[it] / (float)clock_rate);
			//printf ("%d %fms\n", CPU_data_out_index[it], CPU_data_out_time[it] / (float)clock_rate);
		}
		
		checkCudaErrors(hipFree(GPU_data_out_index));
		checkCudaErrors(hipFree(GPU_data_out_time));
		//checkCudaErrors(hipFree(GPU_data_in));
		checkCudaErrors(hipFree(CPU_data_in));
		//free(CPU_data_in);
		free(CPU_data_out_index);
		free(CPU_data_out_time);
	}
	
	for(int mod = 1024 * 2; mod <= 1024 * 2; mod = mod + 32){/////kepler L2 1.5m /////kepler L1 16KB ////////saturate the L1 not L2
		///////////////////////////////////////////////////////////////////CPU data begin
		int data_size = 1024 * 2;/////size = iteration * stride = 30 2mb pages.		
		//int iterations = data_size / data_stride;
		//int iterations = 1024 * 256 * 8;
		int iterations = mod / data_stride;////32 * 32 * 4 / 32 * 2 = 256 ////////we only need to see the first time if it is prefetched or not.
	
		int *CPU_data_in;
		//CPU_data_in = (int*)malloc(sizeof(int) * data_size);
		checkCudaErrors(hipMallocManaged(&CPU_data_in, sizeof(int) * data_size));/////////////using unified memory		
		init_cpu_data(CPU_data_in, data_size, data_stride, mod);
		
		int *CPU_data_out_index;
		CPU_data_out_index = (int*)malloc(sizeof(int) * iterations);
		long long int *CPU_data_out_time;
		CPU_data_out_time = (long long int*)malloc(sizeof(long long int) * iterations);
		///////////////////////////////////////////////////////////////////CPU data end	
	
		///////////////////////////////////////////////////////////////////GPU data in	
		//int *GPU_data_in;
		//checkCudaErrors(hipMalloc(&GPU_data_in, sizeof(int) * data_size));	
		//hipMemcpy(GPU_data_in, CPU_data_in, sizeof(int) * data_size, hipMemcpyHostToDevice);
		
		///////////////////////////////////////////////////////////////////GPU data out
		int *GPU_data_out_index;
		checkCudaErrors(hipMalloc(&GPU_data_out_index, sizeof(int) * iterations));
		long long int *GPU_data_out_time;
		checkCudaErrors(hipMalloc(&GPU_data_out_time, sizeof(long long int) * iterations));
		
		tlb_latency_test<<<1, 1>>>(CPU_data_in, iterations, GPU_data_out, GPU_data_out_index, GPU_data_out_time, clock_rate, mod, data_stride);///////////////kernel is here	
		hipDeviceSynchronize();
				
		hipMemcpy(CPU_data_out_index, GPU_data_out_index, sizeof(int) * iterations, hipMemcpyDeviceToHost);
		hipMemcpy(CPU_data_out_time, GPU_data_out_time, sizeof(long long int) * iterations, hipMemcpyDeviceToHost);
				
		fprintf(pFile, "############data_size%d#########################\n", data_size);
		fprintf(pFile, "###################data_stride%d#########################\n", data_stride);
		fprintf (pFile, "###############Mod%d##############%d\n", mod, (mod - 1024 * 4) / 32);
		for (int it = 0; it < iterations; it++){			
			fprintf (pFile, "%d %fms %lldcycles\n", CPU_data_out_index[it], CPU_data_out_time[it] / (float)clock_rate, CPU_data_out_time[it]);
			//fprintf (pFile, "%d %fms\n", it, CPU_data_out_time[it] / (float)clock_rate);
			//printf ("%d %fms\n", CPU_data_out_index[it], CPU_data_out_time[it] / (float)clock_rate);
		}
		
		checkCudaErrors(hipFree(GPU_data_out_index));
		checkCudaErrors(hipFree(GPU_data_out_time));
		//checkCudaErrors(hipFree(GPU_data_in));
		checkCudaErrors(hipFree(CPU_data_in));
		//free(CPU_data_in);
		free(CPU_data_out_index);
		free(CPU_data_out_time);
	}
		//printf("############################################\n\n");
	}
			
	checkCudaErrors(hipFree(GPU_data_out));	
	//free(CPU_data_out);
	fclose (pFile);
	
    exit(EXIT_SUCCESS);
}
