#include "hip/hip_runtime.h"
#include <cstdio>
#include <ctime>
#include <vector>
#include <algorithm>
#include <stdlib.h>
// utilities
#include <hip/hip_runtime_api.h>
#include <time.h>

///////////per request timing. L1 enabled. 
///////////hipMemAdviseSetAccessedBy is not supported for K40 using dev_id (because hipDeviceAttributeConcurrentManagedAccess = 0).
///////////When using hipCpuDeviceId, both first and second iteration are the same as the hipMemAdviseSetPreferredLocation case.
///////////hipMemAdviseSetPreferredLocation doesn't seem to have noticeable effect on K40.
///////////P.S. The 800s actually happens randomly. Thus it is not another condition.


//typedef unsigned char byte;

void init_cpu_data(int* A, long long int size, int stride, long long int mod){
	for (long long int i = 0; i < size; i = i + stride){
		A[i]=(i + stride) % mod;
   	}
	
	for (long long int i = 32; i < size; i = i + stride){
		A[i]=(i + stride) % mod;
   	}
}

__device__ void P_chasing0(int mark, int *A, int iterations, int *B, int *C, long long int *D, int starting_index, float clock_rate, int data_stride){	
	
	int j = starting_index;/////make them in the same page, and miss near in cache lines
			
	for (int it = 0; it < iterations; it++){	
		j = A[j];		
	}	
		
	B[0] = j;
}

//////////min page size 4kb = 4096b = 32 * 128.
__device__ void P_chasing1(int mark, int *A, long long int iterations, int *B, int *C, long long int *D, int starting_index, float clock_rate, int data_stride){	
	
	int j = starting_index;/////make them in the same page, and miss near in cache lines
	
	//long long int start_time = 0;//////clock
	//long long int end_time = 0;//////clock
	//start_time = clock64();//////clock
			
	for (int it = 0; it < iterations; it++){
		j = A[j];
	}
	
	//end_time=clock64();//////clock
	//long long int total_time = end_time - start_time;//////clock
	//printf("inside%d:%fms\n", mark, (total_time / (float)clock_rate) / ((float)iterations));//////clock, average latency //////////the print will flush the L1?! (
	
	B[0] = j;
	//B[1] = (int) total_time;
}

//////////min page size 4kb = 4096b = 32 * 128.
__device__ void P_chasing2(int mark, int *A, long long int iterations, int *B, int *C, long long int *D, int starting_index, float clock_rate, int data_stride){//////what is the effect of warmup outside vs inside?
	
	//////shared memory: 0xc000 max (49152 Bytes = 48KB)
	__shared__ long long int s_tvalue[1024 * 4];/////must be enough to contain the number of iterations.
	__shared__ int s_index[1024 * 4];
	//__shared__ int s_index[1];
	
	int j = starting_index;/////make them in the same page, and miss near in cache lines
	//int j = B[0];
	
	long long int start_time = 0;//////clock
	long long int end_time = 0;//////clock
	long long int time_interval = 0;//////clock
	//long long int total_time = end_time - start_time;//////clock
	
	/*		
	for (int it = 0; it < iterations; it++){
		
		start_time = clock64();//////clock		
		j = A[j];
		//s_index[it] = j;
		end_time=clock64();//////clock		
		s_tvalue[it] = end_time - start_time;
	}
	*/	
	
	asm(".reg .u32 t1;\n\t"
	".reg .u64 t2;\n\t"
	".reg .u32 t3;\n\t"
	".reg .u32 t4;\n\t"
	".reg .u64 t5;\n\t"
	".reg .u32 t6;\n\t"
	".reg .u64 t7;\n\t"
	"cvta.to.shared.u64 	t5, %0;\n\t"
	"cvt.u32.u64 	t6, t5;\n\t"
	:: "l"(s_index));////////////////////////////////////cvta.to.global.u64 	%rd4, %rd25; needed??
	
	for (int it = 0; it < iterations; it++){//////////it here is limited by the size of the shared memory
		
		asm("shl.b32 	t1, %3, 2;\n\t"
		"cvt.u64.u32 	t7, t1;\n\t"
		"add.s64 	t2, t7, %4;\n\t"
		"shl.b32 	t3, %6, 2;\n\t"
		"add.s32 	t4, t3, t6;\n\t"		
		"mov.u64 	%0, %clock64;\n\t"
		"ld.global.u32 	%2, [t2];\n\t"
		"st.shared.u32 	[t4], %2;\n\t"
		"mov.u64	%1, %clock64;"
		: "=l"(start_time), "=l"(end_time), "=r"(j) : "r"(j), "l"(A), "l"(s_index), "r"(it));		
				
		time_interval = end_time - start_time;
		//if(it >= 4 * 1024){
		s_tvalue[it] = time_interval;
		//}
	}
	
	//printf("inside%d:%fms\n", mark, (total_time / (float)clock_rate) / ((float)iterations));//////clock, average latency
	
	B[0] = j;
	
	for (int it = 0; it < iterations; it++){		
		C[it] = s_index[it];
		D[it] = s_tvalue[it];
	}
}

__global__ void tlb_latency_test(int *A, long long int iterations, int *B, int *C, long long int *D, float clock_rate, long long int mod, int data_stride){
	
	long long int reduced_iter = iterations;
	if(reduced_iter > 4096){
		reduced_iter = 4096;
	}else if(reduced_iter < 16){
		reduced_iter = 16;
	}
	
	///////////kepler L2 has 48 * 1024 = 49152 cache lines. But we only have 1024 * 4 slots in shared memory.
	//P_chasing1(0, A, iterations + 0, B, C, D, 0, clock_rate, data_stride);////////saturate the L2
	P_chasing2(0, A, reduced_iter, B, C, D, 0, clock_rate, data_stride);////////partially print the data
	
	 __syncthreads();
}

int main(int argc, char **argv)
{
	printf("\n");
	
    // set device
    hipDeviceProp_t device_prop;
    //int dev_id = findCudaDevice(argc, (const char **) argv);
	int dev_id = 0;
    checkCudaErrors(hipGetDeviceProperties(&device_prop, dev_id));
	
	int peak_clk = 1;//kHz
	checkCudaErrors(hipDeviceGetAttribute(&peak_clk, hipDeviceAttributeClockRate, dev_id));
	float clock_rate = (float) peak_clk;
	
	//printf("clock_rate_out_kernel:%f\n", clock_rate);

    if (!device_prop.managedMemory) { 
        // This samples requires being run on a device that supports Unified Memory
        fprintf(stderr, "Unified Memory not supported on this device\n");

        exit(EXIT_WAIVED);
    }

    if (device_prop.computeMode == hipComputeModeProhibited)
    {
        // This sample requires being run with a default or process exclusive mode
        fprintf(stderr, "This sample requires a device in either default or process exclusive mode\n");

        exit(EXIT_WAIVED);
    }
	
	if (device_prop.concurrentManagedAccess == 1){
		printf("This device supports concurrent Managed Access.\n");
    }else{
		printf("This device does not support concurrent Managed Access.\n");
	}
	
	int value1 = 1;
	checkCudaErrors(hipDeviceGetAttribute(&value1, hipDeviceAttributeConcurrentManagedAccess, dev_id));
	printf("hipDeviceAttributeConcurrentManagedAccess = %d\n", value1);	
	
	///////////////////////////////////////////////////////////////////GPU data out
	int *GPU_data_out;
	checkCudaErrors(hipMalloc(&GPU_data_out, sizeof(int) * 2));			
	
	FILE * pFile;
    pFile = fopen ("output.txt","w");		
	
	int counter = 0;
	/////////change the data stride as to observe if the latency increase is caused by iteration(cache) or stride(tlb)
	for(int data_stride = 1 * 1 * 1024; data_stride <= 2 * 256 * 1024; data_stride = data_stride * 2){/////////32mb stride
		//data_stride = data_stride + 32;///offset a cache line, trying to cause L2 miss but tlb hit.
		//printf("###################data_stride%d#########################\n", data_stride);
	//for(int mod = 1024 * 256 * 2; mod > 0; mod = mod - 32 * 1024){/////kepler L2 1.5m = 12288 cache lines, L1 16k = 128 cache lines.
	for(long long int mod2 = 1 * 16 * 1024; mod2 <= 2147483648; mod2 = mod2 * 2){////268435456 = 1gb, 536870912 = 2gb, 1073741824 = 4gb, 2147483648 = 8gb, 4294967296 = 16gb.
		counter++;
		///////////////////////////////////////////////////////////////////CPU data begin
		//int data_size = 2 * 256 * 1024 * 32;/////size = iteration * stride = 32 2mb pages.
		long long int mod = mod2;
		if(mod > 2684354560){
			mod = 2684354560;
		}
		long long int data_size = mod;
		if(data_size < 4194304){//////////data size at least 16mb to prevent L2 prefetch
			data_size = 4194304;
		}	
		//int iterations = data_size / data_stride;
		//int iterations = 1024 * 256 * 8;
		long long int iterations = mod / data_stride;////32 * 32 * 4 / 32 * 2 = 256
	
		int *CPU_data_in;
		//CPU_data_in = (int*)malloc(sizeof(int) * data_size);
		checkCudaErrors(hipMallocManaged(&CPU_data_in, sizeof(int) * data_size));/////////////using unified memory
		checkCudaErrors(hipMemAdvise(CPU_data_in, sizeof(int) * data_size, hipMemAdviseSetAccessedBy, dev_id));//////////using hint
		init_cpu_data(CPU_data_in, data_size, data_stride, mod);
		
		long long int reduced_iter = iterations;
		if(reduced_iter > 4096){
			reduced_iter = 4096;
		}else if(reduced_iter < 16){
			reduced_iter = 16;
		}
		
		int *CPU_data_out_index;
		CPU_data_out_index = (int*)malloc(sizeof(int) * reduced_iter);
		long long int *CPU_data_out_time;
		CPU_data_out_time = (long long int*)malloc(sizeof(long long int) * reduced_iter);
		///////////////////////////////////////////////////////////////////CPU data end	
	
		///////////////////////////////////////////////////////////////////GPU data in	
		//int *GPU_data_in;
		//checkCudaErrors(hipMalloc(&GPU_data_in, sizeof(int) * data_size));	
		//hipMemcpy(GPU_data_in, CPU_data_in, sizeof(int) * data_size, hipMemcpyHostToDevice);
		
		///////////////////////////////////////////////////////////////////GPU data out
		int *GPU_data_out_index;
		checkCudaErrors(hipMalloc(&GPU_data_out_index, sizeof(int) * reduced_iter));
		long long int *GPU_data_out_time;
		checkCudaErrors(hipMalloc(&GPU_data_out_time, sizeof(long long int) * reduced_iter));
		
		tlb_latency_test<<<1, 1>>>(CPU_data_in, iterations, GPU_data_out, GPU_data_out_index, GPU_data_out_time, clock_rate, mod, data_stride);///////////////kernel is here	
		hipDeviceSynchronize();
				
		hipMemcpy(CPU_data_out_index, GPU_data_out_index, sizeof(int) * reduced_iter, hipMemcpyDeviceToHost);
		hipMemcpy(CPU_data_out_time, GPU_data_out_time, sizeof(long long int) * reduced_iter, hipMemcpyDeviceToHost);
				

		fprintf(pFile, "###################data_stride%d#########################\n", data_stride);
		fprintf (pFile, "###############Mod%lld##############%lld\n", mod, iterations);
		for (long long int it = 0; it < reduced_iter; it++){		
			fprintf (pFile, "%d %fms %lldcycles\n", CPU_data_out_index[it], CPU_data_out_time[it] / (float)clock_rate, CPU_data_out_time[it]);
			//fprintf (pFile, "%d %fms\n", it, CPU_data_out_time[it] / (float)clock_rate);
			//printf ("%d %fms\n", CPU_data_out_index[it], CPU_data_out_time[it] / (float)clock_rate);
		}
		
		checkCudaErrors(hipFree(GPU_data_out_index));
		checkCudaErrors(hipFree(GPU_data_out_time));
		//checkCudaErrors(hipFree(GPU_data_in));
		checkCudaErrors(hipFree(CPU_data_in));
		//free(CPU_data_in);
		free(CPU_data_out_index);
		free(CPU_data_out_time);
	}
		//printf("############################################\n\n");
	}
			
	checkCudaErrors(hipFree(GPU_data_out));	
	//free(CPU_data_out);
	fclose (pFile);
	
    exit(EXIT_SUCCESS);
}