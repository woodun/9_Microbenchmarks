#include "hip/hip_runtime.h"
#include <cstdio>
#include <ctime>
#include <vector>
#include <algorithm>
#include <stdlib.h>
// utilities
#include <hip/hip_runtime_api.h>
#include <time.h>

///////////per request timing.

//typedef unsigned char byte;

void init_cpu_data(int* A, int size, int stride, int mod){
	for (int i = 0; i < size; ++i){
		A[i]=(i + stride) % mod;
   	}
}

__device__ void P_chasing0(int mark, int *A, int iterations, int *B, int *C, long long int *D, int starting_index, float clock_rate, int data_stride){	
	
	int j = starting_index;/////make them in the same page, and miss near in cache lines
			
	for (int it = 0; it < iterations; it++){	
		j = A[j];		
	}	
		
	B[0] = j;
}

//////////min page size 4kb = 4096b = 32 * 128.
__device__ void P_chasing1(int mark, int *A, int iterations, int *B, int *C, long long int *D, int starting_index, float clock_rate, int data_stride){	
	
	int j = starting_index;/////make them in the same page, and miss near in cache lines
	
	long long int start_time = 0;//////clock
	long long int end_time = 0;//////clock
	start_time = clock64();//////clock
			
	for (int it = 0; it < iterations; it++){
		j = A[j];
	}
	
	end_time=clock64();//////clock
	long long int total_time = end_time - start_time;//////clock
	printf("inside%d:%fms\n", mark, (total_time / (float)clock_rate) / ((float)iterations));//////clock, average latency
	
	B[0] = j;
}

//////////min page size 4kb = 4096b = 32 * 128.
__device__ void P_chasing2(int mark, int *A, int iterations, int *B, int *C, long long int *D, int starting_index, float clock_rate, int data_stride){	
	
	__shared__ long long int s_tvalue[1024 * 2];
	__shared__ int s_index[1024 * 2];
	
	int j = starting_index;/////make them in the same page, and miss near in cache lines
	//int j = B[0];
	
	long long int start_time = 0;//////clock
	long long int end_time = 0;//////clock
	long long int time_interval = 0;//////clock
	//long long int total_time = end_time - start_time;//////clock
	
	/*		
	for (int it = 0; it < iterations; it++){
		
		start_time = clock64();//////clock		
		j = A[j];
		//s_index[it] = j;
		end_time=clock64();//////clock		
		s_tvalue[it] = end_time - start_time;
	}
	*/
	
		asm(".reg .u64 t1;\n\t"
		".reg .u64 t2;\n\t");
	
	for (int it = 0; it < iterations; it++){
		
		/*
		asm("mul.wide.u32 	t1, %3, %5;\n\t"	
		"add.u64 	t2, t1, %4;\n\t"		
		"mov.u64 	%0, %clock64;\n\t"		
		"ld.global.u32 	%2, [t2];\n\t"
		"mov.u64 	%1, %clock64;"
		: "=l"(start_time), "=l"(end_time), "=r"(j) : "r"(j), "l"(A), "r"(4));
		*/

		asm("mul.wide.u32 	t1, %2, %4;\n\t"	
		"add.u64 	t2, t1, %3;\n\t"		
		"mov.u64 	%0, %clock64;\n\t"		
		"ld.global.u32 	%1, [t2];\n\t"		
		: "=l"(start_time), "=r"(j) : "r"(j), "l"(A), "r"(4));
		
		s_index[it] = j;		
		asm volatile ("mov.u64 %0, %clock64;": "=l"(end_time));
		
		time_interval = end_time - start_time;
		//if(it >= 4 * 1024){
		s_tvalue[it] = time_interval;
		//}
	}
	
	//printf("inside%d:%fms\n", mark, (total_time / (float)clock_rate) / ((float)iterations));//////clock, average latency
	
	B[0] = j;
	
	for (int it = 0; it < iterations; it++){		
		C[it] = s_index[it];
		D[it] = s_tvalue[it];
	}
}

__global__ void tlb_latency_test(int *A, int iterations, int *B, int *C, long long int *D, float clock_rate, int mod, int data_stride){
	
	P_chasing0(0, A, iterations, B, C, D, 0, clock_rate, data_stride);
	//P_chasing1(0, A, iterations, B, C, D, 0, clock_rate, data_stride);
	//P_chasing1(0, A, iterations, B, C, D, 0, clock_rate, data_stride);////////saturate the L1 not L2
	//P_chasing1(0, A, iterations, B, C, D, 0, clock_rate, data_stride);////////saturate the L1 not L2
	P_chasing2(0, A, iterations, B, C, D, 0, clock_rate, data_stride);////////saturate the L1 not L2
	
	 __syncthreads();
}

int main(int argc, char **argv)
{
	printf("\n");
	
    // set device
    hipDeviceProp_t device_prop;
    //int dev_id = findCudaDevice(argc, (const char **) argv);
	int dev_id = 0;
    checkCudaErrors(hipGetDeviceProperties(&device_prop, dev_id));
	
	int peak_clk = 1;//kHz
	checkCudaErrors(hipDeviceGetAttribute(&peak_clk, hipDeviceAttributeClockRate, dev_id));
	float clock_rate = (float) peak_clk;
	
	//printf("clock_rate_out_kernel:%f\n", clock_rate);

    if (!device_prop.managedMemory) { 
        // This samples requires being run on a device that supports Unified Memory
        fprintf(stderr, "Unified Memory not supported on this device\n");

        exit(EXIT_WAIVED);
    }

    if (device_prop.computeMode == hipComputeModeProhibited)
    {
        // This sample requires being run with a default or process exclusive mode
        fprintf(stderr, "This sample requires a device in either default or process exclusive mode\n");

        exit(EXIT_WAIVED);
    }
	
	///////////////////////////////////////////////////////////////////GPU data out
	int *GPU_data_out;
	checkCudaErrors(hipMalloc(&GPU_data_out, sizeof(int) * 1));			
	
	FILE * pFile;
    pFile = fopen ("output.txt","w");		
	
	for(int data_stride = 32; data_stride <= 32; data_stride = data_stride + 1){/////////stride shall be L1 cache line size.
		printf("###################data_stride%d#########################\n", data_stride);
	//for(int mod = 1024 * 256 * 2; mod > 0; mod = mod - 32 * 1024){/////kepler L2 1.5m
	for(int mod = 1024 ; mod >= 1024 ; mod = mod / 2){/////kepler L2 1.5m ////////saturate the L1 not L2
		///////////////////////////////////////////////////////////////////CPU data begin
		int data_size = 512 * 1024 * 30;/////size = iteration * stride = 30 2mb pages.		
		//int iterations = data_size / data_stride;
		//int iterations = 1024 * 256 * 8;
		int iterations = mod / data_stride * 2;
	
		int *CPU_data_in;
		CPU_data_in = (int*)malloc(sizeof(int) * data_size);	
		init_cpu_data(CPU_data_in, data_size, data_stride, mod);
		
		int *CPU_data_out_index;
		CPU_data_out_index = (int*)malloc(sizeof(int) * iterations);
		long long int *CPU_data_out_time;
		CPU_data_out_time = (long long int*)malloc(sizeof(long long int) * iterations);
		///////////////////////////////////////////////////////////////////CPU data end	
	
		///////////////////////////////////////////////////////////////////GPU data in	
		int *GPU_data_in;
		checkCudaErrors(hipMalloc(&GPU_data_in, sizeof(int) * data_size));	
		hipMemcpy(GPU_data_in, CPU_data_in, sizeof(int) * data_size, hipMemcpyHostToDevice);
		
		///////////////////////////////////////////////////////////////////GPU data out
		int *GPU_data_out_index;
		checkCudaErrors(hipMalloc(&GPU_data_out_index, sizeof(int) * iterations));
		long long int *GPU_data_out_time;
		checkCudaErrors(hipMalloc(&GPU_data_out_time, sizeof(long long int) * iterations));
		
		tlb_latency_test<<<1, 1>>>(GPU_data_in, iterations, GPU_data_out, GPU_data_out_index, GPU_data_out_time, clock_rate, mod, data_stride);///////////////kernel is here	
		hipDeviceSynchronize();
				
		hipMemcpy(CPU_data_out_index, GPU_data_out_index, sizeof(int) * iterations, hipMemcpyDeviceToHost);
		hipMemcpy(CPU_data_out_time, GPU_data_out_time, sizeof(long long int) * iterations, hipMemcpyDeviceToHost);
				
		for (int it = 0; it < iterations; it++){
			fprintf (pFile, "%d %fms\n", CPU_data_out_index[it], CPU_data_out_time[it] / (float)clock_rate);
			//fprintf (pFile, "%d %fms\n", it, CPU_data_out_time[it] / (float)clock_rate);
		}
		
		checkCudaErrors(hipFree(GPU_data_out_index));
		checkCudaErrors(hipFree(GPU_data_out_time));
		checkCudaErrors(hipFree(GPU_data_in));
		free(CPU_data_in);
		free(CPU_data_out_index);
		free(CPU_data_out_time);
	}
		printf("############################################\n\n");
	}
			
	checkCudaErrors(hipFree(GPU_data_out));	
	//free(CPU_data_out);
	fclose (pFile);
	
    exit(EXIT_SUCCESS);
}
